#include "hip/hip_runtime.h"
#include "Simulation.cuh"
#include "LDPC_Encoder.cuh"
#include "LDPC_Decoder.cuh"
#include "hip/hip_runtime.h"
#include ""

/*
* 仿真函数
* AWGN:AWGNChannel类变量，包含噪声种子等
* 
*/
void Simulation_GPU(AWGNChannel* AWGN, float* sigma_GPU, Simulation* SIM, int* Address_Variablenode, int* Weight_Checknode, int* Weight_Variablenode)
{
	hipError_t cudaStatus;
	hipDeviceProp_t prop;
	int Num_Device;
	int* CodeWord;
	int* CodeWord_GPU;			// 编码所得码字序列,Num_Frames_OneTime_define帧,分配在GPU内存中
	float* Channel_Out_GPU;		// Num_Frames_OneTime_define帧数据经过AWGN信道后的结果,分配在GPU内存中
	float* Channel_Out;
	int* D;						// Num_Frames_OneTime_define帧数据的译码结果+校验结果,分配在CPU内存中
	int ThreadPerBlock, NumBlock;
	int stopflag;
	float TimeGPU;
	LDPCCode* LDPC;


	hipEvent_t GPU_start;			// GPU速率统计参数
	hipEvent_t GPU_stop;
	hipEventCreate(&GPU_start);
	hipEventCreate(&GPU_stop);

	
	// 查找系统中的GPU个数,并指定采用那一块,同时得到该GPU的性能参数
	cudaStatus = hipGetDeviceCount(&Num_Device);
	if (cudaStatus != hipSuccess)	// 没有一块可以用于计算的GPU,则下列所有步骤无法进行
	{
		printf("There is no GPU beyond 1.0, exit!\n");
		//getch();
		exit(0);
	}
	else
	{
		cudaStatus = hipGetDeviceProperties(&prop, Num_Device - 1);	// 选择最后一块GPU用于计算,同时获得它的性能参数
		if (cudaStatus != hipSuccess)	// 没有一块可以用于计算的GPU,则下列所有步骤无法进行
		{
			printf("Cannot get device properties, exit!\n");
			//getch();
			exit(0);
		}
//		printf( "Clock rate:  %d\n", prop.clockRate );
//		printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
	}

	LDPC = (LDPCCode*)malloc(sizeof(LDPCCode));
	if (LDPC == NULL)
	{
		printf("Can not malloc LDPC in main on Host!\n");
		//getch();
		exit(0);
	}
	CodeWord = (int*)malloc(Num_Frames_OneTime * CW_Len * sizeof(int));
	if (CodeWord == NULL)
	{
		printf("Cannot malloc CodeWord in SNR_Simulation_GPU on host, exit!\n");
		//getch();
		exit(0);
	}	
	cudaStatus = hipMalloc((void**)&CodeWord_GPU, Num_Frames_OneTime * CW_Len * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc CodeWord_GPU in SNR_Simulation_GPU on device, exit!\n");
		//getch();
		exit(0);
	}
	cudaStatus = hipMalloc((void**)&Channel_Out_GPU, Num_Frames_OneTime * CW_Len * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc Channel_Out_GPU in SNR_Simulation_GPU on device, exit!\n");
		//getch();
		exit(0);
	}
	Channel_Out = (float*)malloc(Num_Frames_OneTime * CW_Len * sizeof(float));
	if (Channel_Out == NULL)
	{
		printf("Cannot malloc Channel_Out in SNR_Simulation_GPU on host, exit!\n");
		//getch();
		exit(0);
	}
	D = (int*)malloc((CW_Len + 1) * Num_Frames_OneTime * sizeof(int));
	if (D == NULL)
	{
		printf("Cannot malloc D in SNR_Simulation_GPU on host, exit!\n");
		//getch();
		exit(0);
	}
	while (1)
	{
		SIM->num_Frames += Num_Frames_OneTime;

		// 产生码字序列
		if (PN_Message == 0)	// 本版本均用全零序列
		{
			memset(CodeWord, 0, CW_Len * Num_Frames_OneTime * sizeof(int));
			cudaStatus = hipMemset(CodeWord_GPU, 0, Num_Frames_OneTime * CW_Len * sizeof(int));
			if (cudaStatus != hipSuccess)
			{
				printf("hipMemset CodeWord_GPU cannot execute, exit!\n");
				//getch();
				exit(0);
			}
		}
		else if (PN_Message == 1)	// PN序列,需要编码
		{
		}

		// Num_Frames_OneTime_define帧数据依次经过AWGN信道,得到相应的信道输出.其中,各码字的输出穿插在一起.
		ThreadPerBlock = prop.maxThreadsPerBlock;
		NumBlock = (CW_Len % ThreadPerBlock == 0) ? CW_Len / ThreadPerBlock : CW_Len / ThreadPerBlock + 1;//保证够处理	
		if (Add_noise == 1)
		{

			AWGNChannel_CPU(AWGN,Channel_Out,CodeWord);
			hipMemcpy(Channel_Out_GPU, Channel_Out, Num_Frames_OneTime * CW_Len * sizeof(float), hipMemcpyHostToDevice);
		}
		else BPSK << <NumBlock, ThreadPerBlock >> > (Channel_Out_GPU, CodeWord_GPU);
		
		LDPC_Decoder_GPU(D, Channel_Out_GPU, prop, Address_Variablenode, Weight_Checknode, Weight_Variablenode,LDPC);

		// 统计16帧的结果
		stopflag = Statistic(SIM, CodeWord, D, LDPC);
		// if (SIM->num_Frames >= leastTestFrames)
		// {
		// 	exit(0);
		// }
		if (stopflag == 1)
		{
			break;
		}
		//hipDeviceSynchronize();
	}
	hipEventRecord(GPU_stop, 0);
	hipEventSynchronize(GPU_stop);
	hipEventElapsedTime(&TimeGPU, GPU_start, GPU_stop);
	//printf( "Time of GPU for 1 iteration of LDPC code is:  %f us\n", TimeGPU);

	hipEventDestroy(GPU_start);
	hipEventDestroy(GPU_stop);

	free(LDPC);
	free(CodeWord);
	hipFree(CodeWord_GPU);
	hipFree(Channel_Out_GPU);
	free(D);
	free(Channel_Out);
}


/*
* 仿真参数显示函数
*/
void WriteLogo(AWGNChannel* AWGN, Simulation* SIM)
{

	/*将仿真参数打印到屏幕上*/
	printf("*******************Binary LDPC Simulation*******************\n");
	printf("*Author: Lv Yanchen                         Date:2020/9/28\n\n");
	printf("* Message bits' length of LDPC is %d\n", msgLen);
	printf("* Parity bits' length of LDPC is %d\n", parLen);
	printf("* CodeWord length of LDPC is %d\n", CW_Len);
	printf("* H's row is divided into %d blocks, and column divided into %d blocks. Dimension Z is %d\n", J, L, Z);
	printf("* The encoding rate for current LDPC is %f\n", rate);
	if (PN_Message == 0)
	{
		printf("* Information bits are zero sequence, encoder is no need here.\n");
	}
	else if (PN_Message == 1)
	{
		printf("* Information bits are generated by PN sequence.\n");
	}
	printf("* Maximum iterations for LDPC_decoder is %d\n", maxIT);

	if (decoder_method == 0)
	{
		printf("* LDPC decoder use normalized min-sum algorithm!\n");
		//printf("* Optimal factor opt for R in NMS is: %f\n", opt_R);
	}

	if (Add_noise == 0)
	{
		printf("* Not add white gaussin noise on the symbol.\n");
	}
	else if (Add_noise == 1)
	{
		printf("* Add white gaussin noise on the symbol.\n");
	}
	printf("* Initial seeds for each SNR are %d, %d, %d.\n", AWGN->seed[0], AWGN->seed[1], AWGN->seed[2]);

	if (snrtype == 0)
	{
		printf("* The type of SNR is Eb/No\n");
	}
	else if (snrtype == 1)
	{
		printf("* The type of SNR is Es/No\n");
	}
	printf("* Simulation SNR(SNR_start SNR_stop SNR_step) are: %.2f, %.2f, %.2f\n", startSNR, stopSNR, stepSNR);

	printf("* Least error frames to exit the simulation for each SNR is %d.\n", leastErrorFrames);
	printf("* Least test frames to exit the simulation for each SNR is %d.\n", leastTestFrames);
	printf("* Display step is %d.\n", displayStep);

	if (CPU_GPU == 0)
	{
		printf("* Simulation is on CPU.\n");
	}
	else if (CPU_GPU == 1)
	{
		printf("* Simulation is on GPU.\n");
	}

	printf("* %d frames are simulated simultaneously.\n", Num_Frames_OneTime);

	printf("***************************************************************************\n\n\n");
	printf(" SNR   %5s   %5s   %7s    %7s     %7s  %7s   %7s\n", "NTF", "NEF", "FER", "BER", "AverIT", "FER_F", "FER_A");	
}

/*
* 统计函数，统计仿真结果
*/
int Statistic(Simulation* SIM, int* CodeWord_Frames, int* D,LDPCCode *LDPC)
{
	int index0, index1, Length;
	int Error_msgBit[Num_Frames_OneTime];	// ???????д???????λ???????
	Length = (Message_CW == 0) ? msgLen : CW_Len;

	memset(Error_msgBit, 0, Num_Frames_OneTime * sizeof(int));
	for (index0 = 0; index0 < Num_Frames_OneTime; index0++)
	{
		for (index1 = 0; index1 < Length; index1++)
		{
			Error_msgBit[index0] = (D[index1 * Num_Frames_OneTime + index0] != CodeWord_Frames[index1 * Num_Frames_OneTime + index0]) ? Error_msgBit[index0] + 1 : Error_msgBit[index0];
		}
		SIM->num_Error_Bits += Error_msgBit[index0];
		SIM->num_Error_Frames = (Error_msgBit[index0] != 0 || D[index0 + CW_Len * Num_Frames_OneTime] == 0) ? SIM->num_Error_Frames + 1 : SIM->num_Error_Frames;
		SIM->num_Alarm_Frames = (Error_msgBit[index0] == 0 && D[index0 + CW_Len * Num_Frames_OneTime] == 0) ? SIM->num_Alarm_Frames + 1 : SIM->num_Alarm_Frames;
		SIM->num_False_Frames = (Error_msgBit[index0] != 0 && D[index0 + CW_Len * Num_Frames_OneTime] == 1) ? SIM->num_False_Frames + 1 : SIM->num_False_Frames;
		SIM->Total_Iteration += LDPC->iteraTime;
	}
	if (SIM->num_Frames % displayStep == 0)
	{
		SIM->BER = ((double)SIM->num_Error_Bits / (double)(SIM->num_Frames)) / (double)(Length);
		SIM->FER = (double)SIM->num_Error_Frames / (double)SIM->num_Frames;
		SIM->AverageIT = (double)SIM->Total_Iteration / (double)SIM->num_Frames;
		SIM->FER_Alarm = (double)SIM->num_Alarm_Frames / (double)SIM->num_Frames;
		SIM->FER_False = (double)SIM->num_False_Frames / (double)SIM->num_Frames;
		printf(" %.1f %8d  %4d  %6.4e  %6.4e  %.2f  %6.4e %6.4e\n", SIM->SNR, SIM->num_Frames, SIM->num_Error_Frames, SIM->FER, SIM->BER, SIM->AverageIT, SIM->FER_False, SIM->FER_Alarm);
	}

	if (SIM->num_Error_Frames >= leastErrorFrames && SIM->num_Frames >= leastTestFrames)
	{
		SIM->BER = ((double)SIM->num_Error_Bits / (double)(SIM->num_Frames)) / (double)(Length);
		SIM->FER = (double)SIM->num_Error_Frames / (double)SIM->num_Frames;
		SIM->AverageIT = (double)SIM->Total_Iteration / (double)SIM->num_Frames;
		SIM->FER_Alarm = (double)SIM->num_Alarm_Frames / (double)SIM->num_Frames;
		SIM->FER_False = (double)SIM->num_False_Frames / (double)SIM->num_Frames;
		printf(" %.1f %8d  %4d  %6.4e  %6.4e  %.2f  %6.4e %6.4e\n", SIM->SNR, SIM->num_Frames, SIM->num_Error_Frames, SIM->FER, SIM->BER, SIM->AverageIT, SIM->FER_False, SIM->FER_Alarm);
		return 1;
	}
	return 0;
}

/*
H:校验矩阵
Weight_Checknode:按顺序记录每个校验节点的重量，最后一位为最大重量
Weight_Variablenode:按顺序记录每个变量节点的重量，最后一位为最大重量
*/
void Get_H(int* H, int* Weight_Checknode, int* Weight_Variablenode)
{
	int index0, index1;
	char temp[100];
	char file[100];
	FILE* fp_H;
	strcpy(file, "J4");
	//_itoa(J, temp, 10);
	//strcat(file, temp);
	//_itoa(L, temp, 10);
	strcat(file, "_L24");
	//strcat(file, temp);
	//_itoa(Z, temp, 10);
	strcat(file, "_Z96");
	//strcat(file, temp);
	strcat(file, "_BlockH.txt");
	if (NULL == (fp_H = fopen(file, "r")))
	{
		printf("can not open file: %s\n", file);
		getchar();
		exit(0);
	}

	for (index0 = 0; index0 < L *J; index0++)
	{
		fscanf(fp_H, "%d", &index1);
		*(H + index0) = index1;
	}
	fclose(fp_H);

	for (index0 = 0; index0 < J; index0++)
	{
		for (index1 = 0; index1 < L; index1++)
		{
			Weight_Checknode[index0] = (H[index0 * L + index1] != -1) ? Weight_Checknode[index0] + 1 : Weight_Checknode[index0];//????-1??????1???????е?????(У????)
		}
		// 选择最大重量
		Weight_Checknode[J] = (Weight_Checknode[index0] > Weight_Checknode[J]) ? Weight_Checknode[index0] : Weight_Checknode[J];//У????????????
	}


	for (index0 = 0; index0 < L; index0++)
	{
		for (index1 = 0; index1 < J; index1++)
		{
			Weight_Variablenode[index0] = (H[index1 * L + index0] != -1) ? Weight_Variablenode[index0] + 1 : Weight_Variablenode[index0];//????-1??????1???????е?????(???????)
		}
		// 选择最大重量
		Weight_Variablenode[L] = (Weight_Variablenode[index0] > Weight_Variablenode[L]) ? Weight_Variablenode[index0] : Weight_Variablenode[L];//???????????????
	}

	if (Weight_Checknode[J] > maxWeight_checknode || Weight_Checknode[J] < minWeight_checknode)//?????????Χ
	{
		printf("You must input a LDPC code with Weight_Checknode in [%d, %d], exit!\n", minWeight_checknode, maxWeight_checknode);
		//getch();
		exit(0);
	}
	if (Weight_Variablenode[L] > maxWeight_variablenode || Weight_Variablenode[L] < minWeight_variablenode)
	{
		printf("You must input a LDPC code with Weight_variablenode in [%d, %d], exit!\n", minWeight_variablenode, maxWeight_variablenode);
		//getch();
		exit(0);
	}
}

/*
* H:校验矩阵
* Weight_Checknode:校验节点重量
* Weight_Variablenode:变量节点重量
* Address_Variablenode:每个变量节点所对应校验节点的memory_rq的地址
* 校验节点不需要是因为校验节点对应的就是每一行memory_rq，地址是连在一起的
*/
void Transform_H(int* H, int* Weight_Checknode, int* Weight_Variablenode, int* Address_Variablenode)
{
	int index0, index1, index2, index3, index4, position;
	for (index0 = 0; index0 < L; index0++)		// index0为当前所在列
	{
		index2 = 0;
		for (index1 = 0; index1 < J; index1++)	// index1为当前所在行
		{
			if (H[index1 * L + index0] != -1)
			{
				position = 0;	// 变量节点相连的校验节点中,该变量节点的相对位置(每一行1对应的位置)
				for (index3 = 0; index3 < index0; index3++)
				{
					position = (H[index1 * L + index3] != -1) ? position + 1 : position;//每一行第几块非全零矩阵块
				}
				for (index3 = 0; index3 < Z; index3++)//index3(列)，每加一就是统计下一个变量节点的连接关系
				{
					index4 = (((Z - H[index1 * L + index0]) % Z + index3) >= Z) ? (Z - H[index1 * L + index0]) % Z + index3 - Z : index3;//z维方块里，每一列的1在第几行（结合下面式子）
					Address_Variablenode[(index0 * Z + index3) * Weight_Variablenode[L] + index2] = (index1 * Z + index4) * Weight_Checknode[J] + position;//记录每个变量节点（每一列）和哪些点连接（序号为先行后列）
				}
				index2++;//下一行（块）
			}
		}
	}
}