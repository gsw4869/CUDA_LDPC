#include "hip/hip_runtime.h"
#include "Simulation.cuh"
#include "LDPC_Encoder.cuh"
#include "LDPC_Decoder.cuh"
#include "hip/hip_runtime.h"
#include ""

/*
* ���溯��
* AWGN:AWGNChannel������������������ӵ�
* 
*/
void Simulation_GPU(AWGNChannel* AWGN, float* sigma_GPU, Simulation* SIM, int* Address_Variablenode, int* Weight_Checknode, int* Weight_Variablenode)
{
	hipError_t cudaStatus;
	hipDeviceProp_t prop;
	int Num_Device;
	int* CodeWord;
	int* CodeWord_GPU;			// ����������������,Num_Frames_OneTime_define֡,������GPU�ڴ���
	float* Channel_Out_GPU;		// Num_Frames_OneTime_define֡���ݾ���AWGN�ŵ���Ľ��,������GPU�ڴ���
	float* Channel_Out;
	int* D;						// Num_Frames_OneTime_define֡���ݵ�������+У����,������CPU�ڴ���
	int ThreadPerBlock, NumBlock;
	int stopflag;
	float TimeGPU;
	LDPCCode* LDPC;


	hipEvent_t GPU_start;			// GPU����ͳ�Ʋ���
	hipEvent_t GPU_stop;
	hipEventCreate(&GPU_start);
	hipEventCreate(&GPU_stop);

	
	// ����ϵͳ�е�GPU����,��ָ��������һ��,ͬʱ�õ���GPU�����ܲ���
	cudaStatus = hipGetDeviceCount(&Num_Device);
	if (cudaStatus != hipSuccess)	// û��һ��������ڼ����GPU,���������в����޷�����
	{
		printf("There is no GPU beyond 1.0, exit!\n");
		//getch();
		exit(0);
	}
	else
	{
		cudaStatus = hipGetDeviceProperties(&prop, Num_Device - 1);	// ѡ�����һ��GPU���ڼ���,ͬʱ����������ܲ���
		if (cudaStatus != hipSuccess)	// û��һ��������ڼ����GPU,���������в����޷�����
		{
			printf("Cannot get device properties, exit!\n");
			//getch();
			exit(0);
		}
//		printf( "Clock rate:  %d\n", prop.clockRate );
//		printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
	}

	LDPC = (LDPCCode*)malloc(sizeof(LDPCCode));
	if (LDPC == NULL)
	{
		printf("Can not malloc LDPC in main on Host!\n");
		//getch();
		exit(0);
	}
	CodeWord = (int*)malloc(Num_Frames_OneTime * CW_Len * sizeof(int));
	if (CodeWord == NULL)
	{
		printf("Cannot malloc CodeWord in SNR_Simulation_GPU on host, exit!\n");
		//getch();
		exit(0);
	}	
	cudaStatus = hipMalloc((void**)&CodeWord_GPU, Num_Frames_OneTime * CW_Len * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc CodeWord_GPU in SNR_Simulation_GPU on device, exit!\n");
		//getch();
		exit(0);
	}
	cudaStatus = hipMalloc((void**)&Channel_Out_GPU, Num_Frames_OneTime * CW_Len * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc Channel_Out_GPU in SNR_Simulation_GPU on device, exit!\n");
		//getch();
		exit(0);
	}
	Channel_Out = (float*)malloc(Num_Frames_OneTime * CW_Len * sizeof(float));
	if (Channel_Out == NULL)
	{
		printf("Cannot malloc Channel_Out in SNR_Simulation_GPU on host, exit!\n");
		//getch();
		exit(0);
	}
	D = (int*)malloc((CW_Len + 1) * Num_Frames_OneTime * sizeof(int));
	if (D == NULL)
	{
		printf("Cannot malloc D in SNR_Simulation_GPU on host, exit!\n");
		//getch();
		exit(0);
	}
	while (1)
	{
		SIM->num_Frames += Num_Frames_OneTime;

		// ������������
		if (PN_Message == 0)	// ���汾����ȫ������
		{
			memset(CodeWord, 0, CW_Len * Num_Frames_OneTime * sizeof(int));
			cudaStatus = hipMemset(CodeWord_GPU, 0, Num_Frames_OneTime * CW_Len * sizeof(int));
			if (cudaStatus != hipSuccess)
			{
				printf("hipMemset CodeWord_GPU cannot execute, exit!\n");
				//getch();
				exit(0);
			}
		}
		else if (PN_Message == 1)	// PN����,��Ҫ����
		{
		}

		// Num_Frames_OneTime_define֡�������ξ���AWGN�ŵ�,�õ���Ӧ���ŵ����.����,�����ֵ����������һ��.
		ThreadPerBlock = prop.maxThreadsPerBlock;
		NumBlock = (CW_Len % ThreadPerBlock == 0) ? CW_Len / ThreadPerBlock : CW_Len / ThreadPerBlock + 1;//��֤������	
		if (Add_noise == 1)
		{

			AWGNChannel_CPU(AWGN,Channel_Out,CodeWord);
			hipMemcpy(Channel_Out_GPU, Channel_Out, Num_Frames_OneTime * CW_Len * sizeof(float), hipMemcpyHostToDevice);
		}
		else BPSK << <NumBlock, ThreadPerBlock >> > (Channel_Out_GPU, CodeWord_GPU);
		
		LDPC_Decoder_GPU(D, Channel_Out_GPU, prop, Address_Variablenode, Weight_Checknode, Weight_Variablenode,LDPC);

		// ͳ��16֡�Ľ��
		stopflag = Statistic(SIM, CodeWord, D, LDPC);
		// if (SIM->num_Frames >= leastTestFrames)
		// {
		// 	exit(0);
		// }
		if (stopflag == 1)
		{
			break;
		}
		//hipDeviceSynchronize();
	}
	hipEventRecord(GPU_stop, 0);
	hipEventSynchronize(GPU_stop);
	hipEventElapsedTime(&TimeGPU, GPU_start, GPU_stop);
	//printf( "Time of GPU for 1 iteration of LDPC code is:  %f us\n", TimeGPU);

	hipEventDestroy(GPU_start);
	hipEventDestroy(GPU_stop);

	free(LDPC);
	free(CodeWord);
	hipFree(CodeWord_GPU);
	hipFree(Channel_Out_GPU);
	free(D);
	free(Channel_Out);
}


/*
* ���������ʾ����
*/
void WriteLogo(AWGNChannel* AWGN, Simulation* SIM)
{

	/*�����������ӡ����Ļ��*/
	printf("*******************Binary LDPC Simulation*******************\n");
	printf("*Author: Lv Yanchen                         Date:2020/9/28\n\n");
	printf("* Message bits' length of LDPC is %d\n", msgLen);
	printf("* Parity bits' length of LDPC is %d\n", parLen);
	printf("* CodeWord length of LDPC is %d\n", CW_Len);
	printf("* H's row is divided into %d blocks, and column divided into %d blocks. Dimension Z is %d\n", J, L, Z);
	printf("* The encoding rate for current LDPC is %f\n", rate);
	if (PN_Message == 0)
	{
		printf("* Information bits are zero sequence, encoder is no need here.\n");
	}
	else if (PN_Message == 1)
	{
		printf("* Information bits are generated by PN sequence.\n");
	}
	printf("* Maximum iterations for LDPC_decoder is %d\n", maxIT);

	if (decoder_method == 0)
	{
		printf("* LDPC decoder use normalized min-sum algorithm!\n");
		//printf("* Optimal factor opt for R in NMS is: %f\n", opt_R);
	}

	if (Add_noise == 0)
	{
		printf("* Not add white gaussin noise on the symbol.\n");
	}
	else if (Add_noise == 1)
	{
		printf("* Add white gaussin noise on the symbol.\n");
	}
	printf("* Initial seeds for each SNR are %d, %d, %d.\n", AWGN->seed[0], AWGN->seed[1], AWGN->seed[2]);

	if (snrtype == 0)
	{
		printf("* The type of SNR is Eb/No\n");
	}
	else if (snrtype == 1)
	{
		printf("* The type of SNR is Es/No\n");
	}
	printf("* Simulation SNR(SNR_start SNR_stop SNR_step) are: %.2f, %.2f, %.2f\n", startSNR, stopSNR, stepSNR);

	printf("* Least error frames to exit the simulation for each SNR is %d.\n", leastErrorFrames);
	printf("* Least test frames to exit the simulation for each SNR is %d.\n", leastTestFrames);
	printf("* Display step is %d.\n", displayStep);

	if (CPU_GPU == 0)
	{
		printf("* Simulation is on CPU.\n");
	}
	else if (CPU_GPU == 1)
	{
		printf("* Simulation is on GPU.\n");
	}

	printf("* %d frames are simulated simultaneously.\n", Num_Frames_OneTime);

	printf("***************************************************************************\n\n\n");
	printf(" SNR   %5s   %5s   %7s    %7s     %7s  %7s   %7s\n", "NTF", "NEF", "FER", "BER", "AverIT", "FER_F", "FER_A");	
}

/*
* ͳ�ƺ�����ͳ�Ʒ�����
*/
int Statistic(Simulation* SIM, int* CodeWord_Frames, int* D,LDPCCode *LDPC)
{
	int index0, index1, Length;
	int Error_msgBit[Num_Frames_OneTime];	// ???????��???????��???????
	Length = (Message_CW == 0) ? msgLen : CW_Len;

	memset(Error_msgBit, 0, Num_Frames_OneTime * sizeof(int));
	for (index0 = 0; index0 < Num_Frames_OneTime; index0++)
	{
		for (index1 = 0; index1 < Length; index1++)
		{
			Error_msgBit[index0] = (D[index1 * Num_Frames_OneTime + index0] != CodeWord_Frames[index1 * Num_Frames_OneTime + index0]) ? Error_msgBit[index0] + 1 : Error_msgBit[index0];
		}
		SIM->num_Error_Bits += Error_msgBit[index0];
		SIM->num_Error_Frames = (Error_msgBit[index0] != 0 || D[index0 + CW_Len * Num_Frames_OneTime] == 0) ? SIM->num_Error_Frames + 1 : SIM->num_Error_Frames;
		SIM->num_Alarm_Frames = (Error_msgBit[index0] == 0 && D[index0 + CW_Len * Num_Frames_OneTime] == 0) ? SIM->num_Alarm_Frames + 1 : SIM->num_Alarm_Frames;
		SIM->num_False_Frames = (Error_msgBit[index0] != 0 && D[index0 + CW_Len * Num_Frames_OneTime] == 1) ? SIM->num_False_Frames + 1 : SIM->num_False_Frames;
		SIM->Total_Iteration += LDPC->iteraTime;
	}
	if (SIM->num_Frames % displayStep == 0)
	{
		SIM->BER = ((double)SIM->num_Error_Bits / (double)(SIM->num_Frames)) / (double)(Length);
		SIM->FER = (double)SIM->num_Error_Frames / (double)SIM->num_Frames;
		SIM->AverageIT = (double)SIM->Total_Iteration / (double)SIM->num_Frames;
		SIM->FER_Alarm = (double)SIM->num_Alarm_Frames / (double)SIM->num_Frames;
		SIM->FER_False = (double)SIM->num_False_Frames / (double)SIM->num_Frames;
		printf(" %.1f %8d  %4d  %6.4e  %6.4e  %.2f  %6.4e %6.4e\n", SIM->SNR, SIM->num_Frames, SIM->num_Error_Frames, SIM->FER, SIM->BER, SIM->AverageIT, SIM->FER_False, SIM->FER_Alarm);
	}

	if (SIM->num_Error_Frames >= leastErrorFrames && SIM->num_Frames >= leastTestFrames)
	{
		SIM->BER = ((double)SIM->num_Error_Bits / (double)(SIM->num_Frames)) / (double)(Length);
		SIM->FER = (double)SIM->num_Error_Frames / (double)SIM->num_Frames;
		SIM->AverageIT = (double)SIM->Total_Iteration / (double)SIM->num_Frames;
		SIM->FER_Alarm = (double)SIM->num_Alarm_Frames / (double)SIM->num_Frames;
		SIM->FER_False = (double)SIM->num_False_Frames / (double)SIM->num_Frames;
		printf(" %.1f %8d  %4d  %6.4e  %6.4e  %.2f  %6.4e %6.4e\n", SIM->SNR, SIM->num_Frames, SIM->num_Error_Frames, SIM->FER, SIM->BER, SIM->AverageIT, SIM->FER_False, SIM->FER_Alarm);
		return 1;
	}
	return 0;
}

/*
H:У�����
Weight_Checknode:��˳���¼ÿ��У��ڵ�����������һλΪ�������
Weight_Variablenode:��˳���¼ÿ�������ڵ�����������һλΪ�������
*/
void Get_H(int* H, int* Weight_Checknode, int* Weight_Variablenode)
{
	int index0, index1;
	char temp[100];
	char file[100];
	FILE* fp_H;
	strcpy(file, "J4");
	//_itoa(J, temp, 10);
	//strcat(file, temp);
	//_itoa(L, temp, 10);
	strcat(file, "_L24");
	//strcat(file, temp);
	//_itoa(Z, temp, 10);
	strcat(file, "_Z96");
	//strcat(file, temp);
	strcat(file, "_BlockH.txt");
	if (NULL == (fp_H = fopen(file, "r")))
	{
		printf("can not open file: %s\n", file);
		getchar();
		exit(0);
	}

	for (index0 = 0; index0 < L *J; index0++)
	{
		fscanf(fp_H, "%d", &index1);
		*(H + index0) = index1;
	}
	fclose(fp_H);

	for (index0 = 0; index0 < J; index0++)
	{
		for (index1 = 0; index1 < L; index1++)
		{
			Weight_Checknode[index0] = (H[index0 * L + index1] != -1) ? Weight_Checknode[index0] + 1 : Weight_Checknode[index0];//????-1??????1???????��?????(��????)
		}
		// ѡ���������
		Weight_Checknode[J] = (Weight_Checknode[index0] > Weight_Checknode[J]) ? Weight_Checknode[index0] : Weight_Checknode[J];//��????????????
	}


	for (index0 = 0; index0 < L; index0++)
	{
		for (index1 = 0; index1 < J; index1++)
		{
			Weight_Variablenode[index0] = (H[index1 * L + index0] != -1) ? Weight_Variablenode[index0] + 1 : Weight_Variablenode[index0];//????-1??????1???????��?????(???????)
		}
		// ѡ���������
		Weight_Variablenode[L] = (Weight_Variablenode[index0] > Weight_Variablenode[L]) ? Weight_Variablenode[index0] : Weight_Variablenode[L];//???????????????
	}

	if (Weight_Checknode[J] > maxWeight_checknode || Weight_Checknode[J] < minWeight_checknode)//?????????��
	{
		printf("You must input a LDPC code with Weight_Checknode in [%d, %d], exit!\n", minWeight_checknode, maxWeight_checknode);
		//getch();
		exit(0);
	}
	if (Weight_Variablenode[L] > maxWeight_variablenode || Weight_Variablenode[L] < minWeight_variablenode)
	{
		printf("You must input a LDPC code with Weight_variablenode in [%d, %d], exit!\n", minWeight_variablenode, maxWeight_variablenode);
		//getch();
		exit(0);
	}
}

/*
* H:У�����
* Weight_Checknode:У��ڵ�����
* Weight_Variablenode:�����ڵ�����
* Address_Variablenode:ÿ�������ڵ�����ӦУ��ڵ��memory_rq�ĵ�ַ
* У��ڵ㲻��Ҫ����ΪУ��ڵ��Ӧ�ľ���ÿһ��memory_rq����ַ������һ���
*/
void Transform_H(int* H, int* Weight_Checknode, int* Weight_Variablenode, int* Address_Variablenode)
{
	int index0, index1, index2, index3, index4, position;
	for (index0 = 0; index0 < L; index0++)		// index0Ϊ��ǰ������
	{
		index2 = 0;
		for (index1 = 0; index1 < J; index1++)	// index1Ϊ��ǰ������
		{
			if (H[index1 * L + index0] != -1)
			{
				position = 0;	// �����ڵ�������У��ڵ���,�ñ����ڵ�����λ��(ÿһ��1��Ӧ��λ��)
				for (index3 = 0; index3 < index0; index3++)
				{
					position = (H[index1 * L + index3] != -1) ? position + 1 : position;//ÿһ�еڼ����ȫ������
				}
				for (index3 = 0; index3 < Z; index3++)//index3(��)��ÿ��һ����ͳ����һ�������ڵ�����ӹ�ϵ
				{
					index4 = (((Z - H[index1 * L + index0]) % Z + index3) >= Z) ? (Z - H[index1 * L + index0]) % Z + index3 - Z : index3;//zά�����ÿһ�е�1�ڵڼ��У��������ʽ�ӣ�
					Address_Variablenode[(index0 * Z + index3) * Weight_Variablenode[L] + index2] = (index1 * Z + index4) * Weight_Checknode[J] + position;//��¼ÿ�������ڵ㣨ÿһ�У�����Щ�����ӣ����Ϊ���к��У�
				}
				index2++;//��һ�У��飩
			}
		}
	}
}