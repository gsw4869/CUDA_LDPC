#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
//#include <conio.h>
#include <string.h>
#include <memory.h>
#include <time.h>
//#include <direct.h>
#include "define.cuh"
#include "struct.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.cuh"

/*
* D:һ�����
* Channel_Out:����AWGN�ŵ����ź�
* Weight_Checknode:У��ڵ�����
* Weight_Variablenode:�����ڵ�����
* Address_Variablenode:ÿ�������ڵ�����ӦУ��ڵ��memory_rq�ĵ�ַ
* LDPC:��������
*/
void LDPC_Decoder_GPU(int* D, float* Channel_Out, hipDeviceProp_t prop, int* Address_Variablenode, int* Weight_Checknode, int* Weight_Variablenode, LDPCCode *LDPC)
{
	hipError_t cudaStatus;
	int index0, index1, Length;
	int ThreadPerBlock, Num_Block;
	float* Memory_RQ;
	int* Weight_Checknode_GPU, *Weight_Variablenode_GPU;
	int* D_GPU;
	hipEvent_t GPU_start;			// GPU����ͳ�Ʋ���
	hipEvent_t GPU_stop;
	hipEventCreate(&GPU_start);
	hipEventCreate(&GPU_stop);

	Length = (Message_CW == 0) ? msgLen : CW_Len;

	cudaStatus = hipMalloc((void**)&Memory_RQ, parLen * Weight_Checknode[J] * Num_Frames_OneTime * sizeof(float));	// ������GPU��global memory��
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc Memory_RQ in LDPC_Decoder_GPU on device, exit!\n");
		//getch();
		exit(0);
	}
	cudaStatus = hipMalloc((void**)&D_GPU, (CW_Len + 1) * Num_Frames_OneTime * sizeof(int));		// ������GPU��global memory��
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc D_GPU in LDPC_Decoder_GPU on device, exit!\n");
		//getch();
		exit(0);
	}
	cudaStatus = hipMalloc((void**)&Weight_Checknode_GPU, (J + 1) * sizeof(int));		// ������GPU��global memory��
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc Weight_Checknode_GPU in LDPC_Decoder_GPU on device, exit!\n");
		//getch();
		exit(0);
	}
	cudaStatus = hipMalloc((void**)&Weight_Variablenode_GPU, (L + 1) * sizeof(int));		// ������GPU��global memory��
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc Weight_Checknode_GPU in LDPC_Decoder_GPU on device, exit!\n");
		//getch();
		exit(0);
	}
	cudaStatus = hipMemcpy(Weight_Checknode_GPU, Weight_Checknode, (J + 1) * sizeof(int), hipMemcpyHostToDevice);//J�� L�У����һ���?�������?
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Weight_Checknode to Weight_Checknode_GPU in LDPC_Decoder_GPU, exit!\n");
		//getch();
		exit(0);
	}
	cudaStatus = hipMemcpy(Weight_Variablenode_GPU, Weight_Variablenode, (L + 1) * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Weight_Variablenode to Weight_Variablenode_GPU in LDPC_Decoder_GPU, exit!\n");
		//getch();
		exit(0);
	}

	// ��ʼ��
	cudaStatus = hipMemset(Memory_RQ, 0, parLen * Weight_Checknode[J] * Num_Frames_OneTime * sizeof(float));	// �洢������,Ϊ��һ�ε�����׼��,parlenУ��λ���ȣ�J*Z��
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot memset Memory_RQ in LDPC_Decoder_GPU on device, exit!\n");
		//getch();
		exit(0);
	}

	LDPC->iteraTime = 0;

	hipDeviceSynchronize();

	while (LDPC->iteraTime < maxIT)
	{
		LDPC->iteraTime = LDPC->iteraTime + 1;		
		if ((Z * Num_Frames_OneTime) % prop.maxThreadsPerBlock == 0)	// ��ʱ����prop.maxThreadsPerBlock�ֳɵ�ÿ���߳̿������һ���п�����п�Ľڵ�,һ����֮����Թ�������
		{
			hipDeviceSynchronize();
			ThreadPerBlock = prop.maxThreadsPerBlock;
			Num_Block = ((Num_Frames_OneTime * CW_Len) % ThreadPerBlock == 0) ? (Num_Frames_OneTime * CW_Len) / ThreadPerBlock : ((Num_Frames_OneTime * CW_Len) / ThreadPerBlock) + 1;
			Variablenode_Shared_Kernel<<<Num_Block, ThreadPerBlock>>>(Memory_RQ, D_GPU, Channel_Out, Address_Variablenode, Weight_Variablenode_GPU);//�����ڵ���㣬�õ�L

			ThreadPerBlock = prop.maxThreadsPerBlock;
			Num_Block = ((Num_Frames_OneTime * parLen) % ThreadPerBlock == 0) ? (Num_Frames_OneTime * parLen) / ThreadPerBlock : ((Num_Frames_OneTime * parLen) / ThreadPerBlock) + 1;

			if (decoder_method == 0)
			{
				Checknode_Shared_Kernel<<<Num_Block, ThreadPerBlock>>>(Memory_RQ, Weight_Checknode_GPU);
			}

			hipDeviceSynchronize();
		}
		else
		{
			hipDeviceSynchronize();
			ThreadPerBlock = prop.maxThreadsPerBlock;
			Num_Block = ((Num_Frames_OneTime * CW_Len) % ThreadPerBlock == 0) ? (Num_Frames_OneTime * CW_Len) / ThreadPerBlock : ((Num_Frames_OneTime * CW_Len) / ThreadPerBlock) + 1;
			Variablenode_Kernel<<<Num_Block, ThreadPerBlock>>>(Memory_RQ, D_GPU, Channel_Out, Address_Variablenode, Weight_Variablenode_GPU);//�����ڵ���㣬�õ�L

			hipDeviceSynchronize();

			ThreadPerBlock = prop.maxThreadsPerBlock;
			Num_Block = ((Num_Frames_OneTime * parLen) % ThreadPerBlock == 0) ? (Num_Frames_OneTime * parLen) / ThreadPerBlock : ((Num_Frames_OneTime * parLen) / ThreadPerBlock) + 1;
			if (decoder_method == 0)
			{
				Checknode_Kernel<<<Num_Block, ThreadPerBlock>>>(Memory_RQ, Weight_Checknode_GPU);
			}
			
			hipDeviceSynchronize();
		}
		hipDeviceSynchronize();

		memset(D + CW_Len * Num_Frames_OneTime, 0, Num_Frames_OneTime * sizeof(int));
		hipMemcpy(D, D_GPU, CW_Len * Num_Frames_OneTime * sizeof(int), hipMemcpyDeviceToHost);

		for (index0 = 0; index0 < Length; index0++)
		{
			for (index1 = 0; index1 < Num_Frames_OneTime; index1++)
			{
				D[index1 + CW_Len * Num_Frames_OneTime] += D[index0 * Num_Frames_OneTime + index1];//��ÿһ֡�������ֽڼ��������
			}
		}
		index0 = 0;
		for (index1 = 0; index1 < Num_Frames_OneTime; index1++)
		{
			D[index1 + CW_Len * Num_Frames_OneTime] = (D[index1 + CW_Len * Num_Frames_OneTime] == 0) ? 1 : 0;//ȫ�����м�����Ϊȫ0����ȷ���Ϊ1
			index0 += D[index1 + CW_Len * Num_Frames_OneTime];//ͳ�ƶԵ�֡��
		}
		if (index0 == Num_Frames_OneTime)
		{
			break;//����֡��������
		}
		hipDeviceSynchronize();

	}
	hipEventDestroy(GPU_start);
	hipEventDestroy(GPU_stop);

	hipFree(Memory_RQ);
	hipFree(Weight_Checknode_GPU);
	hipFree(Weight_Variablenode_GPU);
	hipFree(D_GPU);
}

/*
* Memory_RQ:���ڴ洢�����ڵ��У��ڵ����ʱ�õ���R��Qֵ
* D:�������
* Weight_Variablenode:�����ڵ�����
* Address_Variablenode:ÿ�������ڵ�����ӦУ��ڵ��memory_rq�ĵ�ַ
*/
__global__ void Variablenode_Kernel(float* Memory_RQ, int* D, float* Channel_Out, int* Address_Variablenode, int* Weight_Variablenode)
{
	int offset, num_Variablenode, num_Frames, num_VariablenodeZ;
	float R[15];
	float Add_result;
	int Ad[15];
	int Weight;

	offset = threadIdx.x + blockIdx.x * blockDim.x;			// �̺߳�
	num_Variablenode = offset / Num_Frames_OneTime;		// �����ڵ���ţ�16��֡�ĵ�һ�������ڵ�-16��֡�ĵڶ����ڵ�-��������16��֡�����һ���ڵ㣩
	num_Frames = offset % Num_Frames_OneTime;		// ֡��
	num_VariablenodeZ = num_Variablenode / Z;					// �ֿ�ʽУ������ж�Ӧ���п��,����offset / (Z*Num_Frames_OneTime_define)��1��zά�����z���ڵ㣩
	num_Variablenode = num_Variablenode * Weight_Variablenode[L];//ת������Address_Variablenode_GPU���λ�ã�ÿ�������ڵ��Ӧ�����ӹ�ϵ��Address_Variablenode_GPUÿһ����һ�������ڵ�����е����ӣ�

	

	if (offset < CW_Len * Num_Frames_OneTime)//memory�������ǣ�֡1�����ڵ�1���ӵĽڵ㡪��֡2�ڵ��1���ӵĽڵ㡪��֡3����������������������
	{
		Weight = Weight_Variablenode[num_VariablenodeZ];
		for (int i = 0; i < Weight; i++)
		{
			Ad[i] = Address_Variablenode[num_Variablenode + i] * Num_Frames_OneTime + num_Frames;
		}
		for (int i = 0; i < Weight; i++)
		{
			R[i] = Memory_RQ[(Ad[i])];
		}
		for (int i = 0; i < Weight; i++)
		{
			Add_result += R[i];
		}
		Add_result += Channel_Out[offset];
		D[offset] = (Add_result < 0) ? 1 : 0;//����R����Q�������ڵ�;
		for (int i = 0; i < Weight;i++)
		{
			Memory_RQ[Ad[i]] = Add_result - R[i];

		}
	}
}
/*
* Memory_RQ:���ڴ洢�����ڵ��У��ڵ����ʱ�õ���R��Qֵ
* D:�������
* Weight_Variablenode:�����ڵ�����
* Address_Variablenode:ÿ�������ڵ�����ӦУ��ڵ��memory_rq�ĵ�ַ
*/
__global__ void Variablenode_Shared_Kernel(float* Memory_RQ, int* D, float* Channel_Out, int* Address_Variablenode, int* Weight_Variablenode)
{
	int offset, num_Variablenode, num_Frames, num_VariablenodeZ;
	float R[15];
	float Add_result;
	int Ad[15];
	__shared__ int Weight;

	offset = threadIdx.x + blockIdx.x * blockDim.x;			// �̺߳�
	num_Variablenode = offset / Num_Frames_OneTime;		// �����ڵ���ţ�16��֡�ĵ�һ�������ڵ�-16��֡�ĵڶ����ڵ�-��������16��֡�����һ���ڵ㣩
	num_Frames = offset % Num_Frames_OneTime;		// ֡��
	num_VariablenodeZ = num_Variablenode / Z;					// �ֿ�ʽУ������ж�Ӧ���п��,����offset / (Z*Num_Frames_OneTime_define)��1��zά�����z���ڵ㣩
	num_Variablenode = num_Variablenode * Weight_Variablenode[L];//ת������Address_Variablenode_GPU���λ�ã�ÿ�������ڵ��Ӧ�����ӹ�ϵ��Address_Variablenode_GPUÿһ����һ�������ڵ�����е����ӣ�

	if (threadIdx.x == 0 && num_VariablenodeZ < L)
	{
		Weight = Weight_Variablenode[num_VariablenodeZ];//ֻ��Ҫ��һ��ֵ
	}
	__syncthreads();

	if (offset < CW_Len * Num_Frames_OneTime)//memory�������ǣ�֡1�����ڵ�1���ӵĽڵ㡪��֡2�ڵ��1���ӵĽڵ㡪��֡3����������������������
	{
		// �����ַ��ʱ����Ҫ��������
		for (int i = 0; i < Weight; i++)
		{
			Ad[i] = Address_Variablenode[num_Variablenode + i] * Num_Frames_OneTime + num_Frames;
		}
		for (int i = 0; i < Weight; i++)
		{
			R[i] = Memory_RQ[(Ad[i])];
		}
		for (int i = 0; i < Weight; i++)
		{
			Add_result += R[i];
		}
		Add_result += Channel_Out[offset];
		D[offset] = (Add_result < 0) ? 1 : 0;//����R����Q�������ڵ�;
		for (int i = 0; i < Weight; i++)
		{
			Memory_RQ[Ad[i]] = Add_result - R[i];

		}
	}
}
__global__ void Checknode_Kernel(float* Memory_RQ, int* Weight_Checknode)
{
	int offset, num_Checknode, num_Frames, num_ChecknodeZ;
	__shared__ int Weight;
	float Q[25], Q0[25];
	int Sign[26];
	float MinQ, SubMinQ;
	int Index_minQ;

	offset = threadIdx.x + blockIdx.x * blockDim.x;
	num_Checknode = offset / Num_Frames_OneTime;													// У��ڵ����
	num_Frames = offset % Num_Frames_OneTime;													// ֡��
	num_Frames = num_Frames + num_Checknode * Num_Frames_OneTime * Weight_Checknode[J];	// ��ǰ֡�ĸ�У��ڵ��0��Qֵ�Ĵ�ŵ�ַ
	num_ChecknodeZ = num_Checknode / Z;																	// ��ǰУ��ڵ����ڵ��п��

	

	if (offset < Num_Frames_OneTime * parLen)//q����memory_rq��һ��
	{
		Weight = Weight_Checknode[num_ChecknodeZ];
		for (int i = 0; i < Weight; i++)
		{
			Q[i] = Memory_RQ[num_Frames + i * Num_Frames_OneTime];
		}
		for (int i = 0; i < Weight; i++)
		{
			Sign[i] = (Q[i] < 0) ? -1 : 1;
			Q[i] = (Q[i] < 0) ? -Q[i] : Q[i];
			Q0[i] = Q[i];
		}
		Sign[25] = 1;
		for (int i = 0; i < Weight; i++)
		{
			Sign[25] *= Sign[i];
		}
		sortQ(&MinQ, &SubMinQ, Q, Weight);
		for (int i = 0; i < Weight; i++)
		{
			if (Q0[i] == MinQ)
			{
				Index_minQ = i;
				break;
			}
		}
		for (int i = 0; i < Weight; i++)
		{
			if (i != Index_minQ)
			{
				Memory_RQ[num_Frames + i * Num_Frames_OneTime] = Sign[25] * Sign[i] * MinQ;
			}
			else Memory_RQ[num_Frames + i * Num_Frames_OneTime] = Sign[25] * Sign[i] * SubMinQ;
		}
	}
}
__global__ void Checknode_Shared_Kernel(float* Memory_RQ, int* Weight_Checknode)
{
	int offset, num_Checknode, num_Frames, num_ChecknodeZ;
	__shared__ int Weight;
	float Q[25],Q0[25];
	int Sign[26];
	float MinQ, SubMinQ;
	int Index_minQ;

	offset = threadIdx.x + blockIdx.x * blockDim.x;
	num_Checknode = offset / Num_Frames_OneTime;													// У��ڵ����
	num_Frames = offset % Num_Frames_OneTime;													// ֡��
	num_Frames = num_Frames + num_Checknode * Num_Frames_OneTime * Weight_Checknode[J];	// ��ǰ֡�ĸ�У��ڵ��0��Qֵ�Ĵ�ŵ�ַ
	num_ChecknodeZ = num_Checknode / Z;																	// ��ǰУ��ڵ����ڵ��п��

	if (threadIdx.x == 0 && num_ChecknodeZ < J)		// �õ�0���߳��ҵ����߳̿��������̶߳�Ӧ������
	{
		Weight = Weight_Checknode[num_ChecknodeZ];
	}
	__syncthreads();

	if (offset < Num_Frames_OneTime * parLen)//q����memory_rq��һ��
	{
		for (int i = 0; i < Weight; i++)
		{
			Q[i] = Memory_RQ[num_Frames + i * Num_Frames_OneTime];
		}
		for (int i = 0; i < Weight; i++)
		{
			Sign[i]= (Q[i] < 0) ? -1 : 1;
			Q[i] = (Q[i] < 0) ? -Q[i] : Q[i];
			Q0[i] = Q[i];
		}
		Sign[25] = 1;
		for (int i = 0; i < Weight; i++)
		{
			Sign[25] *= Sign[i];
		}
		sortQ(&MinQ, &SubMinQ, Q, Weight);
		for (int i = 0; i < Weight;i++)
		{
			if (Q0[i] == MinQ)
			{
				Index_minQ = i;
				break;
			}
		}
		for (int i = 0; i < Weight; i++)
		{
			if (i != Index_minQ)
			{
				Memory_RQ[num_Frames + i * Num_Frames_OneTime] = Sign[25] * Sign[i] * MinQ;
			}
			else Memory_RQ[num_Frames + i * Num_Frames_OneTime] = Sign[25] * Sign[i] * SubMinQ;
		}
	}
}

__device__ void sortQ(float* MinQ, float* SubMinQ, float* Q,int Weight)
{
	float tmp;
	for (int i = 0; i < 2; i++) {

		for (int j = 0; j < Weight-1; j++) 
		{

			if (Q[j] < Q[j + 1]) 
			{

				tmp = Q[j];

				Q[j] = Q[j + 1];

				Q[j + 1] = tmp;

			}

		}

	}
	*MinQ = Q[Weight - 1];
	*SubMinQ = Q[Weight - 2];
}