#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "define.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.cuh"

int main()
{
	AWGNChannel* AWGN;
	Simulation* SIM;


	int* Weight_Checknode;			// LDPC����ֿ���У��ڵ������,����ΪJ_define+1,���һ��Ϊ�������.������CPU��.
	int* Weight_Variablenode;		// LDPC����ֿ��б����ڵ������,����ΪL_define+1,���һ��Ϊ�������.������CPU��.
	int* H;					// LDPC��ֿ�ʽУ�����,���к���,����ΪJ_define*L_define*Z_define.������CPU��.
	int* Address_Variablenode;		// �����ڵ������ӵ�У��ڵ�����(ע��������Num_Frames_OneTime_define��֡������һ��ͬʱ����),����ΪL_define*J_define*Z_define.������CPU��.
	int* Address_Variablenode_GPU;
	float* sigma_GPU;
	hipError_t cudaStatus;

	

	AWGN = (AWGNChannel*)malloc(sizeof(AWGNChannel));
	if (AWGN == NULL)
	{
		printf("Can not malloc AWGN in main on Host!\n");
		//getch();
		exit(0);
	}

	SIM = (Simulation*)malloc(sizeof(Simulation));
	if (SIM == NULL)
	{
		printf("Can not malloc SIM in main on Host!\n");
		//getch();
		exit(0);
	}

	Weight_Checknode = (int*)malloc((J + 1) * sizeof(int));// LDPC����ֿ���У��ڵ������,����ΪJ_define+1,���һ��Ϊ�������.������CPU��.
	if (Weight_Checknode == NULL)
	{
		printf("Can not malloc Weight_Checknode in main on Host!\n");
		//getch();
		exit(0);
	}

	Weight_Variablenode = (int*)malloc((L + 1) * sizeof(int));// LDPC����ֿ��б����ڵ������,����ΪL_define+1,���һ��Ϊ�������.������CPU��.
	if (Weight_Variablenode == NULL)
	{
		printf("Can not malloc Weight_Variablenode in main on Host!\n");
		//getch();
		exit(0);
	}

	H = (int*)malloc(J * L * sizeof(int));// LDPC��ֿ�ʽУ�����,���к���,����ΪJ_define*L_define.������CPU��.
	if (H == NULL)
	{
		printf("Can not malloc Block_H in main on Host!\n");
		//getch();
		exit(0);
	}

	Address_Variablenode = (int*)malloc(J * L * Z * sizeof(int));// �����ڵ������ӵ�У��ڵ�����(ע��������Num_Frames_OneTime_define��֡������һ��ͬʱ����),����ΪL_define*J_define*Z_define.������CPU��.
	if (Address_Variablenode == NULL)
	{
		printf("Can not malloc Address_Variablenode in main on Host!\n");
		//getch();
		exit(0);
	}

	cudaStatus = hipMalloc((void**)&Address_Variablenode_GPU, J * L * Z * sizeof(int));	// ������GPU��global memory��
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc Address_Variablenode_GPU in main on device, exit!\n");
		//getch();
		exit(0);
	}

	cudaStatus = hipMalloc((void**)&sigma_GPU, sizeof(float));	// ������GPU��global memory��
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc sigma_GPU in main on device, exit!\n");
		//getch();
		exit(0);
	}

	memset(Weight_Checknode, 0, (J + 1) * sizeof(int));
	memset(Weight_Variablenode, 0, (L + 1) * sizeof(int));
	memset(H, 0, J * L* sizeof(int));
	memset(Address_Variablenode, -1, J* L* Z * sizeof(int));
	

	// ��define.cuh�ж�ȡ��Ҫ�Ĳ���,�����ⲿ��QC-LDPC��ֿ�ʽУ������ж�ȡ��Ӧ��ƫ�����ͽڵ�����ֵ,������Ӧ��CPU�ڴ���
	Get_H(H, Weight_Checknode, Weight_Variablenode);
	Transform_H(H, Weight_Checknode, Weight_Variablenode, Address_Variablenode);
	cudaStatus = hipMemcpy(Address_Variablenode_GPU, Address_Variablenode, J * L * Z * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Address_Variablenode to Address_Variablenode_GPU in main, exit!\n");
		//getch();
		exit(0);
	}

	*(AWGN->seed + 0) = ix_define;			// ÿһ��SNR��Ҫ��һ�γ�ֵ
	*(AWGN->seed + 1) = iy_define;			// ÿһ��SNR��Ҫ��һ�γ�ֵ
	*(AWGN->seed + 2) = iz_define;			// ÿһ��SNR��Ҫ��һ�γ�ֵ
	AWGN->sigma = 0.0;						// ÿһ��SNR��Ҫ��һ��ֵ

	// ��������Ҫ������ʾ����Ļ��,��д����Ӧ���ĵ���
	WriteLogo(AWGN, SIM);

	for (SIM->SNR = startSNR; SIM->SNR <= stopSNR; SIM->SNR += stepSNR)
	{
		/*��ʼ����֮ǰ,�Բ������г�ʼ��*/
		*(AWGN->seed + 0) = ix_define;			// ��֤ÿһ��SNR�µķ��滷����ȫ��ͬ
		*(AWGN->seed + 1) = iy_define;
		*(AWGN->seed + 2) = iz_define;
		if (snrtype == 0)
		{
			AWGN->sigma = (float)sqrt(0.5 / (rate * (pow(10.0, (SIM->SNR / 10.0)))));//(float)LDPC->msgLen / LDPC->codewordLen;
		}
		else if (snrtype == 1)
		{
			AWGN->sigma = (float)sqrt(0.5 / (pow(10.0, (SIM->SNR / 10.0))));
		}

		SIM->num_Frames = 0;					// ���¿�ʼͳ��
		SIM->num_Error_Frames = 0;
		SIM->num_Error_Bits = 0;
		SIM->Total_Iteration = 0;
		SIM->num_False_Frames = 0;
		SIM->num_Alarm_Frames = 0;

		//hipDeviceSynchronize();

		/*��ʼ������ȵ�ķ���*/
		if (CPU_GPU == 0)
		{
			//SNR_Simulation_CPU(LDPC, AWGN, SIM, Address_Variablenode, Weight_Checknode, Weight_Variablenode);
		}
		else if (CPU_GPU == 1)
		{
			cudaStatus = hipMemcpy(sigma_GPU, &(AWGN->sigma), sizeof(float), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess)
			{
				printf("Cannot copy sigma to sigma_GPU in main, exit!\n");
				//getch();
				exit(0);
			}

			Simulation_GPU(AWGN, sigma_GPU, SIM, Address_Variablenode_GPU, Weight_Checknode, Weight_Variablenode);
		}
		/*��CPU��GPU����ͬ��,��ִֹֻ��һ������ȵ㼴����ѭ��*/
		hipDeviceSynchronize();
	}


	free(AWGN);
	free(SIM);
	free(Weight_Checknode);
	free(Weight_Variablenode);
	free(H);
	free(Address_Variablenode);
	hipFree(sigma_GPU);

	hipDeviceReset();

	printf("\ntask finish\n");
	printf("\nPress any key to stop\n");
	getchar();
	return 0;
}