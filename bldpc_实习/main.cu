#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "define.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.cuh"

int main()
{
	AWGNChannel* AWGN;
	Simulation* SIM;


	int* Weight_Checknode;			// LDPC码各分块中校验节点的重量,长度为J_define+1,最后一个为最大重量.分配在CPU上.
	int* Weight_Variablenode;		// LDPC码各分块中变量节点的重量,长度为L_define+1,最后一个为最大重量.分配在CPU上.
	int* H;					// LDPC码分块式校验矩阵,先行后列,长度为J_define*L_define*Z_define.分配在CPU上.
	int* Address_Variablenode;		// 变量节点相连接的校验节点的序号(注意现在是Num_Frames_OneTime_define个帧穿插在一起同时译码),长度为L_define*J_define*Z_define.分配在CPU上.
	int* Address_Variablenode_GPU;
	float* sigma_GPU;
	hipError_t cudaStatus;

	

	AWGN = (AWGNChannel*)malloc(sizeof(AWGNChannel));
	if (AWGN == NULL)
	{
		printf("Can not malloc AWGN in main on Host!\n");
		//getch();
		exit(0);
	}

	SIM = (Simulation*)malloc(sizeof(Simulation));
	if (SIM == NULL)
	{
		printf("Can not malloc SIM in main on Host!\n");
		//getch();
		exit(0);
	}

	Weight_Checknode = (int*)malloc((J + 1) * sizeof(int));// LDPC码各分块中校验节点的重量,长度为J_define+1,最后一个为最大重量.分配在CPU上.
	if (Weight_Checknode == NULL)
	{
		printf("Can not malloc Weight_Checknode in main on Host!\n");
		//getch();
		exit(0);
	}

	Weight_Variablenode = (int*)malloc((L + 1) * sizeof(int));// LDPC码各分块中变量节点的重量,长度为L_define+1,最后一个为最大重量.分配在CPU上.
	if (Weight_Variablenode == NULL)
	{
		printf("Can not malloc Weight_Variablenode in main on Host!\n");
		//getch();
		exit(0);
	}

	H = (int*)malloc(J * L * sizeof(int));// LDPC码分块式校验矩阵,先行后列,长度为J_define*L_define.分配在CPU上.
	if (H == NULL)
	{
		printf("Can not malloc Block_H in main on Host!\n");
		//getch();
		exit(0);
	}

	Address_Variablenode = (int*)malloc(J * L * Z * sizeof(int));// 变量节点相连接的校验节点的序号(注意现在是Num_Frames_OneTime_define个帧穿插在一起同时译码),长度为L_define*J_define*Z_define.分配在CPU上.
	if (Address_Variablenode == NULL)
	{
		printf("Can not malloc Address_Variablenode in main on Host!\n");
		//getch();
		exit(0);
	}

	cudaStatus = hipMalloc((void**)&Address_Variablenode_GPU, J * L * Z * sizeof(int));	// 分配在GPU的global memory中
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc Address_Variablenode_GPU in main on device, exit!\n");
		//getch();
		exit(0);
	}

	cudaStatus = hipMalloc((void**)&sigma_GPU, sizeof(float));	// 分配在GPU的global memory中
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc sigma_GPU in main on device, exit!\n");
		//getch();
		exit(0);
	}

	memset(Weight_Checknode, 0, (J + 1) * sizeof(int));
	memset(Weight_Variablenode, 0, (L + 1) * sizeof(int));
	memset(H, 0, J * L* sizeof(int));
	memset(Address_Variablenode, -1, J* L* Z * sizeof(int));
	

	// 从define.cuh中读取所要的参数,并从外部的QC-LDPC码分块式校验矩阵中读取相应的偏移量和节点重量值,存于相应的CPU内存中
	Get_H(H, Weight_Checknode, Weight_Variablenode);
	Transform_H(H, Weight_Checknode, Weight_Variablenode, Address_Variablenode);
	cudaStatus = hipMemcpy(Address_Variablenode_GPU, Address_Variablenode, J * L * Z * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Address_Variablenode to Address_Variablenode_GPU in main, exit!\n");
		//getch();
		exit(0);
	}

	*(AWGN->seed + 0) = ix_define;			// 每一个SNR下要赋一次初值
	*(AWGN->seed + 1) = iy_define;			// 每一个SNR下要赋一次初值
	*(AWGN->seed + 2) = iz_define;			// 每一个SNR下要赋一次初值
	AWGN->sigma = 0.0;						// 每一个SNR下要赋一次值

	// 将部分重要参数显示在屏幕上,并写入相应的文档中
	WriteLogo(AWGN, SIM);

	for (SIM->SNR = startSNR; SIM->SNR <= stopSNR; SIM->SNR += stepSNR)
	{
		/*开始仿真之前,对参数进行初始化*/
		*(AWGN->seed + 0) = ix_define;			// 保证每一个SNR下的仿真环境完全相同
		*(AWGN->seed + 1) = iy_define;
		*(AWGN->seed + 2) = iz_define;
		if (snrtype == 0)
		{
			AWGN->sigma = (float)sqrt(0.5 / (rate * (pow(10.0, (SIM->SNR / 10.0)))));//(float)LDPC->msgLen / LDPC->codewordLen;
		}
		else if (snrtype == 1)
		{
			AWGN->sigma = (float)sqrt(0.5 / (pow(10.0, (SIM->SNR / 10.0))));
		}

		SIM->num_Frames = 0;					// 重新开始统计
		SIM->num_Error_Frames = 0;
		SIM->num_Error_Bits = 0;
		SIM->Total_Iteration = 0;
		SIM->num_False_Frames = 0;
		SIM->num_Alarm_Frames = 0;

		//hipDeviceSynchronize();

		/*开始本信噪比点的仿真*/
		if (CPU_GPU == 0)
		{
			//SNR_Simulation_CPU(LDPC, AWGN, SIM, Address_Variablenode, Weight_Checknode, Weight_Variablenode);
		}
		else if (CPU_GPU == 1)
		{
			cudaStatus = hipMemcpy(sigma_GPU, &(AWGN->sigma), sizeof(float), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess)
			{
				printf("Cannot copy sigma to sigma_GPU in main, exit!\n");
				//getch();
				exit(0);
			}

			Simulation_GPU(AWGN, sigma_GPU, SIM, Address_Variablenode_GPU, Weight_Checknode, Weight_Variablenode);
		}
		/*对CPU和GPU进行同步,防止只执行一个信噪比点即跳出循环*/
		hipDeviceSynchronize();
	}


	free(AWGN);
	free(SIM);
	free(Weight_Checknode);
	free(Weight_Variablenode);
	free(H);
	free(Address_Variablenode);
	hipFree(sigma_GPU);

	hipDeviceReset();

	printf("\ntask finish\n");
	printf("\nPress any key to stop\n");
	getchar();
	return 0;
}