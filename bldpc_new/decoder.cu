#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "decoder.cuh"

/**
 * @description:���������õ���С�ĺʹ�С��
 * @param {*}MinQ:��С�� SubMinQ:��С��
 * @return {*}
 */
__device__ void sortQ(float* MinQ, float* SubMinQ, float* Q,int Weight)
{
	float tmp;
	for (int i = 0; i < 2; i++) {
		for (int j = 0; j < Weight-1; j++) 
		{
			if (Q[j] < Q[j + 1]) 
			{
				tmp = Q[j];
				Q[j] = Q[j + 1];
				Q[j + 1] = tmp;
			}
		}
	}
	*MinQ = Q[Weight - 1];
	*SubMinQ = Q[Weight - 2];
}

/**
 * @description:�����ڵ���� 
 * @param {*}
 * @return {*}
 */
__global__ void Variablenode_Kernel(float* Memory_RQ, int* D, float* Channel_Out, int* Address_Variablenode, int* Weight_Variablenode)
{
	int offset, num_Variablenode, num_Frames, num_VariablenodeZ;
	float R[15];
	float Add_result;
	int Ad[15];
	int Weight;

	offset = threadIdx.x + blockIdx.x * blockDim.x;			// �̺߳�
	num_Variablenode = offset / Num_Frames_OneTime;		// �����ڵ���ţ�16��֡�ĵ�һ�������ڵ�-16��֡�ĵڶ����ڵ�-��������16��֡�����һ���ڵ㣩
	num_Frames = offset % Num_Frames_OneTime;		// ֡��
	num_VariablenodeZ = num_Variablenode / Z;					// �ֿ�ʽУ������ж�Ӧ���п��,����offset / (Z*Num_Frames_OneTime_define)��1��zά�����z���ڵ㣩
	num_Variablenode = num_Variablenode * Weight_Variablenode[L];//ת������Address_Variablenode_GPU���λ�ã�ÿ�������ڵ��Ӧ�����ӹ�ϵ��Address_Variablenode_GPUÿһ����һ�������ڵ�����е����ӣ�

	

	if (offset < CW_Len * Num_Frames_OneTime)//memory�������ǣ�֡1�����ڵ�1���ӵĽڵ㡪��֡2�ڵ��1���ӵĽڵ㡪��֡3����������������������
	{
		Weight = Weight_Variablenode[num_VariablenodeZ];
		for (int i = 0; i < Weight; i++)
		{
			Ad[i] = Address_Variablenode[num_Variablenode + i] * Num_Frames_OneTime + num_Frames;
		}
		for (int i = 0; i < Weight; i++)
		{
			R[i] = Memory_RQ[(Ad[i])];
		}
		for (int i = 0; i < Weight; i++)
		{
			Add_result += R[i];
		}
		Add_result += Channel_Out[offset];
		D[offset] = (Add_result < 0) ? 1 : 0;//����R����Q�������ڵ�;
		for (int i = 0; i < Weight;i++)
		{
			Memory_RQ[Ad[i]] = Add_result - R[i];

		}
	}
}