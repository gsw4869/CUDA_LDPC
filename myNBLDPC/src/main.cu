#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "define.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.cuh"
#include "struct.cuh"

int main()
{
	
	CN* Checknode;			// LDPC码各分块中校验节点的重量
	VN* Variablenode;		// LDPC码各分块中变量节点的重量
	Get_H(Variablenode,Checknode);
	return 0;
}