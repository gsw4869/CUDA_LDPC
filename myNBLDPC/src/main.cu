#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "define.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.cuh"
#include "LDPC_Encoder.cuh"

int main()
{
	AWGNChannel* AWGN;
	AWGN=(AWGNChannel* )malloc(sizeof(AWGN));
	Simulation* SIM;
	SIM=(Simulation* )malloc(sizeof(Simulation));

	

	CN* Checknode;			// LDPC码各分块中校验节点的重量
	VN* Variablenode;		// LDPC码各分块中变量节点的重量
	
	LDPCCode* H;
	H=(LDPCCode* )malloc(sizeof(LDPCCode));
	
	Get_H(H,Variablenode,Checknode);


	int* CodeWord;
	CodeWord=(int* )malloc(H->Variablenode_num*sizeof(int));
	memset(CodeWord,0,H->Variablenode_num*sizeof(int));

	// float* BPSK_Out;
	// BPSK_Out=(float* )malloc(H->Variablenode_num*sizeof(float));

	float* Channel_Out;
	Channel_Out=(float* )malloc(H->Variablenode_num*sizeof(float));

	for (SIM->SNR = startSNR; SIM->SNR <= stopSNR; SIM->SNR += stepSNR)
	{
		AWGN->seed[0]=ix_define;
		AWGN->seed[1]=iy_define;
		AWGN->seed[2]=iz_define;
		AWGN->sigma=0;

		if (snrtype == 0)
		{
			AWGN->sigma = (float)sqrt(0.5 / (H->rate * (pow(10.0, (SIM->SNR / 10.0)))));//(float)LDPC->msgLen / LDPC->codewordLen;
		}
		else if (snrtype == 1)
		{
			AWGN->sigma = (float)sqrt(0.5 / (pow(10.0, (SIM->SNR / 10.0))));
		}
		printf("%f\n\n",AWGN->sigma);
		SIM->num_Frames = 0;					// 重新开始统计
		SIM->num_Error_Frames = 0;
		SIM->num_Error_Bits = 0;
		SIM->Total_Iteration = 0;
		SIM->num_False_Frames = 0;
		SIM->num_Alarm_Frames = 0;

		// BPSK(H,BPSK_Out,CodeWord);
		AWGNChannel_CPU(H,AWGN,Channel_Out,CodeWord);

		// for(int i=0;i<H->Variablenode_num;i++)
		// {
		// 	printf("%f ",Channel_Out[i]);
		// }
		// printf("\n");
		// exit(0);
	}

	return 0;
}