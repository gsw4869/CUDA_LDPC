#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "define.h"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.h"
#include "LDPC_Encoder.h"
#include "GF.h"
#include "math.h"
#include "Decode_GPU.cuh"

int main()
{
	hipError_t cudaStatus;
	hipDeviceProp_t prop;
	int Num_Device;

	cudaStatus = hipGetDeviceCount(&Num_Device);
	if (cudaStatus != hipSuccess)
	{
		printf("There is no GPU beyond 1.0, exit!\n");
		exit(0);
	}
	else
	{
		cudaStatus = hipGetDeviceProperties(&prop, Num_Device - 1);
		if (cudaStatus != hipSuccess)
		{
			printf("Cannot get device properties, exit!\n");
			exit(0);
		}
	}
	printf("Device Name : %s.\n", prop.name);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
	printf("maxThreadsPerMultiProcessor : %d.\n",
		   prop.maxThreadsPerMultiProcessor);

	AWGNChannel *AWGN;
	AWGN = (AWGNChannel *)malloc(sizeof(AWGN));
	Simulation *SIM;
	SIM = (Simulation *)malloc(sizeof(Simulation));

	CN *Checknode;	  // LDPC码各分块中校验节点的重量
	VN *Variablenode; // LDPC码各分块中变量节点的重量

	LDPCCode *H;
	H = (LDPCCode *)malloc(sizeof(LDPCCode));

	//	先读取行数和列数,分配空间
	FILE *fp_H;

	if (NULL == (fp_H = fopen(Matrixfile, "r")))
	{
		printf("can not open file: %s\n", Matrixfile);
		exit(0);
	}

	int threadNum =
		THREAD_NUM ? THREAD_NUM : std::thread::hardware_concurrency();

	fscanf(fp_H, "%d", &H->Variablenode_num); // 变量节点个数（行数）
	Variablenode = (VN *)malloc(H->Variablenode_num * threadNum * sizeof(VN));

	fscanf(fp_H, "%d", &H->Checknode_num); // 校验节点个数（列数）
	Checknode = (CN *)malloc(H->Checknode_num * threadNum * sizeof(CN));

	fclose(fp_H);
	//
	Get_H(H, Variablenode, Checknode); //初始化剩下的参数

	GFInitial(GFQ);

	CComplex *CONSTELLATION;
	CONSTELLATION = Get_CONSTELLATION(H);

	CComplex *CComplex_sym;

	int *CodeWord_bit;
	CodeWord_bit = (int *)malloc(H->bit_length * sizeof(int));
	memset(CodeWord_bit, 0, H->bit_length * sizeof(int));

	int *CodeWord_sym;
	CodeWord_sym = (int *)malloc(H->Variablenode_num * sizeof(int));
	memset(CodeWord_sym, 0, H->Variablenode_num * sizeof(int));

	int CodeWord_sym_test[96] = {12, 26, 32, 18, 58, 59, 49, 24, 55, 48, 19, 14, 13, 2, 59, 15, 7, 43, 20, 8, 36, 54, 23, 7, 29, 2, 31, 43, 34, 30, 51, 57, 3, 14, 41, 38, 30, 58, 32, 26, 51, 48, 26, 23, 20, 63, 34, 51, 45, 62, 62, 13, 42, 33, 9, 61, 3, 25, 12, 51, 4, 48, 32, 48, 36, 42, 37, 14, 37, 21, 48, 39, 25, 51, 12, 23, 60, 51, 50, 15, 45, 35, 30, 23, 11, 45, 1, 25, 62, 47, 17, 25, 37, 32, 58, 56};

	unsigned *TableMultiply_GPU;
	hipMalloc((void **)&TableMultiply_GPU, GFQ * GFQ * sizeof(unsigned));
	hipMemcpy(TableMultiply_GPU, TableMultiply[0], GFQ * GFQ * sizeof(unsigned), hipMemcpyHostToDevice); //GPU乘法表

	unsigned *TableAdd_GPU;
	hipMalloc((void **)&TableAdd_GPU, GFQ * GFQ * sizeof(unsigned));
	hipMemcpy(TableAdd_GPU, TableAdd[0], GFQ * GFQ * sizeof(unsigned), hipMemcpyHostToDevice); //GPU加法表

	unsigned *TableInverse_GPU;
	hipMalloc((void **)&TableInverse_GPU, GFQ * sizeof(unsigned));
	hipMemcpy(TableInverse_GPU, TableInverse, GFQ * sizeof(unsigned), hipMemcpyHostToDevice); //GPU除法表

	int *Checknode_weight;
	hipMalloc((void **)&Checknode_weight, H->Checknode_num * sizeof(int));

	int *Checknode_weight_temp = (int *)malloc(H->Checknode_num * sizeof(int));
	for (int i = 0; i < H->Checknode_num; i++)
	{
		Checknode_weight_temp[i] = Checknode[i].weight;
	}
	cudaStatus = hipMemcpy(Checknode_weight, Checknode_weight_temp, H->Checknode_num * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Checknode_weight\n");
		exit(0);
	}
	free(Checknode_weight_temp);

	int *Variablenode_weight;
	hipMalloc((void **)&Variablenode_weight, H->Variablenode_num * sizeof(int));

	int *Variablenode_weight_temp = (int *)malloc(H->Variablenode_num * sizeof(int));
	for (int i = 0; i < H->Variablenode_num; i++)
	{
		Variablenode_weight_temp[i] = Variablenode[i].weight;
	}
	cudaStatus = hipMemcpy(Variablenode_weight, Variablenode_weight_temp, H->Variablenode_num * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Variablenode_weight\n");
		exit(0);
	}
	free(Variablenode_weight_temp);

	int *Variablenode_linkCNs;
	hipMalloc((void **)&Variablenode_linkCNs, H->Variablenode_num * maxdv * sizeof(int));

	int *Variablenode_linkCNs_temp = (int *)malloc(H->Variablenode_num * maxdv * sizeof(int));
	for (int i = 0; i < H->Variablenode_num; i++)
	{
		for (int j = 0; j < Variablenode[i].weight; j++)
		{
			Variablenode_linkCNs_temp[i * maxdv + j] = Variablenode[i].linkCNs[j] * GFQ * maxdc + index_in_CN(Variablenode, i, j, Checknode) * GFQ; //直接给到数组里的序号
		}
	}
	cudaStatus = hipMemcpy(Variablenode_linkCNs, Variablenode_linkCNs_temp, H->Variablenode_num * maxdv * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Variablenode_linkCNs\n");
		exit(0);
	}
	free(Variablenode_linkCNs_temp);

	int *Checknode_linkVNs;
	hipMalloc((void **)&Checknode_linkVNs, H->Checknode_num * maxdc * sizeof(int));

	int *Checknode_linkVNs_temp = (int *)malloc(H->Checknode_num * maxdc * sizeof(int));
	for (int i = 0; i < H->Checknode_num; i++)
	{
		for (int j = 0; j < Checknode[i].weight; j++)
		{
			Checknode_linkVNs_temp[i * maxdc + j] = Checknode[i].linkVNs[j] * GFQ * maxdv + index_in_VN(Checknode, i, j, Variablenode) * GFQ; //直接给到数组里的序号
		}
	}
	cudaStatus = hipMemcpy(Checknode_linkVNs, Checknode_linkVNs_temp, H->Checknode_num * maxdc * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Checknode_linkVNs\n");
		exit(0);
	}
	free(Checknode_linkVNs_temp);

	int *Checknode_linkVNs_GF;
	hipMalloc((void **)&Checknode_linkVNs_GF, H->Checknode_num * maxdc * sizeof(int));

	int *Checknode_linkVNs_GF_temp = (int *)malloc(H->Checknode_num * maxdc * sizeof(int));
	for (int i = 0; i < H->Checknode_num; i++)
	{
		for (int j = 0; j < Checknode[i].weight; j++)
		{
			Checknode_linkVNs_GF_temp[i * maxdc + j] = Checknode[i].linkVNs_GF[j];
		}
	}
	cudaStatus = hipMemcpy(Checknode_linkVNs_GF, Checknode_linkVNs_GF_temp, H->Checknode_num * maxdc * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Checknode_linkVNs_GF\n");
		exit(0);
	}
	free(Checknode_linkVNs_GF_temp);

	if (n_QAM != 2)
	{
		CComplex_sym = (CComplex *)malloc(H->Variablenode_num * sizeof(CComplex));
		BitToSym(H, CodeWord_sym, CodeWord_bit);
		for (int i = 0; i < H->Variablenode_num; i++)
		{
			CodeWord_sym[i] = CodeWord_sym_test[i];
		}
		Modulate(H, CONSTELLATION, CComplex_sym, CodeWord_sym);
	}
	else
	{
		CComplex_sym = (CComplex *)malloc(H->bit_length * sizeof(CComplex));
		for (int i = 0; i < H->Variablenode_num; i++)
		{
			for (int j = 0; j < H->q_bit; j++)
			{
				CodeWord_bit[i * H->q_bit + j] = (CodeWord_sym_test[i] & (1 << j)) >> j;
			}
		}
		BitToSym(H, CodeWord_sym, CodeWord_bit);
		Modulate(H, CONSTELLATION, CComplex_sym, CodeWord_bit);
	}

	printf("sim start\n");
	for (SIM->SNR = startSNR; SIM->SNR <= stopSNR; SIM->SNR += stepSNR)
	{
		AWGN->seed[0] = ix_define;
		AWGN->seed[1] = iy_define;
		AWGN->seed[2] = iz_define;
		AWGN->sigma = 0;
		if (snrtype == 0)
		{
			AWGN->sigma = (float)sqrt(0.5 / (log(n_QAM) / log(2) * H->rate * (pow(10.0, (SIM->SNR / 10.0))))); //(float)LDPC->msgLen / LDPC->codewordLen;
		}
		else if (snrtype == 1)
		{
			AWGN->sigma = (float)sqrt(0.5 / (log(n_QAM) / log(2) * pow(10.0, (SIM->SNR / 10.0))));
		}
		SIM->num_Frames = 0; // 重新开始统计
		SIM->num_Error_Frames = 0;
		SIM->num_Error_Bits = 0;
		SIM->Total_Iteration = 0;
		SIM->num_False_Frames = 0;
		SIM->num_Alarm_Frames = 0;
		SIM->sumTime = 0;
		SIM->FER = 0;
		SIM->BER = 0;
		SIM->AverageIT = 0;
		SIM->FER_False = 0;
		SIM->FER_Alarm = 0;
		if (!CPU_GPU)
		{
			Simulation_CPU((const LDPCCode *)H, AWGN, SIM, (const CComplex *)CONSTELLATION, Variablenode, Checknode, (const CComplex *)CComplex_sym, (const int *)CodeWord_sym);
		}
		else
		{
			Simulation_GPU((const LDPCCode *)H, AWGN, SIM, (const CComplex *)CONSTELLATION, Variablenode, Checknode, (const CComplex *)CComplex_sym, CodeWord_sym, (const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, (const int *)Variablenode_weight, (const int *)Checknode_weight, (const int *)Variablenode_linkCNs, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF);
		}
	}
	hipFree(TableMultiply_GPU);
	hipFree(TableAdd_GPU);
	hipFree(TableInverse_GPU);
	hipFree(Checknode_weight);
	hipFree(Variablenode_linkCNs);
	hipFree(Checknode_linkVNs);
	hipFree(Checknode_linkVNs_GF);
	free(AWGN);
	free(SIM);
	freeCN(H, Checknode);
	freeVN(H, Variablenode);
	free(H);
	free(CodeWord_sym);
	free(CodeWord_bit);
	free(CComplex_sym);
	free(CONSTELLATION);

	return 0;
}