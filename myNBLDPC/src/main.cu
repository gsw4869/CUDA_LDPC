#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "define.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.cuh"
#include "LDPC_Encoder.cuh"

int main()
{
	AWGNChannel* AWGN;
	AWGN=(AWGNChannel* )malloc(sizeof(AWGN));
	Simulation* SIM;
	SIM=(Simulation* )malloc(sizeof(Simulation));

	CN* Checknode;			// LDPC码各分块中校验节点的重量
	VN* Variablenode;		// LDPC码各分块中变量节点的重量
	
	LDPCCode* H;
	H=(LDPCCode* )malloc(sizeof(LDPCCode));
	
	Get_H(H,Variablenode,Checknode);


	int* CodeWord;
	CodeWord=(int* )malloc(H->length*sizeof(int));
	memset(CodeWord,0,H->length*sizeof(int));


	float* Channel_Out;
	Channel_Out=(float* )malloc(H->length*sizeof(float));

	for (SIM->SNR = startSNR; SIM->SNR <= stopSNR; SIM->SNR += stepSNR)
	{
		AWGN->seed[0]=ix_define;
		AWGN->seed[1]=iy_define;
		AWGN->seed[2]=iz_define;
		AWGN->sigma=0;

		if (snrtype == 0)
		{
			AWGN->sigma = (float)sqrt(0.5 / (H->rate * (pow(10.0, (SIM->SNR / 10.0)))));//(float)LDPC->msgLen / LDPC->codewordLen;
		}
		else if (snrtype == 1)
		{
			AWGN->sigma = (float)sqrt(0.5 / (pow(10.0, (SIM->SNR / 10.0))));
		}
		SIM->num_Frames = 0;					// 重新开始统计
		SIM->num_Error_Frames = 0;
		SIM->num_Error_Bits = 0;
		SIM->Total_Iteration = 0;
		SIM->num_False_Frames = 0;
		SIM->num_Alarm_Frames = 0;

		// BPSK(H,BPSK_Out,CodeWord);
		AWGNChannel_CPU(H,AWGN,Channel_Out,CodeWord);

		Simulation_GPU(SIM, Variablenode, Checknode, Channel_Out);

		Statistic(SIM,CodeWord,CodeWord,H);

		// for(int i=0;i<H->Variablenode_num;i++)
		// {
		// 	printf("%f ",Channel_Out[i]);
		// }
		// printf("\n");
		// exit(0);
	}

	return 0;
}