#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "define.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.cuh"
#include "LDPC_Encoder.cuh"
#include "GF.cuh"
#include "math.h"
#include "Decode_GPU.cuh"

int main()
{
	hipError_t cudaStatus;
	hipDeviceProp_t prop;
	int Num_Device;

	cudaStatus = hipGetDeviceCount(&Num_Device);
	if (cudaStatus != hipSuccess)
	{
		printf("There is no GPU beyond 1.0, exit!\n");
		exit(0);
	}
	else
	{
		cudaStatus = hipGetDeviceProperties(&prop, Num_Device - 1);
		if (cudaStatus != hipSuccess)
		{
			printf("Cannot get device properties, exit!\n");
			exit(0);
		}
	}
	printf("Device Name : %s.\n", prop.name);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
	printf("maxThreadsPerMultiProcessor : %d.\n",
		   prop.maxThreadsPerMultiProcessor);

	AWGNChannel *AWGN;
	AWGN = (AWGNChannel *)malloc(sizeof(AWGN));
	Simulation *SIM;
	SIM = (Simulation *)malloc(sizeof(Simulation));

	CN *Checknode;	  // LDPC码各分块中校验节点的重量
	VN *Variablenode; // LDPC码各分块中变量节点的重量

	LDPCCode *H;
	H = (LDPCCode *)malloc(sizeof(LDPCCode));

	//	先读取行数和列数,分配空间
	FILE *fp_H;

	if (NULL == (fp_H = fopen(Matrixfile, "r")))
	{
		printf("can not open file: %s\n", Matrixfile);
		exit(0);
	}

	fscanf(fp_H, "%d", &H->Variablenode_num); // 变量节点个数（行数）
	Variablenode = (VN *)malloc(H->Variablenode_num * sizeof(VN));

	fscanf(fp_H, "%d", &H->Checknode_num); // 校验节点个数（列数）
	Checknode = (CN *)malloc(H->Checknode_num * sizeof(CN));

	fclose(fp_H);
	//
	Get_H(H, Variablenode, Checknode); //初始化剩下的参数

	GFInitial(H->GF);

	CComplex *CONSTELLATION;
	CONSTELLATION = Get_CONSTELLATION(H);

	CComplex *CComplex_sym;

	int *CodeWord_bit;
	CodeWord_bit = (int *)malloc(H->bit_length * sizeof(int));
	memset(CodeWord_bit, 0, H->bit_length * sizeof(int));

	int *CodeWord_sym;
	CodeWord_sym = (int *)malloc(H->Variablenode_num * sizeof(int));
	memset(CodeWord_sym, 0, H->Variablenode_num * sizeof(int));

	int *DecodeOutput;
	DecodeOutput = (int *)malloc(H->Variablenode_num * sizeof(int));
	memset(DecodeOutput, 0, H->Variablenode_num * sizeof(int));

	int CodeWord_sym_test[96] = {12, 26, 32, 18, 58, 59, 49, 24, 55, 48, 19, 14, 13, 2, 59, 15, 7, 43, 20, 8, 36, 54, 23, 7, 29, 2, 31, 43, 34, 30, 51, 57, 3, 14, 41, 38, 30, 58, 32, 26, 51, 48, 26, 23, 20, 63, 34, 51, 45, 62, 62, 13, 42, 33, 9, 61, 3, 25, 12, 51, 4, 48, 32, 48, 36, 42, 37, 14, 37, 21, 48, 39, 25, 51, 12, 23, 60, 51, 50, 15, 45, 35, 30, 23, 11, 45, 1, 25, 62, 47, 17, 25, 37, 32, 58, 56};

	unsigned *TableMultiply_GPU;
	hipMalloc((void **)&TableMultiply_GPU, H->GF * H->GF * sizeof(unsigned));
	hipMemcpy(TableMultiply_GPU, TableMultiply[0], H->GF * H->GF * sizeof(unsigned), hipMemcpyHostToDevice); //GPU乘法表

	unsigned *TableAdd_GPU;
	hipMalloc((void **)&TableAdd_GPU, H->GF * H->GF * sizeof(unsigned));
	hipMemcpy(TableAdd_GPU, TableAdd[0], H->GF * H->GF * sizeof(unsigned), hipMemcpyHostToDevice); //GPU加法表

	unsigned *TableInverse_GPU;
	hipMalloc((void **)&TableInverse_GPU, H->GF * sizeof(unsigned));
	hipMemcpy(TableInverse_GPU, TableInverse, H->GF * sizeof(unsigned), hipMemcpyHostToDevice); //GPU除法表

	//GPU Array
	// int *Checknode_weight;
	// int *Variablenode_linkCNs;
	// int *Checknode_linkVNs;
	// int *Checknode_linkVNs_GF;
	// GPUArray_initial(H, Variablenode, Checknode, Checknode_weight, Variablenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF);

	int *Checknode_weight;
	hipMalloc((void **)&Checknode_weight, H->Checknode_num * sizeof(int));

	int *Checknode_weight_temp = (int *)malloc(H->Checknode_num * sizeof(int));
	for (int i = 0; i < H->Checknode_num; i++)
	{
		Checknode_weight_temp[i] = Checknode[i].weight;
	}
	cudaStatus = hipMemcpy(Checknode_weight, Checknode_weight_temp, H->Checknode_num * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Checknode_weight\n");
		exit(0);
	}
	free(Checknode_weight_temp);

	int *Variablenode_linkCNs;
	hipMalloc((void **)&Variablenode_linkCNs, H->Variablenode_num * maxdv * sizeof(int));

	int *Variablenode_linkCNs_temp = (int *)malloc(H->Variablenode_num * maxdv * sizeof(int));
	for (int i = 0; i < H->Variablenode_num; i++)
	{
		for (int j = 0; j < Variablenode[i].weight; j++)
		{
			Variablenode_linkCNs_temp[i * maxdv + j] = Variablenode[i].linkCNs[j];
		}
	}
	cudaStatus = hipMemcpy(Variablenode_linkCNs, Variablenode_linkCNs_temp, H->Variablenode_num * maxdv * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Variablenode_linkCNs\n");
		exit(0);
	}
	free(Variablenode_linkCNs_temp);

	int *Checknode_linkVNs;
	hipMalloc((void **)&Checknode_linkVNs, H->Checknode_num * maxdc * sizeof(int));

	int *Checknode_linkVNs_temp = (int *)malloc(H->Checknode_num * maxdc * sizeof(int));
	for (int i = 0; i < H->Checknode_num; i++)
	{
		for (int j = 0; j < Checknode[i].weight; j++)
		{
			Checknode_linkVNs_temp[i * maxdc + j] = Checknode[i].linkVNs[j];
		}
	}
	cudaStatus = hipMemcpy(Checknode_linkVNs, Checknode_linkVNs_temp, H->Checknode_num * maxdc * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Checknode_linkVNs\n");
		exit(0);
	}
	free(Checknode_linkVNs_temp);

	int *Checknode_linkVNs_GF;
	hipMalloc((void **)&Checknode_linkVNs_GF, H->Checknode_num * maxdc * sizeof(int));

	int *Checknode_linkVNs_GF_temp = (int *)malloc(H->Checknode_num * maxdc * sizeof(int));
	for (int i = 0; i < H->Checknode_num; i++)
	{
		for (int j = 0; j < Checknode[i].weight; j++)
		{
			Checknode_linkVNs_GF_temp[i * maxdc + j] = Checknode[i].linkVNs_GF[j];
		}
	}
	cudaStatus = hipMemcpy(Checknode_linkVNs_GF, Checknode_linkVNs_GF_temp, H->Checknode_num * maxdc * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Checknode_linkVNs_GF\n");
		exit(0);
	}
	free(Checknode_linkVNs_GF_temp);

	if (n_QAM != 2)
	{
		CComplex_sym = (CComplex *)malloc(H->Variablenode_num * sizeof(CComplex));
		BitToSym(H, CodeWord_sym, CodeWord_bit);
		for (int i = 0; i < H->Variablenode_num; i++)
		{
			CodeWord_sym[i] = CodeWord_sym_test[i];
		}
		Modulate(H, CONSTELLATION, CComplex_sym, CodeWord_sym);
	}
	else
	{
		CComplex_sym = (CComplex *)malloc(H->bit_length * sizeof(CComplex));
		for (int i = 0; i < H->Variablenode_num; i++)
		{
			for (int j = 0; j < H->q_bit; j++)
			{
				CodeWord_bit[i * H->q_bit + j] = (CodeWord_sym_test[i] & (1 << j)) >> j;
			}
		}
		BitToSym(H, CodeWord_sym, CodeWord_bit);
		Modulate(H, CONSTELLATION, CComplex_sym, CodeWord_bit);
	}

	for (SIM->SNR = startSNR; SIM->SNR <= stopSNR; SIM->SNR += stepSNR)
	{
		AWGN->seed[0] = ix_define;
		AWGN->seed[1] = iy_define;
		AWGN->seed[2] = iz_define;
		AWGN->sigma = 0;

		if (snrtype == 0)
		{
			AWGN->sigma = (float)sqrt(0.5 / (log(n_QAM) / log(2) * H->rate * (pow(10.0, (SIM->SNR / 10.0))))); //(float)LDPC->msgLen / LDPC->codewordLen;
		}
		else if (snrtype == 1)
		{
			AWGN->sigma = (float)sqrt(0.5 / (log(n_QAM) / log(2) * pow(10.0, (SIM->SNR / 10.0))));
		}
		SIM->num_Frames = 0; // 重新开始统计
		SIM->num_Error_Frames = 0;
		SIM->num_Error_Bits = 0;
		SIM->Total_Iteration = 0;
		SIM->num_False_Frames = 0;
		SIM->num_Alarm_Frames = 0;

		// BPSK(H,BPSK_Out,CodeWord);

		// Simulation_CPU(H, AWGN, SIM, CONSTELLATION, Variablenode, Checknode, CComplex_sym, CodeWord_sym, DecodeOutput);
		Simulation_GPU(H, AWGN, SIM, CONSTELLATION, Variablenode, Checknode, CComplex_sym, CodeWord_sym, DecodeOutput, TableMultiply_GPU, TableAdd_GPU, Checknode_weight, Variablenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF);

		// for(int i=0;i<H->Variablenode_num;i++)
		// {
		// 	printf("%f + %f i\n",CComplex_sym_Channelout[i].Real,CComplex_sym_Channelout[i].Image);
		// }
		// printf("\n");
		// exit(0);
	}
	hipFree(TableMultiply_GPU);
	hipFree(TableAdd_GPU);
	hipFree(TableInverse_GPU);
	hipFree(Checknode_weight);
	hipFree(Variablenode_linkCNs);
	hipFree(Checknode_linkVNs);
	hipFree(Checknode_linkVNs_GF);
	free(AWGN);
	free(SIM);
	free(H);
	free(Checknode);
	free(Variablenode);
	free(CodeWord_sym);
	free(CodeWord_bit);
	free(CComplex_sym);
	free(CONSTELLATION);

	return 0;
}