#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "define.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.cuh"
#include "LDPC_Encoder.cuh"
#include "GF.cuh"

int main()
{
	AWGNChannel* AWGN;
	AWGN=(AWGNChannel* )malloc(sizeof(AWGN));
	Simulation* SIM;
	SIM=(Simulation* )malloc(sizeof(Simulation));

	CN* Checknode;			// LDPC码各分块中校验节点的重量
	VN* Variablenode;		// LDPC码各分块中变量节点的重量
	
	LDPCCode* H;
	H=(LDPCCode* )malloc(sizeof(LDPCCode));

//	先读取行数和列数,分配空间
	FILE* fp_H;
	
	if (NULL == (fp_H = fopen(Matrixfile, "r")))
	{
		printf("can not open file: %s\n", Matrixfile);
		exit(0);
	}

	fscanf(fp_H, "%d", &H->Variablenode_num);// 变量节点个数（行数）
	Variablenode=(VN *)malloc(H->Variablenode_num*sizeof(VN));

	fscanf(fp_H, "%d", &H->Checknode_num);// 校验节点个数（列数）
	Checknode=(CN *)malloc(H->Checknode_num*sizeof(CN));

	fclose(fp_H);
//
	Get_H(H,Variablenode,Checknode);//初始化剩下的参数
	
	GFInitial(H->GF);

	CComplex* CONSTELLATION;
	CONSTELLATION=Get_CONSTELLATION(H);

	CComplex* CComplex_sym;
	CComplex_sym=(CComplex* )malloc(H->Variablenode_num*sizeof(CComplex));

	int* CodeWord_bit;
	CodeWord_bit=(int* )malloc(H->bit_length*sizeof(int));
	memset(CodeWord_bit,0,H->bit_length*sizeof(int));

	for(int i=0;i<H->bit_length;i++)
	{
		CodeWord_bit[i]=1;
	}

	int* CodeWord_sym;
	CodeWord_sym=(int* )malloc(H->Variablenode_num*sizeof(int));
	memset(CodeWord_sym,0,H->Variablenode_num*sizeof(int));


	int* DecodeOutput;
	DecodeOutput=(int* )malloc(H->Variablenode_num*sizeof(int));
	memset(DecodeOutput,0,H->Variablenode_num*sizeof(int));

	BitToSym(H,CodeWord_sym,CodeWord_bit);
	Modulate(H,CONSTELLATION,CComplex_sym,CodeWord_sym);

	for (SIM->SNR = startSNR; SIM->SNR <= stopSNR; SIM->SNR += stepSNR)
	{
		AWGN->seed[0]=ix_define;
		AWGN->seed[1]=iy_define;
		AWGN->seed[2]=iz_define;
		AWGN->sigma=0;

		if (snrtype == 0)
		{
			AWGN->sigma = (float)sqrt(0.5 / (H->rate * (pow(10.0, (SIM->SNR / 10.0)))));//(float)LDPC->msgLen / LDPC->codewordLen;
		}
		else if (snrtype == 1)
		{
			AWGN->sigma = (float)sqrt(0.5 / (pow(10.0, (SIM->SNR / 10.0))));
		}
		SIM->num_Frames = 0;					// 重新开始统计
		SIM->num_Error_Frames = 0;
		SIM->num_Error_Bits = 0;
		SIM->Total_Iteration = 0;
		SIM->num_False_Frames = 0;
		SIM->num_Alarm_Frames = 0;

		// BPSK(H,BPSK_Out,CodeWord);

		Simulation_GPU(H,AWGN,SIM,CONSTELLATION,Variablenode, Checknode, CComplex_sym,DecodeOutput);

		Statistic(SIM,CodeWord_sym,DecodeOutput,H);

		// for(int i=0;i<H->Variablenode_num;i++)
		// {
		// 	printf("%f + %f i\n",CComplex_sym_Channelout[i].Real,CComplex_sym_Channelout[i].Image);
		// }
		// printf("\n");
		// exit(0);
	}

	free(AWGN);
	free(SIM);
	free(H);
	free(Checknode);
	free(Variablenode);
	free(CodeWord_sym);
	free(CodeWord_bit);
	free(CComplex_sym);
	free(CONSTELLATION);

	return 0;
}