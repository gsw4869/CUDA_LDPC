#include "hip/hip_runtime.h"
#include "Decode_GPU.cuh"
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <assert.h>

__device__ void BubleSort_GPU(float a[], int n, int index[])
{
    int i, j;
    float x;
    for (i = 0; i < n; i++)
    {
        for (j = 1; j < n - i; j++)
        {
            if (a[j - 1] < a[j])
            {
                x = a[j];
                a[j] = a[j - 1];
                a[j - 1] = x;
                x = index[j];
                index[j] = index[j - 1];
                index[j - 1] = x;
            }
        }
    }
}

__device__ int SortLLRVector_GPU(int GF, float *Entr_v2c, int *index)
{
    BubleSort_GPU(Entr_v2c, GF, index);
    return 1;
}

__device__ int DecideLLRVector_GPU(float *LLR, int GF)
{
    float max = 0;
    int alpha_i;
    for (int q = 0; q < GF - 1; q++)
    {
        if (LLR[q] > max)
        {
            max = LLR[q];
            alpha_i = q + 1;
        }
    }
    if (max <= 0)
    {
        return 0;
    }
    else
    {
        return alpha_i;
    }
}
__device__ void d_BubleSort_GPU(float a[], int n, int index[])
{
    int i, j;
    float x;
    for (i = 0; i < n; i++)
    {
        for (j = 1; j < n - i; j++)
        {
            if (a[j - 1] > a[j])
            {
                x = a[j];
                a[j] = a[j - 1];
                a[j - 1] = x;
                x = index[j];
                index[j] = index[j - 1];
                index[j - 1] = x;
            }
        }
    }
}

__device__ int d_SortLLRVector_GPU(int GF, float *Entr_v2c, int *index)
{
    d_BubleSort_GPU(Entr_v2c, GF, index);
    return 1;
}

__device__ int d_DecideLLRVector_GPU(float *LLR, int GF)
{
    float min = DBL_MAX;
    int alpha_i;
    for (int q = 0; q < GFQ; q++)
    {
        if (LLR[q] < min)
        {
            min = LLR[q];
            alpha_i = q;
        }
    }
    return alpha_i;
}

__device__ int GetCombCount(int n, int m)
{
    long int i;
    long int a, b, c, s; // s = a/(b*c)
    a = b = c = 1;
    for (i = 1; i <= n; i++)
        a *= i;
    for (i = 1; i <= m; i++)
        b *= i;
    for (i = 1; i <= n - m; i++)
        c *= i;
    s = a / (b * c);
    return s;
}
__device__ void swap(int &a, int &b)
{
    int temp = a;
    a = b;
    b = temp;
}
__device__ int GFAdd_GPU(int ele1, int ele2, const unsigned *TableAdd_GPU)
{
    return ele1 ^ ele2;
}

__device__ int GFMultiply_GPU(int ele1, int ele2, const unsigned *TableMultiply_GPU)
{
    return TableMultiply_GPU[GFQ * ele1 + ele2];
}

__device__ int GFInverse_GPU(int ele, const unsigned *TableInverse_GPU)
{
    if (ele == 0)
    {
        printf("Div 0 Error!\n");
    }
    return TableInverse_GPU[ele];
}

int Decoding_EMS_GPU(const LDPCCode *H, VN *Variablenode, CN *Checknode, int EMS_Nm, int EMS_Nc, int *DecodeOutput, const unsigned *TableMultiply_GPU, const unsigned *TableAdd_GPU, const int *Variablenode_weight, const int *Checknode_weight, const int *Variablenode_linkCNs, const int *Checknode_linkVNs, const int *Checknode_linkVNs_GF, int &iter_number)
{
    hipError_t cudaStatus;
    // int *sort_Entr_v2c_temp = (int *)malloc(H->Variablenode_num * maxdv * GFQ * sizeof(int));
    // memset(sort_Entr_v2c_temp, 0, H->Variablenode_num * maxdv * GFQ * sizeof(int));
    int *sort_Entr_v2c;
    hipMalloc((void **)&sort_Entr_v2c, H->Variablenode_num * maxdv * GFQ * sizeof(int));

    float *sort_L_v2c_temp = (float *)malloc(H->Variablenode_num * maxdv * GFQ * sizeof(float));
    memset(sort_L_v2c_temp, 0, H->Variablenode_num * maxdv * GFQ * sizeof(float));
    float *sort_L_v2c;
    hipMalloc((void **)&sort_L_v2c, H->Variablenode_num * maxdv * GFQ * sizeof(float));

    float *Checknode_L_c2v_temp = (float *)malloc(H->Checknode_num * maxdc * GFQ * sizeof(float));
    memset(Checknode_L_c2v_temp, 0, H->Checknode_num * maxdc * GFQ * sizeof(float));

    float *Checknode_L_c2v;
    hipMalloc((void **)&Checknode_L_c2v, H->Checknode_num * maxdc * GFQ * sizeof(float));

    // int *index = (int *)malloc((GFQ) * sizeof(int));

    float *L_ch_temp = (float *)malloc(H->Variablenode_num * (GFQ - 1) * sizeof(float));
    memset(L_ch_temp, 0, H->Variablenode_num * (GFQ - 1) * sizeof(float));

    float *L_ch;
    hipMalloc((void **)&L_ch, H->Variablenode_num * (GFQ - 1) * sizeof(float));

    float *LLR_temp = (float *)malloc(H->Variablenode_num * (GFQ - 1) * sizeof(float));
    float *LLR;
    hipMalloc((void **)&LLR, H->Variablenode_num * (GFQ - 1) * sizeof(float));

    for (int col = 0; col < H->Variablenode_num; col++)
    {
        for (int d = 0; d < Variablenode[col].weight; d++)
        {
            Variablenode[col].L_ch[GFQ - 1] = 0;
            for (int q = 0; q < GFQ; q++)
            {
                sort_L_v2c_temp[col * maxdv * GFQ + d * GFQ + q] = Variablenode[col].L_ch[q];
                Variablenode[col].sort_L_v2c[d][q] = Variablenode[col].L_ch[q];
            }
        }
        for (int q = 0; q < GFQ; q++)
        {
            L_ch_temp[col * (GFQ - 1) + q] = Variablenode[col].L_ch[q];
        }
    }
    cudaStatus = hipMemcpy(L_ch, L_ch_temp, H->Variablenode_num * (GFQ - 1) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy L_ch\n");
        exit(0);
    }

    cudaStatus = hipMemcpy(sort_L_v2c, sort_L_v2c_temp, H->Variablenode_num * maxdv * GFQ * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy sort_L_v2c\n");
        exit(0);
    }

    cudaStatus = hipMemcpy(Checknode_L_c2v, Checknode_L_c2v_temp, H->Checknode_num * maxdc * GFQ * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy Checknode_L_c2v\n");
        exit(0);
    }

    for (int row = 0; row < H->Checknode_num; row++)
    {
        for (int d = 0; d < Checknode[row].weight; d++)
        {
            memset(Checknode[row].L_c2v[d], 0, (GFQ - 1) * sizeof(float));
        }
    }

    iter_number = 0;
    bool decode_correct = true;
    int *DecodeOutput_GPU;
    hipMalloc((void **)&DecodeOutput_GPU, H->Variablenode_num * sizeof(int));

    while (iter_number < maxIT)
    {
        iter_number++;
        Variablenode_EMS<<<H->Variablenode_num,1>>>((const int *)Variablenode_weight, (const int *)Variablenode_linkCNs, sort_Entr_v2c, sort_L_v2c, Checknode_L_c2v, (const float *)L_ch, LLR, DecodeOutput_GPU, H->Variablenode_num);

        cudaStatus = hipMemcpy(DecodeOutput, DecodeOutput_GPU, H->Variablenode_num * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            printf("Cannot copy DecodeOutput\n");
            exit(0);
        }
        // for (int i = 0; i < H->Variablenode_num; i++)
        // {
        //     for (int q = 0; q < GFQ - 1; q++)
        //     {
        //         Variablenode[i].LLR[q] = LLR_temp[i * (GFQ - 1) + q];
        //     }
        //     DecodeOutput[i] = DecideLLRVector(Variablenode[i].LLR, GFQ);
        // }

        decode_correct = true;
        int sum_temp = 0;
        for (int row = 0; row < H->Checknode_num; row++)
        {
            for (int i = 0; i < Checknode[row].weight; i++)
            {
                sum_temp = GFAdd(sum_temp, GFMultiply(DecodeOutput[Checknode[row].linkVNs[i]], Checknode[row].linkVNs_GF[i]));
            }
            if (sum_temp)
            {
                decode_correct = false;
                break;
            }
        }

        if (decode_correct)
        {

            hipFree(sort_Entr_v2c);
            hipFree(sort_L_v2c);
            hipFree(Checknode_L_c2v);
            hipFree(LLR);
            hipFree(L_ch);
            hipFree(DecodeOutput_GPU);
            free(L_ch_temp);
            free(LLR_temp);
            // free(index);
            // free(sort_Entr_v2c_temp);
            free(sort_L_v2c_temp);
            free(Checknode_L_c2v_temp);
            iter_number--;
            return 1;
        }
        // message from var to check
        // for (int col = 0; col < H->Variablenode_num; col++)
        // {
        //     for (int dv = 0; dv < Variablenode[col].weight; dv++)
        //     {
        //         for (int q = 0; q < GFQ - 1; q++)
        //         {
        //             Variablenode[col].sort_L_v2c[dv][q] = Variablenode[col].LLR[q] - Checknode[Variablenode[col].linkCNs[dv]].L_c2v[index_in_CN(Variablenode, col, dv, Checknode)][q];
        //         }
        //         Variablenode[col].sort_L_v2c[dv][GFQ - 1] = 0;
        //     }
        // }

        // for (int col = 0; col < H->Variablenode_num; col++)
        // {
        //     for (int dv = 0; dv < Variablenode[col].weight; dv++)
        //     {
        //         for (int i = 0; i < GFQ - 1; i++)
        //         {
        //             index[i] = i + 1;
        //         }
        //         index[GFQ - 1] = 0;
        //         SortLLRVector(GFQ, Variablenode[col].sort_L_v2c[dv], index);
        //         for (int i = 0; i < GFQ; i++)
        //         {
        //             Variablenode[col].sort_Entr_v2c[dv][i] = index[i];

        //             sort_Entr_v2c_temp[col * maxdv * GFQ + dv * GFQ + i] = index[i];
        //             sort_L_v2c_temp[col * maxdv * GFQ + dv * GFQ + i] = Variablenode[col].sort_L_v2c[dv][i];
        //         }
        //     }
        // }

        Variablenode_Update_EMS<<<H->Variablenode_num,1>>>((const int *)Variablenode_weight, (const int *)Variablenode_linkCNs, sort_Entr_v2c, sort_L_v2c, Checknode_L_c2v, (const float *)L_ch, LLR, H->Variablenode_num);

        // cudaStatus = hipMemcpy(sort_Entr_v2c, sort_Entr_v2c_temp, H->Variablenode_num * maxdv * GFQ * sizeof(int), hipMemcpyHostToDevice);
        // if (cudaStatus != hipSuccess)
        // {
        //     printf("Cannot copy sort_Entr_v2c\n");
        //     exit(0);
        // }
        // cudaStatus = hipMemcpy(sort_L_v2c, sort_L_v2c_temp, H->Variablenode_num * maxdv * GFQ * sizeof(float), hipMemcpyHostToDevice);
        // if (cudaStatus != hipSuccess)
        // {
        //     printf("Cannot copy sort_L_v2c\n");
        //     exit(0);
        // }
        // // message from check to var

        Checknode_EMS<<<H->Checknode_num,1>>>((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, EMS_Nm, EMS_Nc, (const int *)Checknode_weight, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c, Checknode_L_c2v, H->Checknode_num);
        // Checknode_EMS<<<1, 1>>>((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, EMS_Nm, EMS_Nc, (const int *)Checknode_weight, (const int *)Variablenode_linkCNs, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c, Checknode_L_c2v, H->Checknode_num);

        // cudaStatus = hipMemcpy(Checknode_L_c2v_temp, Checknode_L_c2v, H->Checknode_num * maxdc * GFQ * sizeof(float), hipMemcpyDeviceToHost);
        // if (cudaStatus != hipSuccess)
        // {
        //     printf("Cannot copy Checknode_L_c2v D2V\n");
        //     exit(0);
        // }

        // for (int i = 0; i < H->Checknode_num; i++)
        // {
        //     for (int j = 0; j < Checknode[i].weight; j++)
        //     {
        //         for (int q = 0; q < GFQ - 1; q++)
        //         {
        //             Checknode[i].L_c2v[j][q] = Checknode_L_c2v_temp[i * maxdc * GFQ + j * GFQ + q];
        //         }
        //     }
        // }
    }

    hipFree(sort_Entr_v2c);
    hipFree(sort_L_v2c);
    hipFree(Checknode_L_c2v);
    hipFree(LLR);
    hipFree(L_ch);
    free(L_ch_temp);
    free(LLR_temp);
    hipFree(DecodeOutput_GPU);
    // free(index);
    // free(sort_Entr_v2c_temp);
    free(sort_L_v2c_temp);
    free(Checknode_L_c2v_temp);
    return 0;
}

__global__ void Variablenode_EMS(const int *Variablenode_weight, const int *Variablenode_linkCNs, int *sort_Entr_v2c, float *sort_L_v2c, float *Checknode_L_c2v, const float *L_ch, float *LLR, int *DecodeOutput, int Variablenode_num)
{

    int offset;
    offset = threadIdx.x + blockDim.x * blockIdx.x;
    if (offset < Variablenode_num)
    {
        for (int q = 0; q < GFQ - 1; q++)
        {
            LLR[offset * (GFQ - 1) + q] = L_ch[offset * (GFQ - 1) + q];
        }
        for (int d = 0; d < Variablenode_weight[offset]; d++)
        {
            for (int q = 0; q < GFQ - 1; q++)
            {
                LLR[offset * (GFQ - 1) + q] += Checknode_L_c2v[Variablenode_linkCNs[offset * maxdv + d] + q];
            }
        }
        DecodeOutput[offset] = DecideLLRVector_GPU(LLR + offset * (GFQ - 1), GFQ);
    }
}

__global__ void Variablenode_Update_EMS(const int *Variablenode_weight, const int *Variablenode_linkCNs, int *sort_Entr_v2c, float *sort_L_v2c, float *Checknode_L_c2v, const float *L_ch, float *LLR, int Variablenode_num)
{

    int offset;
    offset = threadIdx.x + blockDim.x * blockIdx.x;
    int *index = (int *)malloc(GFQ * sizeof(int));
    if (offset < Variablenode_num)
    {

        for (int dv = 0; dv < Variablenode_weight[offset]; dv++)
        {
            for (int q = 0; q < GFQ - 1; q++)
            {
                sort_L_v2c[offset * maxdv * GFQ + dv * GFQ + q] = LLR[offset * (GFQ - 1) + q] - Checknode_L_c2v[Variablenode_linkCNs[offset * maxdv + dv] + q];
            }
            sort_L_v2c[offset * maxdv * GFQ + dv * GFQ + GFQ - 1] = 0;
        }
        for (int dv = 0; dv < Variablenode_weight[offset]; dv++)
        {
            for (int i = 0; i < GFQ - 1; i++)
            {
                index[i] = i + 1;
            }
            index[GFQ - 1] = 0;
            SortLLRVector_GPU(GFQ, sort_L_v2c + offset * maxdv * GFQ + dv * GFQ, index);
            for (int i = 0; i < GFQ; i++)
            {
                sort_Entr_v2c[offset * maxdv * GFQ + dv * GFQ + i] = index[i];
            }
        }
    }
    free(index);
}

/*
Checknode_weight:每一个校验节点的重量
L_c2v:Q个信息，Q个信息，Q个信息，一共校验节点数量*Q个
Variblenode_linkCNs:最大重量dv，每dv个元素代表连接的dv个校验节点的序号
Checknode_linkVNS:最大重量dc，每dc个元素代表连接的dc个变量节点的序号
Checknode_linkVNS_GF:最大重量dc，每dc个元素代表连接的dc个变量节点的多元域值
sort_Entr_v2c:每个变量节点重量dv，q,q,q一共dv个，然后再乘以变量节点个数[变量节点个数][变量节点重量][q]
sort_L_v2c:和sort_Entr_v2c对应的LLR
Checknode_L_c2v:每个校验节点重量dc，q一共dc个，然后再乘以变量节点个数[校验节点个数][校验节点重量][q]
*/
__global__ void Checknode_EMS(const unsigned *TableMultiply_GPU, const unsigned *TableAdd_GPU, int EMS_Nm, int EMS_Nc, const int *Checknode_weight, const int *Checknode_linkVNs, const int *Checknode_linkVNs_GF, int *sort_Entr_v2c, float *sort_L_v2c, float *Checknode_L_c2v, int Checknode_num)
{
    int offset;
    offset = threadIdx.x + blockDim.x * blockIdx.x;
    if (offset < Checknode_num)
    {
        float EMS_L_c2v[GFQ];
        for (int dc = 0; dc < maxdc; dc++)
        {
            if (dc < Checknode_weight[offset])
            {
                // reset the sum store vector to the munimum
                for (int q = 0; q < GFQ; q++)
                {
                    EMS_L_c2v[q] = -DBL_MAX;
                }

                // recursly exhaustly
                int sumNonele;
                float sumNonLLR;
                // conf(q, 1)
                sumNonele = 0;
                sumNonLLR = 0;
                // ConstructConf_GPU((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, GFQ, 1, sumNonele, sumNonLLR, diff, 0, dc, Checknode_weight[offset] - 1, offset, EMS_L_c2v, (const int *)Variblenode_linkCNs, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);

                for (int i = 0; i < Checknode_weight[offset]; i++)
                {
                    if (i == dc)
                    {
                        continue;
                    }

                    sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                    sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i]];
                }
                if (sumNonLLR > EMS_L_c2v[sumNonele])
                {
                    EMS_L_c2v[sumNonele] = sumNonLLR;
                }
                int sumNonele_all_max = sumNonele;
                float sumNonLLR_all_max = sumNonLLR;
                for (int i = 0; i < Checknode_weight[offset]; i++)
                {
                    if (i == dc)
                    {
                        continue;
                    }

                    sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele_all_max, TableAdd_GPU);
                    sumNonLLR = sumNonLLR_all_max - sort_L_v2c[Checknode_linkVNs[offset * maxdc + i]];

                    for (int k = 1; k < GFQ; k++)
                    {

                        int sumNonele1 = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i] + k], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                        float sumNonLLR1 = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i] + k];

                        if (sumNonLLR1 > EMS_L_c2v[sumNonele1])
                        {
                            EMS_L_c2v[sumNonele1] = sumNonLLR1;
                        }
                    }
                }

                // conf(nm, nc)
                // sumNonele = 0;
                // sumNonLLR = 0;
                // diff = 0;
                // ConstructConf_GPU((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, EMS_Nm, EMS_Nc, sumNonele, sumNonLLR, diff, 0, dc, Checknode_weight[offset] - 1, offset, EMS_L_c2v, (const int *)Variblenode_linkCNs, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);
                int *bit = new int[Checknode_weight[offset] - 1];
                int EMS_Nc_temp;
                if (EMS_Nc == maxdc - 1)
                {
                    EMS_Nc_temp = Checknode_weight[offset] - 1;
                }
                else
                {
                    EMS_Nc_temp = EMS_Nc;
                }
                for (int choose_n = 2; choose_n <= EMS_Nc_temp; choose_n++)
                {

                    for (int k = 0; k < Checknode_weight[offset] - 1; k++)
                    {
                        if (k < choose_n)
                            bit[k] = 1;
                        else
                            bit[k] = 0;
                    }

                    int i, j, beg, end;
                    int len = Checknode_weight[offset] - 1;
                    int N = GetCombCount(Checknode_weight[offset] - 1, choose_n); //C(n,count)  C(5,3)

                    int *conf_index = (int *)malloc(choose_n * sizeof(int));
                    memset(conf_index, 0, (choose_n) * sizeof(int));

                    int flag = 0;

                    while (!flag)
                    {
                        sumNonele = 0;
                        sumNonLLR = 0;
                        for (int i = 0; i < choose_n; i++)
                        {
                            conf_index[i] += 1; // move confset[i] to smaller one

                            if (i == choose_n - 1 && conf_index[i] == EMS_Nm)
                            { // reaches end
                                flag = 1;
                                break;
                            }
                            else if (conf_index[i] >= EMS_Nm)
                            {
                                conf_index[i] = 0;
                                // continue to modify next VN
                            }
                            else
                            {
                                break; // don't modify next VN
                            }
                        }
                        if (!flag)
                        {
                            int k = 0;
                            int t = 0;
                            for (int i = 0; i < Checknode_weight[offset]; i++)
                            {
                                if (i == dc)
                                {
                                    continue;
                                }
                                if (bit[t] == 1)
                                {
                                    sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i] + conf_index[k]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                                    sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i] + conf_index[k]];
                                    k++;
                                }
                                else
                                {
                                    sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                                    sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i]];
                                }
                                t++;
                            }
                            if (sumNonLLR > EMS_L_c2v[sumNonele])
                            {
                                EMS_L_c2v[sumNonele] = sumNonLLR;
                            }
                        }
                    }
                    for (j = 1; j < N; j++)
                    {
                        for (i = len - 1; i > 0; i--)
                        {
                            if (bit[i] == 0 && bit[i - 1] == 1)
                            {
                                swap(bit[i], bit[i - 1]);

                                //from index: [i to len-1] , make all bit 1 in the right
                                beg = i;
                                end = len - 1;
                                while (1)
                                {
                                    while (bit[beg] == 1)
                                    {
                                        beg++;
                                        if (beg >= len)
                                            break;
                                    }
                                    while (bit[end] == 0)
                                    {
                                        end--;
                                        if (end < i)
                                            break;
                                    }

                                    if (beg < end)
                                        swap(bit[beg], bit[end]);
                                    else
                                        break;

                                } //end of "while"
                                break;
                            } //end of "if"
                        }
                        flag = 0;
                        memset(conf_index, 0, (choose_n) * sizeof(int));

                        while (!flag)
                        {
                            sumNonele = 0;
                            sumNonLLR = 0;
                            for (int i = 0; i < choose_n; i++)
                            {
                                conf_index[i] += 1; // move confset[i] to smaller one

                                if (i == choose_n - 1 && conf_index[i] == EMS_Nm)
                                { // reaches end
                                    flag = 1;
                                    break;
                                }
                                else if (conf_index[i] >= EMS_Nm)
                                {
                                    conf_index[i] = 0;
                                    // continue to modify next VN
                                }
                                else
                                {
                                    break; // don't modify next VN
                                }
                            }
                            if (!flag)
                            {
                                int k = 0;
                                int t = 0;
                                for (int i = 0; i < Checknode_weight[offset]; i++)
                                {
                                    if (i == dc)
                                    {
                                        continue;
                                    }
                                    if (bit[t] == 1)
                                    {
                                        sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i] + conf_index[k]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                                        sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i] + conf_index[k]];
                                        k++;
                                    }
                                    else
                                    {
                                        sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                                        sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i]];
                                    }
                                    t++;
                                }
                                if (sumNonLLR > EMS_L_c2v[sumNonele])
                                {
                                    EMS_L_c2v[sumNonele] = sumNonLLR;
                                }
                            }
                        }
                    }
                    free(conf_index);
                }

                free(bit);
                // calculate each c2v LLR
                int v = 0;
                Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + GFQ - 1] = 0;
                for (int k = 1; k < GFQ; k++)
                {
                    v = GFMultiply_GPU(k, Checknode_linkVNs_GF[offset * maxdc + dc], TableMultiply_GPU);
                    Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + k - 1] = (EMS_L_c2v[v] - EMS_L_c2v[0]) / 1.2;
                }
            }
            else
            {
                for (int k = 0; k < GFQ; k++)
                {
                    Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + k] = 0;
                }
            }
        }
    }
}

int Decoding_TMM_GPU(const LDPCCode *H, VN *Variablenode, CN *Checknode, int EMS_Nm, int EMS_Nc, int *DecodeOutput, const unsigned *TableMultiply_GPU, const unsigned *TableAdd_GPU, const unsigned *TableInverse_GPU, const int *Variablenode_weight, const int *Checknode_weight, const int *Variablenode_linkCNs, const int *Checknode_linkVNs, const int *Checknode_linkVNs_GF, int &iter_number)
{
    hipError_t cudaStatus;

    float *sort_L_v2c_temp = (float *)malloc(H->Variablenode_num * maxdv * GFQ * sizeof(float));
    memset(sort_L_v2c_temp, 0, H->Variablenode_num * maxdv * GFQ * sizeof(float));
    float *sort_L_v2c;
    hipMalloc((void **)&sort_L_v2c, H->Variablenode_num * maxdv * GFQ * sizeof(float));

    float *Checknode_L_c2v_temp = (float *)malloc(H->Checknode_num * maxdc * GFQ * sizeof(float));
    memset(Checknode_L_c2v_temp, 0, H->Checknode_num * maxdc * GFQ * sizeof(float));

    float *Checknode_L_c2v;
    hipMalloc((void **)&Checknode_L_c2v, H->Checknode_num * maxdc * GFQ * sizeof(float));

    float *LLR_temp = (float *)malloc(H->Variablenode_num * GFQ * sizeof(float));
    memset(LLR_temp, 0, H->Variablenode_num * GFQ * sizeof(float));
    float *LLR;
    hipMalloc((void **)&LLR, H->Variablenode_num * GFQ * sizeof(float));

    float max = -DBL_MAX;
    for (int col = 0; col < H->Variablenode_num; col++)
    {
        max = -DBL_MAX;
        for (int q = 0; q < GFQ - 1; q++)
        {
            if (Variablenode[col].L_ch[q] > max)
            {
                max = Variablenode[col].L_ch[q];
            }
        }
        for (int d = 0; d < Variablenode[col].weight; d++)
        {
            for (int q = 0; q < GFQ; q++)
            {
                if (q == 0)
                {
                    Variablenode[col].sort_L_v2c[d][q] = max;
                    Variablenode[col].LLR[q] = max;
                    sort_L_v2c_temp[col * maxdv * GFQ + d * GFQ + q] = max;
                    LLR_temp[col * GFQ + q] = max;
                }
                else
                {
                    sort_L_v2c_temp[col * maxdv * GFQ + d * GFQ + q] = max - Variablenode[col].L_ch[q - 1];
                    Variablenode[col].sort_L_v2c[d][q] = max - Variablenode[col].L_ch[q - 1];
                    Variablenode[col].LLR[q] = max - Variablenode[col].L_ch[q - 1];
                    LLR_temp[col * GFQ + q] = max - Variablenode[col].L_ch[q - 1];
                }
            }
        }
    }
    cudaStatus = hipMemcpy(LLR, LLR_temp, H->Variablenode_num * GFQ * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy LLR\n");
        exit(0);
    }

    cudaStatus = hipMemcpy(sort_L_v2c, sort_L_v2c_temp, H->Variablenode_num * maxdv * GFQ * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy sort_L_v2c\n");
        exit(0);
    }

    cudaStatus = hipMemcpy(Checknode_L_c2v, Checknode_L_c2v_temp, H->Checknode_num * maxdc * GFQ * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy Checknode_L_c2v\n");
        exit(0);
    }

    for (int row = 0; row < H->Checknode_num; row++)
    {
        for (int d = 0; d < Checknode[row].weight; d++)
        {
            memset(Checknode[row].L_c2v[d], 0, GFQ * sizeof(float));
        }
    }

    iter_number = 0;
    bool decode_correct = true;
    int *DecodeOutput_GPU;
    hipMalloc((void **)&DecodeOutput_GPU, H->Variablenode_num * sizeof(int));

    while (iter_number < maxIT)
    {
        iter_number++;
        Variablenode_TMM<<<H->Variablenode_num,1>>>((const int *)Variablenode_weight, (const int *)Variablenode_linkCNs, sort_L_v2c, Checknode_L_c2v, LLR, DecodeOutput_GPU, H->Variablenode_num);

        cudaStatus = hipMemcpy(DecodeOutput, DecodeOutput_GPU, H->Variablenode_num * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            printf("Cannot copy DecodeOutput\n");
            exit(0);
        }

        decode_correct = true;
        int sum_temp = 0;
        for (int row = 0; row < H->Checknode_num; row++)
        {
            for (int i = 0; i < Checknode[row].weight; i++)
            {
                sum_temp = GFAdd(sum_temp, GFMultiply(DecodeOutput[Checknode[row].linkVNs[i]], Checknode[row].linkVNs_GF[i]));
            }
            if (sum_temp)
            {
                decode_correct = false;
                break;
            }
        }

        if (decode_correct)
        {
            hipFree(sort_L_v2c);
            hipFree(Checknode_L_c2v);
            hipFree(LLR);
            hipFree(DecodeOutput_GPU);
            free(LLR_temp);
            free(sort_L_v2c_temp);
            free(Checknode_L_c2v_temp);
            iter_number--;
            return 1;
        }

        Variablenode_Update_TMM<<<H->Variablenode_num,1>>>((const int *)Variablenode_weight, (const int *)Variablenode_linkCNs, sort_L_v2c, Checknode_L_c2v, LLR, H->Variablenode_num);

        Checknode_TMM<<<H->Checknode_num,1>>>((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, (const unsigned *)TableInverse_GPU, (const int *)Checknode_weight, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_L_v2c, Checknode_L_c2v, H->Checknode_num);
    }
    hipFree(sort_L_v2c);
    hipFree(Checknode_L_c2v);
    hipFree(LLR);
    free(LLR_temp);
    hipFree(DecodeOutput_GPU);
    free(sort_L_v2c_temp);
    free(Checknode_L_c2v_temp);
    return 0;
}

__global__ void Variablenode_TMM(const int *Variablenode_weight, const int *Variablenode_linkCNs, float *sort_L_v2c, float *Checknode_L_c2v, float *LLR, int *DecodeOutput, int Variablenode_num)
{

    int offset;
    offset = threadIdx.x + blockDim.x * blockIdx.x;
    if (offset < Variablenode_num)
    {
        for (int d = 0; d < Variablenode_weight[offset]; d++)
        {
            for (int q = 0; q < GFQ; q++)
            {
                LLR[offset * GFQ + q] += Checknode_L_c2v[Variablenode_linkCNs[offset * maxdv + d] + q];
            }
        }
        DecodeOutput[offset] = d_DecideLLRVector_GPU(LLR + offset * GFQ, GFQ);
    }
}

__global__ void Variablenode_Update_TMM(const int *Variablenode_weight, const int *Variablenode_linkCNs, float *sort_L_v2c, float *Checknode_L_c2v, float *LLR, int Variablenode_num)
{

    int offset;
    offset = threadIdx.x + blockDim.x * blockIdx.x;
    if (offset < Variablenode_num)
    {
        for (int dv = 0; dv < Variablenode_weight[offset]; dv++)
        {
            for (int q = 0; q < GFQ; q++)
            {
                sort_L_v2c[offset * maxdv * GFQ + dv * GFQ + q] = LLR[offset * GFQ + q] - Checknode_L_c2v[Variablenode_linkCNs[offset * maxdv + dv] + q];
            }
        }
    }
}

__global__ void Checknode_TMM(const unsigned *TableMultiply_GPU, const unsigned *TableAdd_GPU, const unsigned *TableInverse_GPU, const int *Checknode_weight, const int *Checknode_linkVNs, const int *Checknode_linkVNs_GF, float *sort_L_v2c, float *Checknode_L_c2v, int Checknode_num)
{
    int offset;
    offset = threadIdx.x + blockDim.x * blockIdx.x;
    if (offset < Checknode_num)
    {
        int *TMM_Zn = (int *)malloc(maxdc * sizeof(int));
        float *TMM_deltaU = (float *)malloc(maxdc * GFQ * sizeof(float));
        float *TMM_Min1 = (float *)malloc(GFQ * sizeof(float));
        float *TMM_Min2 = (float *)malloc(GFQ * sizeof(float));
        int *TMM_Min1_Col = (int *)malloc(GFQ * sizeof(int));
        float *TMM_I = (float *)malloc(GFQ * sizeof(float));
        int *TMM_Path = (int *)malloc(GFQ * 2 * sizeof(int));
        float *TMM_E = (float *)malloc(GFQ * sizeof(float));
        float *TMM_Lc2p = (float *)malloc(GFQ * sizeof(float));
        int TMM_Syndrome = 0;
        d_TMM_Get_Zn_GPU((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, (const unsigned *)TableInverse_GPU, (const int *)Checknode_weight, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_L_v2c, Checknode_L_c2v, TMM_Zn, offset, TMM_Syndrome);

        d_TMM_Get_deltaU_GPU((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, (const unsigned *)TableInverse_GPU, (const int *)Checknode_weight, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_L_v2c, Checknode_L_c2v, TMM_Zn, TMM_deltaU, offset);

        TMM_Get_Min_GPU(Checknode_weight, TMM_Zn, TMM_deltaU, TMM_Min1, TMM_Min2, TMM_Min1_Col, offset);

        TMM_ConstructConf_GPU((const unsigned *)TableInverse_GPU, TMM_deltaU, TMM_Min1, TMM_Min2, TMM_Min1_Col, TMM_I, TMM_Path, TMM_E);

        for (int dc = 0; dc < Checknode_weight[offset]; dc++)
        {
            // choose to output
            TMM_Lc2p[0] = 0;
            for (int eta = 1; eta < GFQ; eta++)
            {
                if (dc != TMM_Path[eta * 2 + 0] && dc != TMM_Path[eta * 2 + 1])
                {
                    TMM_Lc2p[eta] = TMM_I[eta];
                }
                else
                {
                    TMM_Lc2p[eta] = TMM_E[eta];
                }
            }

            int h_inverse = GFInverse_GPU(Checknode_linkVNs_GF[offset * maxdc + dc], TableInverse_GPU);
            int beta_syn = GFAdd_GPU(TMM_Syndrome, TMM_Zn[dc], TableAdd_GPU);
            double L0 = TMM_Lc2p[beta_syn];
            for (int eta = 0; eta < GFQ; eta++)
            {
                int beta =
                    GFMultiply_GPU(h_inverse, GFAdd_GPU(eta, beta_syn, TableAdd_GPU), TableMultiply_GPU);
                Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + beta] = (TMM_Lc2p[eta]) * 0.8;
            }
        }
        free(TMM_Zn);
        free(TMM_deltaU);
        free(TMM_Min1);
        free(TMM_Min2);
        free(TMM_Min1_Col);
        free(TMM_I);
        free(TMM_Path);
        free(TMM_E);
        free(TMM_Lc2p);
    }
}

__device__ int d_TMM_Get_Zn_GPU(const unsigned *TableMultiply_GPU, const unsigned *TableAdd_GPU, const unsigned *TableInverse_GPU, const int *Checknode_weight, const int *Checknode_linkVNs, const int *Checknode_linkVNs_GF, float *sort_L_v2c, float *Checknode_L_c2v, int *TMM_Zn, int row, int &TMM_Syndrome)
{
    TMM_Syndrome = 0;
    for (int dc = 0; dc < Checknode_weight[row]; dc++)
    {
        double min = DBL_MAX;
        int min_ele = 0;
        for (int q = 0; q < GFQ; q++)
        {
            if (sort_L_v2c[Checknode_linkVNs[row * maxdc + dc] + q] < min)
            {
                min = sort_L_v2c[Checknode_linkVNs[row * maxdc + dc] + q];
                min_ele = GFMultiply_GPU(q, Checknode_linkVNs_GF[row * maxdc + dc], TableMultiply_GPU);
            }
        }
        TMM_Zn[dc] = min_ele;
        TMM_Syndrome = GFAdd_GPU(TMM_Syndrome, min_ele, TableAdd_GPU);
    }
    return 0;
}

__device__ int d_TMM_Get_deltaU_GPU(const unsigned *TableMultiply_GPU, const unsigned *TableAdd_GPU, const unsigned *TableInverse_GPU, const int *Checknode_weight, const int *Checknode_linkVNs, const int *Checknode_linkVNs_GF, float *sort_L_v2c, float *Checknode_L_c2v, int *TMM_Zn, float *TMM_deltaU, int row)
{
    for (int dc = 0; dc < Checknode_weight[row]; dc++)
    {

        int h_inverse = GFInverse_GPU(Checknode_linkVNs_GF[row * maxdc + dc], TableInverse_GPU);

        int beta_p = GFMultiply_GPU(h_inverse, TMM_Zn[dc], TableMultiply_GPU);
        float min = sort_L_v2c[Checknode_linkVNs[row * maxdc + dc] + beta_p];

        for (int x = 0; x < GFQ; x++)
        {
            int eta = GFAdd_GPU(x, TMM_Zn[dc], TableAdd_GPU);
            TMM_deltaU[dc * GFQ + eta] =
                sort_L_v2c[Checknode_linkVNs[row * maxdc + dc] + GFMultiply_GPU(h_inverse, x, TableMultiply_GPU)] - min;
        }
    }
    return 0;
}

__device__ int TMM_Get_Min_GPU(const int *Checknode_weight, int *TMM_Zn, float *TMM_deltaU, float *TMM_Min1, float *TMM_Min2, int *TMM_Min1_Col, int row)
{
    // sort
    for (int q = 0; q < GFQ; q++)
    {
        // clear
        TMM_Min1[q] = DBL_MAX;
        TMM_Min2[q] = DBL_MAX;
        // search min and submin
        for (int dc = 0; dc < Checknode_weight[row]; dc++)
        {
            if (TMM_deltaU[dc * GFQ + q] < TMM_Min1[q])
            {
                TMM_Min2[q] = TMM_Min1[q];
                TMM_Min1[q] = TMM_deltaU[dc * GFQ + q];
                TMM_Min1_Col[q] = dc;
            }
            else if (TMM_deltaU[dc * GFQ + q] < TMM_Min2[q])
            {
                TMM_Min2[q] = TMM_deltaU[dc * GFQ + q];
            }
        }
    }

    return 0;
}

__device__ int TMM_ConstructConf_GPU(const unsigned *TableAdd_GPU, float *TMM_deltaU, float *TMM_Min1, float *TMM_Min2, int *TMM_Min1_Col, float *TMM_I, int *TMM_Path, float *TMM_E)
{
    // dQ[0]
    TMM_I[0] = 0;
    TMM_Path[0] = TMM_Path[1] = -1;
    TMM_E[0] = 0;

    double deviation1, deviation2;
    for (int i = 1; i < GFQ; i++)
    {
        // 1 deviation
        TMM_I[i] = TMM_deltaU[TMM_Min1_Col[i] * GFQ + i];
        TMM_Path[i * 2 + 0] = TMM_Path[i * 2 + 1] = TMM_Min1_Col[i];
        TMM_E[i] = TMM_Min2[i];

        // 2 deviation
        for (int j = 0; j < GFQ; j++)
        {
            if (j != i)
            {
                int k = GFAdd_GPU(i, j, TableAdd_GPU);
                if (TMM_Min1_Col[j] != TMM_Min1_Col[k]) // 不在同一列
                {
                    deviation1 = TMM_deltaU[TMM_Min1_Col[j] * GFQ + j];
                    deviation2 = TMM_deltaU[TMM_Min1_Col[k] * GFQ + k];
                    if (deviation1 > deviation2 && deviation1 < TMM_I[i])
                    {
                        TMM_I[i] = deviation1;
                        TMM_Path[i * 2 + 0] = TMM_Min1_Col[j];
                        TMM_Path[i * 2 + 1] = TMM_Min1_Col[k];
                        TMM_E[i] = TMM_Min1[i];
                    }
                    else if (deviation1 < deviation2 &&
                             deviation2 < TMM_I[i])
                    {
                        TMM_I[i] = deviation2;
                        TMM_Path[i * 2 + 0] = TMM_Min1_Col[j];
                        TMM_Path[i * 2 + 1] = TMM_Min1_Col[k];
                        TMM_E[i] = TMM_Min1[i];
                    }
                }
            }
        }
    }
    return 0;
}