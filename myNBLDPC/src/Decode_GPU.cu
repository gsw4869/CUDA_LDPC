#include "hip/hip_runtime.h"
#include "Decode_GPU.cuh"
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <assert.h>

__device__ int GFAdd_GPU(int ele1, int ele2, unsigned *TableAdd_GPU)
{
    if (ele1 >= GFQ | ele2 >= GFQ)
    {
        printf("error");
    }
    return TableAdd_GPU[GFQ * ele1 + ele2];
}

__device__ int GFMultiply_GPU(int ele1, int ele2, unsigned *TableMultiply_GPU)
{
    if (ele1 >= GFQ | ele2 >= GFQ)
    {
        printf("error");
    }
    return TableMultiply_GPU[GFQ * ele1 + ele2];
}

__device__ int GFInverse_GPU(int ele, unsigned *TableInverse_GPU)
{
    if (ele == 0)
    {
        printf("Div 0 Error!\n");
    }
    return TableInverse_GPU[ele];
}

__device__ int index_in_VN_GPU(int *Checknode_linkVNs, int Checknode_num, int index_in_linkVNs, int *Variablenode_linkCNs)
{
    for (int i = 0; i < maxdv; i++)
    {
        if (Variablenode_linkCNs[maxdv * Checknode_linkVNs[maxdc * Checknode_num + index_in_linkVNs] + i] == Checknode_num)
        {
            return i;
        }
    }
    printf("index_in_VN_GPU error\n");
}

int Decoding_EMS_GPU(LDPCCode *H, VN *Variablenode, CN *Checknode, int EMS_Nm, int EMS_Nc, int *DecodeOutput, unsigned *TableMultiply_GPU, unsigned *TableAdd_GPU, int *Checknode_weight, int *Variablenode_linkCNs, int *Checknode_linkVNs, int *Checknode_linkVNs_GF)
{

    for (int col = 0; col < H->Variablenode_num; col++)
    {
        for (int d = 0; d < Variablenode[col].weight; d++)
        {
            for (int q = 0; q < H->GF; q++)
            {
                Variablenode[col].Entr_v2c[d][q] = Variablenode[col].L_ch[q];
            }
        }
    }
    for (int row = 0; row < H->Checknode_num; row++)
    {
        for (int d = 0; d < Checknode[row].weight; d++)
        {
            for (int q = 0; q < H->GF - 1; q++)
            {
                Checknode[row].L_c2v[d][q] = 0;
            }
        }
    }
    int *sort_Entr_v2c_temp = (int *)malloc(H->Variablenode_num * maxdv * GFQ * sizeof(int));
    memset(sort_Entr_v2c_temp, 0, H->Variablenode_num * maxdv * GFQ * sizeof(int));
    int *sort_Entr_v2c;
    hipMalloc((void **)&sort_Entr_v2c, H->Variablenode_num * maxdv * GFQ * sizeof(int));

    float *sort_L_v2c_temp = (float *)malloc(H->Variablenode_num * maxdv * GFQ * sizeof(float));
    memset(sort_L_v2c_temp, 0, H->Variablenode_num * maxdv * GFQ * sizeof(float));
    float *sort_L_v2c;
    hipMalloc((void **)&sort_L_v2c, H->Variablenode_num * maxdv * GFQ * sizeof(float));

    float *Checknode_L_c2v_temp = (float *)malloc(H->Checknode_num * maxdc * GFQ * sizeof(float));
    memset(Checknode_L_c2v_temp, 0, H->Checknode_num * maxdc * GFQ * sizeof(float));
    float *Checknode_L_c2v;
    hipMalloc((void **)&Checknode_L_c2v, H->Checknode_num * maxdc * GFQ * sizeof(float));

    int iter_number = 0;
    bool decode_correct = true;
    while (iter_number++ < maxIT)
    {
        // printf("it_time: %d\n",iter_number);
        for (int col = 0; col < H->Variablenode_num; col++)
        {
            for (int d = 0; d < Variablenode[col].weight; d++)
            {
                for (int q = 0; q < H->GF - 1; q++)
                {
                    Variablenode[col].LLR[q] = Variablenode[col].L_ch[q];
                }
            }
        }
        for (int col = 0; col < H->Variablenode_num; col++)
        {
            for (int d = 0; d < Variablenode[col].weight; d++)
            {
                for (int q = 0; q < H->GF - 1; q++)
                {
                    Variablenode[col].LLR[q] += Checknode[Variablenode[col].linkCNs[d]].L_c2v[index_in_CN(Variablenode, col, d, Checknode)][q];
                }
            }
            DecodeOutput[col] = DecideLLRVector(Variablenode[col].LLR, H->GF);
            // printf("%d ", DecodeOutput[col]);
        }
        // printf("\n");

        decode_correct = true;
        int sum_temp = 0;
        for (int row = 0; row < H->Checknode_num; row++)
        {
            for (int i = 0; i < Checknode[row].weight; i++)
            {
                sum_temp = GFAdd(sum_temp, GFMultiply(DecodeOutput[Checknode[row].linkVNs[i]], Checknode[row].linkVNs_GF[i]));
            }
            if (sum_temp)
            {
                decode_correct = false;
                break;
            }
        }
        if (decode_correct)
        {
            return 1;
        }

        // message from var to check
        for (int col = 0; col < H->Variablenode_num; col++)
        {
            for (int dv = 0; dv < Variablenode[col].weight; dv++)
            {
                for (int q = 0; q < H->GF - 1; q++)
                {
                    Variablenode[col].Entr_v2c[dv][q] = Variablenode[col].LLR[q] - Checknode[Variablenode[col].linkCNs[dv]].L_c2v[index_in_CN(Variablenode, col, dv, Checknode)][q];
                }
            }
        }

        int *index = (int *)malloc((H->GF) * sizeof(int));
        for (int col = 0; col < H->Variablenode_num; col++)
        {
            memcpy(Variablenode[col].sort_L_v2c[0], Variablenode[col].Entr_v2c[0], Variablenode[col].weight * H->GF * sizeof(float));

            for (int dv = 0; dv < Variablenode[col].weight; dv++)
            {
                for (int i = 0; i < H->GF - 1; i++)
                {
                    index[i] = i + 1;
                }
                index[H->GF - 1] = 0;
                SortLLRVector(H->GF, Variablenode[col].sort_L_v2c[dv], index);
                for (int i = 0; i < H->GF; i++)
                {
                    Variablenode[col].sort_Entr_v2c[dv][i] = index[i];

                    sort_Entr_v2c_temp[col * maxdv * GFQ + dv * GFQ + i] = index[i];
                    sort_L_v2c_temp[col * maxdv * GFQ + dv * GFQ + i] = Variablenode[col].sort_L_v2c[dv][i];
                }
            }
        }
        hipError_t cudaStatus;
        cudaStatus = hipMemcpy(sort_Entr_v2c, sort_Entr_v2c_temp, H->Variablenode_num * maxdv * GFQ * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("Cannot copy sort_Entr_v2c\n");
            exit(0);
        }
        cudaStatus = hipMemcpy(sort_L_v2c, sort_L_v2c_temp, H->Variablenode_num * maxdv * GFQ * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("Cannot copy sort_L_v2c\n");
            exit(0);
        }
        // // message from check to var

        Checknode_EMS<<<((H->Checknode_num % 128) ? (H->Checknode_num / 128 + 1) : (H->Checknode_num / 128)), 128>>>(TableMultiply_GPU, TableAdd_GPU, EMS_Nm, EMS_Nc, Checknode_weight, Variablenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c, Checknode_L_c2v, H->Checknode_num);
        // Checknode_EMS<<<1, 1>>>(TableMultiply_GPU, TableAdd_GPU, EMS_Nm, EMS_Nc, Checknode_weight, Variablenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c, Checknode_L_c2v, H->Checknode_num);

        cudaStatus = hipMemcpy(Checknode_L_c2v_temp, Checknode_L_c2v, H->Checknode_num * maxdc * GFQ * sizeof(float), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            printf("Cannot copy Checknode_L_c2v\n");
            exit(0);
        }

        for (int i = 0; i < H->Checknode_num; i++)
        {
            for (int j = 0; j < Checknode[i].weight; j++)
            {
                for (int q = 0; q < GFQ - 1; q++)
                {
                    Checknode[i].L_c2v[j][q] = Checknode_L_c2v_temp[i * maxdc * GFQ + j * GFQ + q];
                }
            }
        }
    }
    hipFree(sort_Entr_v2c);
    hipFree(sort_L_v2c);
    hipFree(Checknode_L_c2v);
    free(sort_Entr_v2c_temp);
    free(sort_L_v2c_temp);
    free(Checknode_L_c2v_temp);
    return 0;
}

/*
Checknode_weight:每一个校验节点的重量
L_c2v:Q个信息，Q个信息，Q个信息，一共校验节点数量*Q个
Variblenode_linkCNs:最大重量dv，每dv个元素代表连接的dv个校验节点的序号
Checknode_linkVNS:最大重量dc，每dc个元素代表连接的dc个变量节点的序号
Checknode_linkVNS_GF:最大重量dc，每dc个元素代表连接的dc个变量节点的多元域值
sort_Entr_v2c:每个变量节点重量dv，q,q,q一共dv个，然后再乘以变量节点个数[变量节点个数][变量节点重量][q]
sort_L_v2c:和sort_Entr_v2c对应的LLR
Checknode_L_c2v:每个校验节点重量dc，q一共dc个，然后再乘以变量节点个数[校验节点个数][校验节点重量][q]
*/
__global__ void Checknode_EMS(unsigned *TableMultiply_GPU, unsigned *TableAdd_GPU, int EMS_Nm, int EMS_Nc, int *Checknode_weight, int *Variblenode_linkCNs, int *Checknode_linkVNs, int *Checknode_linkVNs_GF, int *sort_Entr_v2c, float *sort_L_v2c, float *Checknode_L_c2v, int Checknode_num)
{
    int offset;
    offset = threadIdx.x + blockDim.x * blockIdx.x;
    if (offset < Checknode_num)
    {
        float EMS_L_c2v[GFQ];
        for (int dc = 0; dc < maxdc; dc++)
        {
            if (dc < Checknode_weight[offset])
            {
                // reset the sum store vector to the munimum
                for (int q = 0; q < GFQ; q++)
                {
                    EMS_L_c2v[q] = -DBL_MAX;
                }

                // recursly exhaustly
                int sumNonele, diff;
                float sumNonLLR;
                // conf(q, 1)
                sumNonele = 0;
                sumNonLLR = 0;
                diff = 0;
                ConstructConf_GPU(TableMultiply_GPU, TableAdd_GPU, GFQ, 1, sumNonele, sumNonLLR, diff, 0, dc, Checknode_weight[offset] - 1, offset, EMS_L_c2v, Variblenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);

                // conf(nm, nc)
                // sumNonele = 0;
                // sumNonLLR = 0;
                // diff = 0;
                // ConstructConf_GPU(TableMultiply_GPU, TableAdd_GPU, EMS_Nm, EMS_Nc, sumNonele, sumNonLLR, diff, 0, dc, Checknode_weight[offset] - 1, offset, EMS_L_c2v, Variblenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);

                // calculate each c2v LLR
                int v = 0;
                Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + GFQ - 1] = 0;
                for (int k = 1; k < GFQ; k++)
                {
                    v = GFMultiply_GPU(k, Checknode_linkVNs_GF[offset * maxdc + dc], TableMultiply_GPU);
                    Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + k - 1] = (EMS_L_c2v[v] - EMS_L_c2v[0]) / 1.2;
                }
            }
            else
            {
                for (int k = 0; k < GFQ; k++)
                {
                    Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + k] = 0;
                }
            }
        }
    }
}
__device__ int ConstructConf_GPU(unsigned *TableMultiply_GPU, unsigned *TableAdd_GPU, int Nm, int Nc, int &sumNonele, float &sumNonLLR, int &diff, int begin, int except, int end, int row, float *EMS_L_c2v, int *Variblenode_linkCNs, int *Checknode_linkVNs, int *Checknode_linkVNs_GF, int *sort_Entr_v2c, float *sort_L_v2c)
{
    // if (begin > end)
    // {
    //     if (sumNonLLR > EMS_L_c2v[sumNonele])
    //     {
    //         EMS_L_c2v[sumNonele] = sumNonLLR;
    //     }
    // }
    // else if (begin == except)
    // {
    //     ConstructConf_GPU(TableMultiply_GPU, TableAdd_GPU, Nm, Nc, sumNonele, sumNonLLR, diff, begin + 1, except, end, row, EMS_L_c2v, Variblenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);
    //     return 0;
    // }
    // else
    // {
    //     int index = index_in_VN_GPU(Checknode_linkVNs, row, begin, Variblenode_linkCNs);
    //     for (int k = 0; k < Nm; k++)
    //     {

    //         sumNonele = GFAdd_GPU(GFMultiply_GPU(23, 45, TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //         sumNonLLR = sumNonLLR + 0.3;

    //         // sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + begin], TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //         // sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k];
    //         diff += (k != 0) ? 1 : 0;
    //         if (diff <= Nc)
    //         {
    //             ConstructConf_GPU(TableMultiply_GPU, TableAdd_GPU, Nm, Nc, sumNonele, sumNonLLR, diff, begin + 1, except, end, row, EMS_L_c2v, Variblenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);
    //             // sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + begin], TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //             // sumNonLLR = sumNonLLR - sort_L_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k];

    //             sumNonele = GFAdd_GPU(GFMultiply_GPU(21, 25, TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //             sumNonLLR = sumNonLLR - 0.3;

    //             diff -= (k != 0) ? 1 : 0;
    //         }
    //         else
    //         {
    //             // sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + begin], TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //             // sumNonLLR = sumNonLLR - sort_L_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k];

    //             sumNonele = GFAdd_GPU(GFMultiply_GPU(34, 42, TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //             sumNonLLR = sumNonLLR - 0.3;

    //             diff -= (k != 0) ? 1 : 0;
    //             break;
    //         }
    //     }
    // }
    // return 0;
    for (int i = 0; i < 4; i++)
    {
        if (i == except)
        {
            continue;
        }
        int index = index_in_VN_GPU(Checknode_linkVNs, row, i, Variblenode_linkCNs);
        sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + i] * maxdv * GFQ + index * GFQ], Checknode_linkVNs_GF[row * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
        sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[row * maxdc + i] * maxdv * GFQ + index * GFQ];
    }
    if (sumNonLLR > EMS_L_c2v[sumNonele])
    {
        EMS_L_c2v[sumNonele] = sumNonLLR;
    }
    for (int i = 0; i < 4; i++)
    {
        if (i == except)
        {
            continue;
        }
        for (int k = 1; k < GFQ; k++)
        {

            int index = index_in_VN_GPU(Checknode_linkVNs, row, i, Variblenode_linkCNs);
            sumNonele = 0;
            sumNonLLR = 0;
            sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + i] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
            sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[row * maxdc + i] * maxdv * GFQ + index * GFQ + k];

            for (int j = 0; j < 4; j++)
            {
                if (j == i | j == except)
                {
                    continue;
                }
                int index = index_in_VN_GPU(Checknode_linkVNs, row, j, Variblenode_linkCNs);
                sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + j] * maxdv * GFQ + index * GFQ], Checknode_linkVNs_GF[row * maxdc + j], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[row * maxdc + j] * maxdv * GFQ + index * GFQ];
            }
            if (sumNonLLR > EMS_L_c2v[sumNonele])
            {
                EMS_L_c2v[sumNonele] = sumNonLLR;
            }
        }
    }
}

void GPUArray_initial(LDPCCode *H, VN *Variablenode, CN *Checknode, int *Checknode_weight, int *Variablenode_linkCNs, int *Checknode_linkVNs, int *Checknode_linkVNs_GF)
{
    // int *Checknode_weight;
    hipError_t cudaStatus;

    hipMalloc((void **)&Checknode_weight, H->Checknode_num * sizeof(int));

    int *Checknode_weight_temp = (int *)malloc(H->Checknode_num * sizeof(int));
    for (int i = 0; i < H->Checknode_num; i++)
    {
        Checknode_weight_temp[i] = Checknode[i].weight;
    }
    cudaStatus = hipMemcpy(Checknode_weight, Checknode_weight_temp, H->Checknode_num * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy Checknode_weight\n");
        exit(0);
    }
    free(Checknode_weight_temp);

    // int *Variablenode_linkCNs;
    hipMalloc((void **)&Variablenode_linkCNs, H->Variablenode_num * maxdv * sizeof(int));

    int *Variablenode_linkCNs_temp = (int *)malloc(H->Variablenode_num * maxdv * sizeof(int));
    for (int i = 0; i < H->Variablenode_num; i++)
    {
        for (int j = 0; j < Variablenode[i].weight; j++)
        {
            Variablenode_linkCNs_temp[i * maxdv + j] = Variablenode[i].linkCNs[j];
        }
    }
    cudaStatus = hipMemcpy(Variablenode_linkCNs, Variablenode_linkCNs_temp, H->Variablenode_num * maxdv * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy Variablenode_linkCNs\n");
        exit(0);
    }
    free(Variablenode_linkCNs_temp);

    // int *Checknode_linkVNs;
    hipMalloc((void **)&Checknode_linkVNs, H->Checknode_num * maxdc * sizeof(int));

    int *Checknode_linkVNs_temp = (int *)malloc(H->Checknode_num * maxdc * sizeof(int));
    for (int i = 0; i < H->Checknode_num; i++)
    {
        for (int j = 0; j < Checknode[i].weight; j++)
        {
            Checknode_linkVNs_temp[i * maxdc + j] = Checknode[i].linkVNs[j];
        }
    }
    cudaStatus = hipMemcpy(Checknode_linkVNs, Checknode_linkVNs_temp, H->Checknode_num * maxdc * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy Checknode_linkVNs\n");
        exit(0);
    }
    free(Checknode_linkVNs_temp);

    // int *Checknode_linkVNs_GF;
    hipMalloc((void **)&Checknode_linkVNs_GF, H->Checknode_num * maxdc * sizeof(int));

    int *Checknode_linkVNs_GF_temp = (int *)malloc(H->Checknode_num * maxdc * sizeof(int));
    for (int i = 0; i < H->Checknode_num; i++)
    {
        for (int j = 0; j < Checknode[i].weight; j++)
        {
            Checknode_linkVNs_GF_temp[i * maxdc + j] = Checknode[i].linkVNs_GF[j];
        }
    }
    cudaStatus = hipMemcpy(Checknode_linkVNs_GF, Checknode_linkVNs_GF_temp, H->Checknode_num * maxdc * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy Checknode_linkVNs_GF\n");
        exit(0);
    }
    free(Checknode_linkVNs_GF_temp);
}