#include "hip/hip_runtime.h"
#include "Decode_GPU.cuh"
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <assert.h>

__device__ int GFAdd_GPU(int ele1, int ele2, unsigned *TableAdd_GPU)
{
    return TableAdd_GPU[GFQ * ele1 + ele2];
}

__device__ int GFMultiply_GPU(int ele1, int ele2, unsigned *TableMultiply_GPU)
{
    return TableMultiply_GPU[GFQ * ele1 + ele2];
}

__device__ int GFInverse_GPU(int ele, unsigned *TableInverse_GPU)
{
    if (ele == 0)
    {
        printf("Div 0 Error!\n");
    }
    return TableInverse_GPU[ele];
}

__device__ int index_in_VN_GPU(int *Checknode_linkVNs, int Checknode_num, int index_in_linkVNs, int *Variablenode_linkCNs)
{
    for (int i = 0; i < maxdv; i++)
    {
        if (Variablenode_linkCNs[maxdv * Checknode_linkVNs[maxdc * Checknode_num + index_in_linkVNs] + i] == Checknode_num)
        {
            return i;
        }
    }
    printf("index_in_VN_GPU error\n");
}

int Decoding_EMS_GPU(LDPCCode *H, VN *Variablenode, CN *Checknode, int EMS_Nm, int EMS_Nc, int *DecodeOutput)
{

    for (int col = 0; col < H->Variablenode_num; col++)
    {
        for (int d = 0; d < Variablenode[col].weight; d++)
        {
            for (int q = 0; q < H->GF; q++)
            {
                Variablenode[col].Entr_v2c[d][q] = Variablenode[col].L_ch[q];
            }
        }
    }
    for (int row = 0; row < H->Checknode_num; row++)
    {
        for (int d = 0; d < Checknode[row].weight; d++)
        {
            for (int q = 0; q < H->GF - 1; q++)
            {
                Checknode[row].L_c2v[d][q] = 0;
            }
        }
    }

    int iter_number = 0;
    bool decode_correct = true;
    while (iter_number++ < maxIT)
    {
        // printf("it_time: %d\n",iter_number);
        for (int col = 0; col < H->Variablenode_num; col++)
        {
            for (int d = 0; d < Variablenode[col].weight; d++)
            {
                for (int q = 0; q < H->GF - 1; q++)
                {
                    Variablenode[col].LLR[q] = Variablenode[col].L_ch[q];
                }
            }
        }
        for (int col = 0; col < H->Variablenode_num; col++)
        {
            for (int d = 0; d < Variablenode[col].weight; d++)
            {
                for (int q = 0; q < H->GF - 1; q++)
                {
                    Variablenode[col].LLR[q] += Checknode[Variablenode[col].linkCNs[d]].L_c2v[index_in_CN(Variablenode, col, d, Checknode)][q];
                }
            }
            DecodeOutput[col] = DecideLLRVector(Variablenode[col].LLR, H->GF);
            // printf("%d ", DecodeOutput[col]);
        }
        // printf("\n");

        decode_correct = true;
        int sum_temp = 0;
        for (int row = 0; row < H->Checknode_num; row++)
        {
            for (int i = 0; i < Checknode[row].weight; i++)
            {
                sum_temp = GFAdd(sum_temp, GFMultiply(DecodeOutput[Checknode[row].linkVNs[i]], Checknode[row].linkVNs_GF[i]));
            }
            if (sum_temp)
            {
                decode_correct = false;
                break;
            }
        }
        if (decode_correct)
        {
            return 1;
        }

        // message from var to check
        for (int col = 0; col < H->Variablenode_num; col++)
        {
            for (int dv = 0; dv < Variablenode[col].weight; dv++)
            {
                for (int q = 0; q < H->GF - 1; q++)
                {
                    Variablenode[col].Entr_v2c[dv][q] = Variablenode[col].LLR[q] - Checknode[Variablenode[col].linkCNs[dv]].L_c2v[index_in_CN(Variablenode, col, dv, Checknode)][q];
                }
            }
        }

        int *index = (int *)malloc((H->GF) * sizeof(int));
        for (int col = 0; col < H->Variablenode_num; col++)
        {
            memcpy(Variablenode[col].sort_L_v2c[0], Variablenode[col].Entr_v2c[0], Variablenode[col].weight * H->GF * sizeof(float));

            for (int dv = 0; dv < Variablenode[col].weight; dv++)
            {
                for (int i = 0; i < H->GF - 1; i++)
                {
                    index[i] = i + 1;
                }
                index[H->GF - 1] = 0;
                SortLLRVector(H->GF, Variablenode[col].sort_L_v2c[dv], index);
                for (int i = 0; i < H->GF; i++)
                {
                    Variablenode[col].sort_Entr_v2c[dv][i] = index[i];
                }
            }
        }

        float *EMS_L_c2v = (float *)malloc(H->GF * sizeof(float));

        // message from check to var
        for (int row = 0; row < H->Checknode_num; row++)
        {
        }
        free(EMS_L_c2v);
    }
    return 0;
}

/*
Checknode_weight:每一个校验节点的重量
L_c2v:Q个信息，Q个信息，Q个信息，一共校验节点数量*Q个
Variblenode_linkCNs:最大重量dv，每dv个元素代表连接的dv个校验节点的序号
Checknode_linkVNS:最大重量dc，每dc个元素代表连接的dc个变量节点的序号
Checknode_linkVNS_GF:最大重量dc，每dc个元素代表连接的dc个变量节点的多元域值
sort_Entr_v2c:每个变量节点重量dv，q,q,q一共dv个，然后再乘以变量节点个数[变量节点个数][变量节点重量][q]
sort_L_v2c:和sort_Entr_v2c对应的LLR
*/
__global__ void Checknode_EMS(unsigned *TableMultiply_GPU, unsigned *TableAdd_GPU, int EMS_Nm, int EMS_Nc, int *Checknode_weight, float *L_c2v, int *Variblenode_linkCNs, int *Checknode_linkVNs, int *Checknode_linkVNs_GF, int *sort_Entr_v2c, int *sort_L_v2c)
{
    int offset;
    offset = threadIdx.x + blockDim.x * blockIdx.x;
    float EMS_L_c2v[GFQ];
    for (int dc = 0; dc < Checknode_weight[offset]; dc++)
    {
        // reset the sum store vector to the munimum
        for (int q = 0; q < GFQ; q++)
        {
            EMS_L_c2v[q] = -DBL_MAX;
        }

        // recursly exhaustly
        int sumNonele, diff;
        float sumNonLLR;
        // conf(q, 1)
        sumNonele = 0;
        sumNonLLR = 0;
        diff = 0;
        ConstructConf_GPU(TableMultiply_GPU, TableAdd_GPU, GFQ, 1, sumNonele, sumNonLLR, diff, 0, dc, Checknode_weight[offset] - 1, offset, EMS_L_c2v, Variblenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs, sort_Entr_v2c, sort_L_v2c);

        // conf(nm, nc)
        sumNonele = 0;
        sumNonLLR = 0;
        diff = 0;
        ConstructConf_GPU(TableMultiply_GPU, TableAdd_GPU, EMS_Nm, EMS_Nc, sumNonele, sumNonLLR, diff, 0, dc, Checknode_weight[offset] - 1, offset, EMS_L_c2v, Variblenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs, sort_Entr_v2c, sort_L_v2c);

        // calculate each c2v LLR
        // int v = 0;
        // for (int k = 1; k < GFQ; k++)
        // {
        // 	v = GFMultiply_GPU(k, Checknode[row].linkVNs_GF[dc]);
        // 	Checknode[row].L_c2v[dc][k - 1] = (EMS_L_c2v[v] - EMS_L_c2v[0]) / 1.2;
        // }
    }
}
__device__ int ConstructConf_GPU(unsigned *TableMultiply_GPU, unsigned *TableAdd_GPU, int Nm, int Nc, int &sumNonele, float &sumNonLLR, int &diff, int begin, int except, int end, int row, float *EMS_L_c2v, int *Variblenode_linkCNs, int *Checknode_linkVNs, int *Checknode_linkVNs_GF, int *sort_Entr_v2c, int *sort_L_v2c)
{
    int index;
    if (begin > end)
    {
        if (sumNonLLR > EMS_L_c2v[sumNonele])
        {
            EMS_L_c2v[sumNonele] = sumNonLLR;
        }
    }
    else if (begin == except)
    {
        ConstructConf_GPU(TableMultiply_GPU, TableAdd_GPU, Nm, Nc, sumNonele, sumNonLLR, diff, begin + 1, except, end, row, EMS_L_c2v, Variblenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs, sort_Entr_v2c, sort_L_v2c);
        return 0;
    }
    else
    {
        index = index_in_VN_GPU(Checknode_linkVNs, row, begin, Variblenode_linkCNs);
        for (int k = 0; k < Nm; k++)
        {
            sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + begin], TableMultiply_GPU), sumNonele, TableAdd_GPU);
            sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k];
            diff += (k != 0) ? 1 : 0;
            if (diff <= Nc)
            {
                ConstructConf_GPU(TableMultiply_GPU, TableAdd_GPU, Nm, Nc, sumNonele, sumNonLLR, diff, begin + 1, except, end, row, EMS_L_c2v, Variblenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs, sort_Entr_v2c, sort_L_v2c);
                sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + begin], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                sumNonLLR = sumNonLLR - sort_L_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k];
                diff -= (k != 0) ? 1 : 0;
            }
            else
            {
                sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + begin], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                sumNonLLR = sumNonLLR - sort_L_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k];
                diff -= (k != 0) ? 1 : 0;
                break;
            }
        }
    }
    return 0;
}
