#include "hip/hip_runtime.h"
#include "Decode_GPU.cuh"
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <assert.h>

__device__ int GFAdd_GPU(int ele1, int ele2, const unsigned *TableAdd_GPU)
{
    return ele1 ^ ele2;
}

__device__ int GFMultiply_GPU(int ele1, int ele2, const unsigned *TableMultiply_GPU)
{
    return TableMultiply_GPU[GFQ * ele1 + ele2];
}

__device__ int GFInverse_GPU(int ele, const unsigned *TableInverse_GPU)
{
    if (ele == 0)
    {
        printf("Div 0 Error!\n");
    }
    return TableInverse_GPU[ele];
}

int Decoding_EMS_GPU(const LDPCCode *H, VN *Variablenode, CN *Checknode, int EMS_Nm, int EMS_Nc, int *DecodeOutput, const unsigned *TableMultiply_GPU, const unsigned *TableAdd_GPU, const int *Checknode_weight, const int *Variablenode_linkCNs, const int *Checknode_linkVNs, const int *Checknode_linkVNs_GF, int &iter_number)
{

    for (int col = 0; col < H->Variablenode_num; col++)
    {
        for (int d = 0; d < Variablenode[col].weight; d++)
        {
            memcpy(Variablenode[col].sort_L_v2c[d], Variablenode[col].L_ch, (GFQ - 1) * sizeof(float));
        }
    }

    for (int row = 0; row < H->Checknode_num; row++)
    {
        for (int d = 0; d < Checknode[row].weight; d++)
        {
            memset(Checknode[row].L_c2v[d], 0, (GFQ - 1) * sizeof(float));
        }
    }
    int *sort_Entr_v2c_temp = (int *)malloc(H->Variablenode_num * maxdv * GFQ * sizeof(int));
    memset(sort_Entr_v2c_temp, 0, H->Variablenode_num * maxdv * GFQ * sizeof(int));
    int *sort_Entr_v2c;
    hipMalloc((void **)&sort_Entr_v2c, H->Variablenode_num * maxdv * GFQ * sizeof(int));

    float *sort_L_v2c_temp = (float *)malloc(H->Variablenode_num * maxdv * GFQ * sizeof(float));
    memset(sort_L_v2c_temp, 0, H->Variablenode_num * maxdv * GFQ * sizeof(float));
    float *sort_L_v2c;
    hipMalloc((void **)&sort_L_v2c, H->Variablenode_num * maxdv * GFQ * sizeof(float));

    float *Checknode_L_c2v_temp = (float *)malloc(H->Checknode_num * maxdc * GFQ * sizeof(float));
    memset(Checknode_L_c2v_temp, 0, H->Checknode_num * maxdc * GFQ * sizeof(float));
    float *Checknode_L_c2v;
    hipMalloc((void **)&Checknode_L_c2v, H->Checknode_num * maxdc * GFQ * sizeof(float));

    int *index = (int *)malloc((GFQ) * sizeof(int));

    iter_number = 0;
    bool decode_correct = true;
    while (iter_number++ < maxIT - 1)
    {
        for (int col = 0; col < H->Variablenode_num; col++)
        {
            memcpy(Variablenode[col].LLR, Variablenode[col].L_ch, (GFQ - 1) * sizeof(float));
        }
        for (int col = 0; col < H->Variablenode_num; col++)
        {
            for (int d = 0; d < Variablenode[col].weight; d++)
            {
                for (int q = 0; q < GFQ - 1; q++)
                {
                    Variablenode[col].LLR[q] += Checknode[Variablenode[col].linkCNs[d]].L_c2v[index_in_CN(Variablenode, col, d, Checknode)][q];
                }
            }
            DecodeOutput[col] = DecideLLRVector(Variablenode[col].LLR, GFQ);
        }

        decode_correct = true;
        int sum_temp = 0;
        for (int row = 0; row < H->Checknode_num; row++)
        {
            for (int i = 0; i < Checknode[row].weight; i++)
            {
                sum_temp = GFAdd(sum_temp, GFMultiply(DecodeOutput[Checknode[row].linkVNs[i]], Checknode[row].linkVNs_GF[i]));
            }
            if (sum_temp)
            {
                decode_correct = false;
                break;
            }
        }
        if (decode_correct)
        {
            hipFree(sort_Entr_v2c);
            hipFree(sort_L_v2c);
            hipFree(Checknode_L_c2v);
            free(index);
            free(sort_Entr_v2c_temp);
            free(sort_L_v2c_temp);
            free(Checknode_L_c2v_temp);
            return 1;
        }

        // message from var to check
        for (int col = 0; col < H->Variablenode_num; col++)
        {
            for (int dv = 0; dv < Variablenode[col].weight; dv++)
            {
                for (int q = 0; q < GFQ - 1; q++)
                {
                    Variablenode[col].sort_L_v2c[dv][q] = Variablenode[col].LLR[q] - Checknode[Variablenode[col].linkCNs[dv]].L_c2v[index_in_CN(Variablenode, col, dv, Checknode)][q];
                }
                Variablenode[col].sort_L_v2c[dv][GFQ - 1] = 0;
            }
        }

        for (int col = 0; col < H->Variablenode_num; col++)
        {
            for (int dv = 0; dv < Variablenode[col].weight; dv++)
            {
                for (int i = 0; i < GFQ - 1; i++)
                {
                    index[i] = i + 1;
                }
                index[GFQ - 1] = 0;
                SortLLRVector(GFQ, Variablenode[col].sort_L_v2c[dv], index);
                for (int i = 0; i < GFQ; i++)
                {
                    Variablenode[col].sort_Entr_v2c[dv][i] = index[i];

                    sort_Entr_v2c_temp[col * maxdv * GFQ + dv * GFQ + i] = index[i];
                    sort_L_v2c_temp[col * maxdv * GFQ + dv * GFQ + i] = Variablenode[col].sort_L_v2c[dv][i];
                }
            }
        }
        hipError_t cudaStatus;
        cudaStatus = hipMemcpy(sort_Entr_v2c, sort_Entr_v2c_temp, H->Variablenode_num * maxdv * GFQ * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("Cannot copy sort_Entr_v2c\n");
            exit(0);
        }
        cudaStatus = hipMemcpy(sort_L_v2c, sort_L_v2c_temp, H->Variablenode_num * maxdv * GFQ * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("Cannot copy sort_L_v2c\n");
            exit(0);
        }
        // // message from check to var

        Checknode_EMS<<<((H->Checknode_num % 128) ? (H->Checknode_num / 128 + 1) : (H->Checknode_num / 128)), 128>>>((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, EMS_Nm, EMS_Nc, (const int *)Checknode_weight, (const int *)Variablenode_linkCNs, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c, Checknode_L_c2v, H->Checknode_num);
        // Checknode_EMS<<<1, 1>>>((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, EMS_Nm, EMS_Nc, (const int *)Checknode_weight, (const int *)Variablenode_linkCNs, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c, Checknode_L_c2v, H->Checknode_num);

        cudaStatus = hipMemcpy(Checknode_L_c2v_temp, Checknode_L_c2v, H->Checknode_num * maxdc * GFQ * sizeof(float), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            printf("Cannot copy Checknode_L_c2v\n");
            exit(0);
        }

        for (int i = 0; i < H->Checknode_num; i++)
        {
            for (int j = 0; j < Checknode[i].weight; j++)
            {
                for (int q = 0; q < GFQ - 1; q++)
                {
                    Checknode[i].L_c2v[j][q] = Checknode_L_c2v_temp[i * maxdc * GFQ + j * GFQ + q];
                }
            }
        }
    }
    hipFree(sort_Entr_v2c);
    hipFree(sort_L_v2c);
    hipFree(Checknode_L_c2v);
    free(index);
    free(sort_Entr_v2c_temp);
    free(sort_L_v2c_temp);
    free(Checknode_L_c2v_temp);
    return 0;
}

/*
Checknode_weight:每一个校验节点的重量
L_c2v:Q个信息，Q个信息，Q个信息，一共校验节点数量*Q个
Variblenode_linkCNs:最大重量dv，每dv个元素代表连接的dv个校验节点的序号
Checknode_linkVNS:最大重量dc，每dc个元素代表连接的dc个变量节点的序号
Checknode_linkVNS_GF:最大重量dc，每dc个元素代表连接的dc个变量节点的多元域值
sort_Entr_v2c:每个变量节点重量dv，q,q,q一共dv个，然后再乘以变量节点个数[变量节点个数][变量节点重量][q]
sort_L_v2c:和sort_Entr_v2c对应的LLR
Checknode_L_c2v:每个校验节点重量dc，q一共dc个，然后再乘以变量节点个数[校验节点个数][校验节点重量][q]
*/
__global__ void Checknode_EMS(const unsigned *TableMultiply_GPU, const unsigned *TableAdd_GPU, int EMS_Nm, int EMS_Nc, const int *Checknode_weight, const int *Variblenode_linkCNs, const int *Checknode_linkVNs, const int *Checknode_linkVNs_GF, int *sort_Entr_v2c, float *sort_L_v2c, float *Checknode_L_c2v, int Checknode_num)
{
    int offset;
    offset = threadIdx.x + blockDim.x * blockIdx.x;
    if (offset < Checknode_num)
    {
        float EMS_L_c2v[GFQ];
        for (int dc = 0; dc < maxdc; dc++)
        {
            if (dc < Checknode_weight[offset])
            {
                // reset the sum store vector to the munimum
                for (int q = 0; q < GFQ; q++)
                {
                    EMS_L_c2v[q] = -DBL_MAX;
                }

                // recursly exhaustly
                int sumNonele;
                float sumNonLLR;
                // conf(q, 1)
                sumNonele = 0;
                sumNonLLR = 0;
                // ConstructConf_GPU((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, GFQ, 1, sumNonele, sumNonLLR, diff, 0, dc, Checknode_weight[offset] - 1, offset, EMS_L_c2v, (const int *)Variblenode_linkCNs, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);

                for (int i = 0; i < Checknode_weight[offset]; i++)
                {
                    if (i == dc)
                    {
                        continue;
                    }

                    sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                    sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i]];
                }
                if (sumNonLLR > EMS_L_c2v[sumNonele])
                {
                    EMS_L_c2v[sumNonele] = sumNonLLR;
                }
                int sumNonele_all_max = sumNonele;
                float sumNonLLR_all_max = sumNonLLR;
                for (int i = 0; i < Checknode_weight[offset]; i++)
                {
                    if (i == dc)
                    {
                        continue;
                    }

                    sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele_all_max, TableAdd_GPU);
                    sumNonLLR = sumNonLLR_all_max - sort_L_v2c[Checknode_linkVNs[offset * maxdc + i]];

                    for (int k = 1; k < GFQ; k++)
                    {

                        int sumNonele1 = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i] + k], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                        float sumNonLLR1 = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i] + k];

                        if (sumNonLLR1 > EMS_L_c2v[sumNonele1])
                        {
                            EMS_L_c2v[sumNonele1] = sumNonLLR1;
                        }
                    }
                }

                // conf(nm, nc)
                // sumNonele = 0;
                // sumNonLLR = 0;
                // diff = 0;
                // ConstructConf_GPU((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, EMS_Nm, EMS_Nc, sumNonele, sumNonLLR, diff, 0, dc, Checknode_weight[offset] - 1, offset, EMS_L_c2v, (const int *)Variblenode_linkCNs, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);

                int *conf_index = (int *)malloc((Checknode_weight[offset] - 1) * sizeof(int));
                memset(conf_index, 0, (Checknode_weight[offset] - 1) * sizeof(int));

                int flag = 0;

                while (!flag)
                {
                    sumNonele = 0;
                    sumNonLLR = 0;
                    for (int i = 0; i < Checknode_weight[offset] - 1; i++)
                    {
                        conf_index[i] += 1; // move confset[i] to smaller one

                        if (i == Checknode_weight[offset] - 2 && conf_index[i] == EMS_Nm)
                        { // reaches end
                            flag = 1;
                            break;
                        }
                        else if (conf_index[i] >= EMS_Nm)
                        {
                            conf_index[i] = 0;
                            // continue to modify next VN
                        }
                        else
                        {
                            break; // don't modify next VN
                        }
                    }
                    if (!flag)
                    {
                        int k = 0;
                        for (int i = 0; i < Checknode_weight[offset]; i++)
                        {
                            if (i == dc)
                            {
                                continue;
                            }

                            sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i] + conf_index[k]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                            sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i] + conf_index[k]];
                            k++;
                        }
                        if (sumNonLLR > EMS_L_c2v[sumNonele])
                        {
                            EMS_L_c2v[sumNonele] = sumNonLLR;
                        }
                    }
                }
                free(conf_index);
                // calculate each c2v LLR
                int v = 0;
                Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + GFQ - 1] = 0;
                for (int k = 1; k < GFQ; k++)
                {
                    v = GFMultiply_GPU(k, Checknode_linkVNs_GF[offset * maxdc + dc], TableMultiply_GPU);
                    Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + k - 1] = (EMS_L_c2v[v] - EMS_L_c2v[0]) / 1.2;
                }
            }
            else
            {
                for (int k = 0; k < GFQ; k++)
                {
                    Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + k] = 0;
                }
            }
        }
    }
}
__device__ int ConstructConf_GPU(const unsigned *TableMultiply_GPU, const unsigned *TableAdd_GPU, int Nm, int Nc, int &sumNonele, float &sumNonLLR, int &diff, int begin, int except, int end, int row, float *EMS_L_c2v, const int *Variblenode_linkCNs, const int *Checknode_linkVNs, const int *Checknode_linkVNs_GF, int *sort_Entr_v2c, float *sort_L_v2c)
{
    // if (begin > end)
    // {
    //     if (sumNonLLR > EMS_L_c2v[sumNonele])
    //     {
    //         EMS_L_c2v[sumNonele] = sumNonLLR;
    //     }
    // }
    // else if (begin == except)
    // {
    //     ConstructConf_GPU(TableMultiply_GPU, TableAdd_GPU, Nm, Nc, sumNonele, sumNonLLR, diff, begin + 1, except, end, row, EMS_L_c2v, Variblenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);
    //     return 0;
    // }
    // else
    // {
    //     int index = index_in_VN_GPU(Checknode_linkVNs, row, begin, Variblenode_linkCNs);
    //     for (int k = 0; k < Nm; k++)
    //     {

    //         sumNonele = GFAdd_GPU(GFMultiply_GPU(23, 45, TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //         sumNonLLR = sumNonLLR + 0.3;

    //         // sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + begin], TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //         // sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k];
    //         diff += (k != 0) ? 1 : 0;
    //         if (diff <= Nc)
    //         {
    //             ConstructConf_GPU(TableMultiply_GPU, TableAdd_GPU, Nm, Nc, sumNonele, sumNonLLR, diff, begin + 1, except, end, row, EMS_L_c2v, Variblenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);
    //             // sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + begin], TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //             // sumNonLLR = sumNonLLR - sort_L_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k];

    //             sumNonele = GFAdd_GPU(GFMultiply_GPU(21, 25, TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //             sumNonLLR = sumNonLLR - 0.3;

    //             diff -= (k != 0) ? 1 : 0;
    //         }
    //         else
    //         {
    //             // sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + begin], TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //             // sumNonLLR = sumNonLLR - sort_L_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k];

    //             sumNonele = GFAdd_GPU(GFMultiply_GPU(34, 42, TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //             sumNonLLR = sumNonLLR - 0.3;

    //             diff -= (k != 0) ? 1 : 0;
    //             break;
    //         }
    //     }
    // }
    // return 0;
}