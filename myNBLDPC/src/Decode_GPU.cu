#include "hip/hip_runtime.h"
#include "Decode_GPU.cuh"
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <assert.h>

__device__ void BubleSort_GPU(float a[], int n, int index[])
{
    int i, j;
    float x;
    for (i = 0; i < n; i++)
    {
        for (j = 1; j < n - i; j++)
        {
            if (a[j - 1] < a[j])
            {
                x = a[j];
                a[j] = a[j - 1];
                a[j - 1] = x;
                x = index[j];
                index[j] = index[j - 1];
                index[j - 1] = x;
            }
        }
    }
}

__device__ int SortLLRVector_GPU(int GF, float *Entr_v2c, int *index)
{
    BubleSort_GPU(Entr_v2c, GF, index);
    return 1;
}

__device__ int DecideLLRVector_GPU(float *LLR, int GF)
{
    float max = 0;
    int alpha_i;
    for (int q = 0; q < GF - 1; q++)
    {
        if (LLR[q] > max)
        {
            max = LLR[q];
            alpha_i = q + 1;
        }
    }
    if (max <= 0)
    {
        return 0;
    }
    else
    {
        return alpha_i;
    }
}

__device__ int GetCombCount(int n, int m)
{
    long int i;
    long int a, b, c, s; // s = a/(b*c)
    a = b = c = 1;
    for (i = 1; i <= n; i++)
        a *= i;
    for (i = 1; i <= m; i++)
        b *= i;
    for (i = 1; i <= n - m; i++)
        c *= i;
    s = a / (b * c);
    return s;
}
__device__ void swap(int &a, int &b)
{
    int temp = a;
    a = b;
    b = temp;
}
__device__ int GFAdd_GPU(int ele1, int ele2, const unsigned *TableAdd_GPU)
{
    return ele1 ^ ele2;
}

__device__ int GFMultiply_GPU(int ele1, int ele2, const unsigned *TableMultiply_GPU)
{
    return TableMultiply_GPU[GFQ * ele1 + ele2];
}

__device__ int GFInverse_GPU(int ele, const unsigned *TableInverse_GPU)
{
    if (ele == 0)
    {
        printf("Div 0 Error!\n");
    }
    return TableInverse_GPU[ele];
}

int Decoding_EMS_GPU(const LDPCCode *H, VN *Variablenode, CN *Checknode, int EMS_Nm, int EMS_Nc, int *DecodeOutput, const unsigned *TableMultiply_GPU, const unsigned *TableAdd_GPU, const int *Variablenode_weight, const int *Checknode_weight, const int *Variablenode_linkCNs, const int *Checknode_linkVNs, const int *Checknode_linkVNs_GF, int &iter_number)
{
    hipError_t cudaStatus;
    // int *sort_Entr_v2c_temp = (int *)malloc(H->Variablenode_num * maxdv * GFQ * sizeof(int));
    // memset(sort_Entr_v2c_temp, 0, H->Variablenode_num * maxdv * GFQ * sizeof(int));
    int *sort_Entr_v2c;
    hipMalloc((void **)&sort_Entr_v2c, H->Variablenode_num * maxdv * GFQ * sizeof(int));

    float *sort_L_v2c_temp = (float *)malloc(H->Variablenode_num * maxdv * GFQ * sizeof(float));
    memset(sort_L_v2c_temp, 0, H->Variablenode_num * maxdv * GFQ * sizeof(float));
    float *sort_L_v2c;
    hipMalloc((void **)&sort_L_v2c, H->Variablenode_num * maxdv * GFQ * sizeof(float));

    float *Checknode_L_c2v_temp = (float *)malloc(H->Checknode_num * maxdc * GFQ * sizeof(float));
    memset(Checknode_L_c2v_temp, 0, H->Checknode_num * maxdc * GFQ * sizeof(float));

    float *Checknode_L_c2v;
    hipMalloc((void **)&Checknode_L_c2v, H->Checknode_num * maxdc * GFQ * sizeof(float));

    // int *index = (int *)malloc((GFQ) * sizeof(int));

    float *L_ch_temp = (float *)malloc(H->Variablenode_num * (GFQ - 1) * sizeof(float));
    memset(L_ch_temp, 0, H->Variablenode_num * (GFQ - 1) * sizeof(float));

    float *L_ch;
    hipMalloc((void **)&L_ch, H->Variablenode_num * (GFQ - 1) * sizeof(float));

    float *LLR_temp = (float *)malloc(H->Variablenode_num * (GFQ - 1) * sizeof(float));
    float *LLR;
    hipMalloc((void **)&LLR, H->Variablenode_num * (GFQ - 1) * sizeof(float));

    for (int col = 0; col < H->Variablenode_num; col++)
    {
        for (int d = 0; d < Variablenode[col].weight; d++)
        {
            Variablenode[col].L_ch[GFQ - 1] = 0;
            for (int q = 0; q < GFQ; q++)
            {
                sort_L_v2c_temp[col * maxdv * GFQ + d * GFQ + q] = Variablenode[col].L_ch[q];
                Variablenode[col].sort_L_v2c[d][q] = Variablenode[col].L_ch[q];
            }
        }
        for (int q = 0; q < GFQ; q++)
        {
            L_ch_temp[col * (GFQ - 1) + q] = Variablenode[col].L_ch[q];
        }
    }
    cudaStatus = hipMemcpy(L_ch, L_ch_temp, H->Variablenode_num * (GFQ - 1) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy L_ch\n");
        exit(0);
    }

    cudaStatus = hipMemcpy(sort_L_v2c, sort_L_v2c_temp, H->Variablenode_num * maxdv * GFQ * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy sort_L_v2c\n");
        exit(0);
    }

    cudaStatus = hipMemcpy(Checknode_L_c2v, Checknode_L_c2v_temp, H->Checknode_num * maxdc * GFQ * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("Cannot copy Checknode_L_c2v\n");
        exit(0);
    }

    for (int row = 0; row < H->Checknode_num; row++)
    {
        for (int d = 0; d < Checknode[row].weight; d++)
        {
            memset(Checknode[row].L_c2v[d], 0, (GFQ - 1) * sizeof(float));
        }
    }

    iter_number = 0;
    bool decode_correct = true;
    int *DecodeOutput_GPU;
    hipMalloc((void **)&DecodeOutput_GPU, H->Variablenode_num * sizeof(int));

    while (iter_number < maxIT)
    {
        iter_number++;
        Variablenode_EMS<<<((H->Variablenode_num % 128) ? (H->Variablenode_num / 128 + 1) : (H->Variablenode_num / 128)), 128>>>((const int *)Variablenode_weight, (const int *)Variablenode_linkCNs, sort_Entr_v2c, sort_L_v2c, Checknode_L_c2v, (const float *)L_ch, LLR, DecodeOutput_GPU, H->Variablenode_num);

        cudaStatus = hipMemcpy(DecodeOutput, DecodeOutput_GPU, H->Variablenode_num * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            printf("Cannot copy DecodeOutput\n");
            exit(0);
        }
        // for (int i = 0; i < H->Variablenode_num; i++)
        // {
        //     for (int q = 0; q < GFQ - 1; q++)
        //     {
        //         Variablenode[i].LLR[q] = LLR_temp[i * (GFQ - 1) + q];
        //     }
        //     DecodeOutput[i] = DecideLLRVector(Variablenode[i].LLR, GFQ);
        // }

        decode_correct = true;
        int sum_temp = 0;
        for (int row = 0; row < H->Checknode_num; row++)
        {
            for (int i = 0; i < Checknode[row].weight; i++)
            {
                sum_temp = GFAdd(sum_temp, GFMultiply(DecodeOutput[Checknode[row].linkVNs[i]], Checknode[row].linkVNs_GF[i]));
            }
            if (sum_temp)
            {
                decode_correct = false;
                break;
            }
        }

        if (decode_correct)
        {

            hipFree(sort_Entr_v2c);
            hipFree(sort_L_v2c);
            hipFree(Checknode_L_c2v);
            hipFree(LLR);
            hipFree(L_ch);
            hipFree(DecodeOutput_GPU);
            free(L_ch_temp);
            free(LLR_temp);
            // free(index);
            // free(sort_Entr_v2c_temp);
            free(sort_L_v2c_temp);
            free(Checknode_L_c2v_temp);
            iter_number--;
            return 1;
        }
        // message from var to check
        // for (int col = 0; col < H->Variablenode_num; col++)
        // {
        //     for (int dv = 0; dv < Variablenode[col].weight; dv++)
        //     {
        //         for (int q = 0; q < GFQ - 1; q++)
        //         {
        //             Variablenode[col].sort_L_v2c[dv][q] = Variablenode[col].LLR[q] - Checknode[Variablenode[col].linkCNs[dv]].L_c2v[index_in_CN(Variablenode, col, dv, Checknode)][q];
        //         }
        //         Variablenode[col].sort_L_v2c[dv][GFQ - 1] = 0;
        //     }
        // }

        // for (int col = 0; col < H->Variablenode_num; col++)
        // {
        //     for (int dv = 0; dv < Variablenode[col].weight; dv++)
        //     {
        //         for (int i = 0; i < GFQ - 1; i++)
        //         {
        //             index[i] = i + 1;
        //         }
        //         index[GFQ - 1] = 0;
        //         SortLLRVector(GFQ, Variablenode[col].sort_L_v2c[dv], index);
        //         for (int i = 0; i < GFQ; i++)
        //         {
        //             Variablenode[col].sort_Entr_v2c[dv][i] = index[i];

        //             sort_Entr_v2c_temp[col * maxdv * GFQ + dv * GFQ + i] = index[i];
        //             sort_L_v2c_temp[col * maxdv * GFQ + dv * GFQ + i] = Variablenode[col].sort_L_v2c[dv][i];
        //         }
        //     }
        // }

        Variablenode_Update<<<((H->Variablenode_num % 128) ? (H->Variablenode_num / 128 + 1) : (H->Variablenode_num / 128)), 128>>>((const int *)Variablenode_weight, (const int *)Variablenode_linkCNs, sort_Entr_v2c, sort_L_v2c, Checknode_L_c2v, (const float *)L_ch, LLR, H->Variablenode_num);

        // cudaStatus = hipMemcpy(sort_Entr_v2c, sort_Entr_v2c_temp, H->Variablenode_num * maxdv * GFQ * sizeof(int), hipMemcpyHostToDevice);
        // if (cudaStatus != hipSuccess)
        // {
        //     printf("Cannot copy sort_Entr_v2c\n");
        //     exit(0);
        // }
        // cudaStatus = hipMemcpy(sort_L_v2c, sort_L_v2c_temp, H->Variablenode_num * maxdv * GFQ * sizeof(float), hipMemcpyHostToDevice);
        // if (cudaStatus != hipSuccess)
        // {
        //     printf("Cannot copy sort_L_v2c\n");
        //     exit(0);
        // }
        // // message from check to var

        Checknode_EMS<<<((H->Checknode_num % 128) ? (H->Checknode_num / 128 + 1) : (H->Checknode_num / 128)), 128>>>((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, EMS_Nm, EMS_Nc, (const int *)Checknode_weight, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c, Checknode_L_c2v, H->Checknode_num);
        // Checknode_EMS<<<1, 1>>>((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, EMS_Nm, EMS_Nc, (const int *)Checknode_weight, (const int *)Variablenode_linkCNs, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c, Checknode_L_c2v, H->Checknode_num);

        // cudaStatus = hipMemcpy(Checknode_L_c2v_temp, Checknode_L_c2v, H->Checknode_num * maxdc * GFQ * sizeof(float), hipMemcpyDeviceToHost);
        // if (cudaStatus != hipSuccess)
        // {
        //     printf("Cannot copy Checknode_L_c2v D2V\n");
        //     exit(0);
        // }

        // for (int i = 0; i < H->Checknode_num; i++)
        // {
        //     for (int j = 0; j < Checknode[i].weight; j++)
        //     {
        //         for (int q = 0; q < GFQ - 1; q++)
        //         {
        //             Checknode[i].L_c2v[j][q] = Checknode_L_c2v_temp[i * maxdc * GFQ + j * GFQ + q];
        //         }
        //     }
        // }
    }

    hipFree(sort_Entr_v2c);
    hipFree(sort_L_v2c);
    hipFree(Checknode_L_c2v);
    hipFree(LLR);
    hipFree(L_ch);
    free(L_ch_temp);
    free(LLR_temp);
    hipFree(DecodeOutput_GPU);
    // free(index);
    // free(sort_Entr_v2c_temp);
    free(sort_L_v2c_temp);
    free(Checknode_L_c2v_temp);
    return 0;
}

__global__ void Variablenode_EMS(const int *Variablenode_weight, const int *Variablenode_linkCNs, int *sort_Entr_v2c, float *sort_L_v2c, float *Checknode_L_c2v, const float *L_ch, float *LLR, int *DecodeOutput, int Variablenode_num)
{

    int offset;
    offset = threadIdx.x + blockDim.x * blockIdx.x;
    if (offset < Variablenode_num)
    {
        for (int q = 0; q < GFQ - 1; q++)
        {
            LLR[offset * (GFQ - 1) + q] = L_ch[offset * (GFQ - 1) + q];
        }
        for (int d = 0; d < Variablenode_weight[offset]; d++)
        {
            for (int q = 0; q < GFQ - 1; q++)
            {
                LLR[offset * (GFQ - 1) + q] += Checknode_L_c2v[Variablenode_linkCNs[offset * maxdv + d] + q];
            }
        }
        DecodeOutput[offset] = DecideLLRVector_GPU(LLR + offset * (GFQ - 1), GFQ);
    }
}

__global__ void Variablenode_Update(const int *Variablenode_weight, const int *Variablenode_linkCNs, int *sort_Entr_v2c, float *sort_L_v2c, float *Checknode_L_c2v, const float *L_ch, float *LLR, int Variablenode_num)
{

    int offset;
    offset = threadIdx.x + blockDim.x * blockIdx.x;
    int *index = (int *)malloc(GFQ * sizeof(int));
    if (offset < Variablenode_num)
    {

        for (int dv = 0; dv < Variablenode_weight[offset]; dv++)
        {
            for (int q = 0; q < GFQ - 1; q++)
            {
                sort_L_v2c[offset * maxdv * GFQ + dv * GFQ + q] = LLR[offset * (GFQ - 1) + q] - Checknode_L_c2v[Variablenode_linkCNs[offset * maxdv + dv] + q];
            }
            sort_L_v2c[offset * maxdv * GFQ + dv * GFQ + GFQ - 1] = 0;
        }
        for (int dv = 0; dv < Variablenode_weight[offset]; dv++)
        {
            for (int i = 0; i < GFQ - 1; i++)
            {
                index[i] = i + 1;
            }
            index[GFQ - 1] = 0;
            SortLLRVector_GPU(GFQ, sort_L_v2c + offset * maxdv * GFQ + dv * GFQ, index);
            for (int i = 0; i < GFQ; i++)
            {
                sort_Entr_v2c[offset * maxdv * GFQ + dv * GFQ + i] = index[i];
            }
        }
    }
    free(index);
}

/*
Checknode_weight:每一个校验节点的重量
L_c2v:Q个信息，Q个信息，Q个信息，一共校验节点数量*Q个
Variblenode_linkCNs:最大重量dv，每dv个元素代表连接的dv个校验节点的序号
Checknode_linkVNS:最大重量dc，每dc个元素代表连接的dc个变量节点的序号
Checknode_linkVNS_GF:最大重量dc，每dc个元素代表连接的dc个变量节点的多元域值
sort_Entr_v2c:每个变量节点重量dv，q,q,q一共dv个，然后再乘以变量节点个数[变量节点个数][变量节点重量][q]
sort_L_v2c:和sort_Entr_v2c对应的LLR
Checknode_L_c2v:每个校验节点重量dc，q一共dc个，然后再乘以变量节点个数[校验节点个数][校验节点重量][q]
*/
__global__ void Checknode_EMS(const unsigned *TableMultiply_GPU, const unsigned *TableAdd_GPU, int EMS_Nm, int EMS_Nc, const int *Checknode_weight, const int *Checknode_linkVNs, const int *Checknode_linkVNs_GF, int *sort_Entr_v2c, float *sort_L_v2c, float *Checknode_L_c2v, int Checknode_num)
{
    int offset;
    offset = threadIdx.x + blockDim.x * blockIdx.x;
    if (offset < Checknode_num)
    {
        float EMS_L_c2v[GFQ];
        for (int dc = 0; dc < maxdc; dc++)
        {
            if (dc < Checknode_weight[offset])
            {
                // reset the sum store vector to the munimum
                for (int q = 0; q < GFQ; q++)
                {
                    EMS_L_c2v[q] = -DBL_MAX;
                }

                // recursly exhaustly
                int sumNonele;
                float sumNonLLR;
                // conf(q, 1)
                sumNonele = 0;
                sumNonLLR = 0;
                // ConstructConf_GPU((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, GFQ, 1, sumNonele, sumNonLLR, diff, 0, dc, Checknode_weight[offset] - 1, offset, EMS_L_c2v, (const int *)Variblenode_linkCNs, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);

                for (int i = 0; i < Checknode_weight[offset]; i++)
                {
                    if (i == dc)
                    {
                        continue;
                    }

                    sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                    sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i]];
                }
                if (sumNonLLR > EMS_L_c2v[sumNonele])
                {
                    EMS_L_c2v[sumNonele] = sumNonLLR;
                }
                int sumNonele_all_max = sumNonele;
                float sumNonLLR_all_max = sumNonLLR;
                for (int i = 0; i < Checknode_weight[offset]; i++)
                {
                    if (i == dc)
                    {
                        continue;
                    }

                    sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele_all_max, TableAdd_GPU);
                    sumNonLLR = sumNonLLR_all_max - sort_L_v2c[Checknode_linkVNs[offset * maxdc + i]];

                    for (int k = 1; k < GFQ; k++)
                    {

                        int sumNonele1 = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i] + k], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                        float sumNonLLR1 = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i] + k];

                        if (sumNonLLR1 > EMS_L_c2v[sumNonele1])
                        {
                            EMS_L_c2v[sumNonele1] = sumNonLLR1;
                        }
                    }
                }

                // conf(nm, nc)
                // sumNonele = 0;
                // sumNonLLR = 0;
                // diff = 0;
                // ConstructConf_GPU((const unsigned *)TableMultiply_GPU, (const unsigned *)TableAdd_GPU, EMS_Nm, EMS_Nc, sumNonele, sumNonLLR, diff, 0, dc, Checknode_weight[offset] - 1, offset, EMS_L_c2v, (const int *)Variblenode_linkCNs, (const int *)Checknode_linkVNs, (const int *)Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);
                int *bit = new int[Checknode_weight[offset] - 1];
                int EMS_Nc_temp;
                if (EMS_Nc == maxdc - 1)
                {
                    EMS_Nc_temp = Checknode_weight[offset] - 1;
                }
                else
                {
                    EMS_Nc_temp = EMS_Nc;
                }
                for (int choose_n = 2; choose_n <= EMS_Nc_temp; choose_n++)
                {

                    for (int k = 0; k < Checknode_weight[offset] - 1; k++)
                    {
                        if (k < choose_n)
                            bit[k] = 1;
                        else
                            bit[k] = 0;
                    }

                    int i, j, beg, end;
                    int len = Checknode_weight[offset] - 1;
                    int N = GetCombCount(Checknode_weight[offset] - 1, choose_n); //C(n,count)  C(5,3)

                    int *conf_index = (int *)malloc(choose_n * sizeof(int));
                    memset(conf_index, 0, (choose_n) * sizeof(int));

                    int flag = 0;

                    while (!flag)
                    {
                        sumNonele = 0;
                        sumNonLLR = 0;
                        for (int i = 0; i < choose_n; i++)
                        {
                            conf_index[i] += 1; // move confset[i] to smaller one

                            if (i == choose_n - 1 && conf_index[i] == EMS_Nm)
                            { // reaches end
                                flag = 1;
                                break;
                            }
                            else if (conf_index[i] >= EMS_Nm)
                            {
                                conf_index[i] = 0;
                                // continue to modify next VN
                            }
                            else
                            {
                                break; // don't modify next VN
                            }
                        }
                        if (!flag)
                        {
                            int k = 0;
                            int t = 0;
                            for (int i = 0; i < Checknode_weight[offset]; i++)
                            {
                                if (i == dc)
                                {
                                    continue;
                                }
                                if (bit[t] == 1)
                                {
                                    sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i] + conf_index[k]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                                    sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i] + conf_index[k]];
                                    k++;
                                }
                                else
                                {
                                    sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                                    sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i]];
                                }
                                t++;
                            }
                            if (sumNonLLR > EMS_L_c2v[sumNonele])
                            {
                                EMS_L_c2v[sumNonele] = sumNonLLR;
                            }
                        }
                    }
                    for (j = 1; j < N; j++)
                    {
                        for (i = len - 1; i > 0; i--)
                        {
                            if (bit[i] == 0 && bit[i - 1] == 1)
                            {
                                swap(bit[i], bit[i - 1]);

                                //from index: [i to len-1] , make all bit 1 in the right
                                beg = i;
                                end = len - 1;
                                while (1)
                                {
                                    while (bit[beg] == 1)
                                    {
                                        beg++;
                                        if (beg >= len)
                                            break;
                                    }
                                    while (bit[end] == 0)
                                    {
                                        end--;
                                        if (end < i)
                                            break;
                                    }

                                    if (beg < end)
                                        swap(bit[beg], bit[end]);
                                    else
                                        break;

                                } //end of "while"
                                break;
                            } //end of "if"
                        }
                        flag = 0;
                        memset(conf_index, 0, (choose_n) * sizeof(int));

                        while (!flag)
                        {
                            sumNonele = 0;
                            sumNonLLR = 0;
                            for (int i = 0; i < choose_n; i++)
                            {
                                conf_index[i] += 1; // move confset[i] to smaller one

                                if (i == choose_n - 1 && conf_index[i] == EMS_Nm)
                                { // reaches end
                                    flag = 1;
                                    break;
                                }
                                else if (conf_index[i] >= EMS_Nm)
                                {
                                    conf_index[i] = 0;
                                    // continue to modify next VN
                                }
                                else
                                {
                                    break; // don't modify next VN
                                }
                            }
                            if (!flag)
                            {
                                int k = 0;
                                int t = 0;
                                for (int i = 0; i < Checknode_weight[offset]; i++)
                                {
                                    if (i == dc)
                                    {
                                        continue;
                                    }
                                    if (bit[t] == 1)
                                    {
                                        sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i] + conf_index[k]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                                        sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i] + conf_index[k]];
                                        k++;
                                    }
                                    else
                                    {
                                        sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[offset * maxdc + i]], Checknode_linkVNs_GF[offset * maxdc + i], TableMultiply_GPU), sumNonele, TableAdd_GPU);
                                        sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[offset * maxdc + i]];
                                    }
                                    t++;
                                }
                                if (sumNonLLR > EMS_L_c2v[sumNonele])
                                {
                                    EMS_L_c2v[sumNonele] = sumNonLLR;
                                }
                            }
                        }
                    }
                    free(conf_index);
                }

                free(bit);
                // calculate each c2v LLR
                int v = 0;
                Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + GFQ - 1] = 0;
                for (int k = 1; k < GFQ; k++)
                {
                    v = GFMultiply_GPU(k, Checknode_linkVNs_GF[offset * maxdc + dc], TableMultiply_GPU);
                    Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + k - 1] = (EMS_L_c2v[v] - EMS_L_c2v[0]) / 1.2;
                }
            }
            else
            {
                for (int k = 0; k < GFQ; k++)
                {
                    Checknode_L_c2v[offset * maxdc * GFQ + dc * GFQ + k] = 0;
                }
            }
        }
    }
}
__device__ int ConstructConf_GPU(const unsigned *TableMultiply_GPU, const unsigned *TableAdd_GPU, int Nm, int Nc, int &sumNonele, float &sumNonLLR, int &diff, int begin, int except, int end, int row, float *EMS_L_c2v, const int *Variblenode_linkCNs, const int *Checknode_linkVNs, const int *Checknode_linkVNs_GF, int *sort_Entr_v2c, float *sort_L_v2c)
{
    // if (begin > end)
    // {
    //     if (sumNonLLR > EMS_L_c2v[sumNonele])
    //     {
    //         EMS_L_c2v[sumNonele] = sumNonLLR;
    //     }
    // }
    // else if (begin == except)
    // {
    //     ConstructConf_GPU(TableMultiply_GPU, TableAdd_GPU, Nm, Nc, sumNonele, sumNonLLR, diff, begin + 1, except, end, row, EMS_L_c2v, Variblenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);
    //     return 0;
    // }
    // else
    // {
    //     int index = index_in_VN_GPU(Checknode_linkVNs, row, begin, Variblenode_linkCNs);
    //     for (int k = 0; k < Nm; k++)
    //     {

    //         sumNonele = GFAdd_GPU(GFMultiply_GPU(23, 45, TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //         sumNonLLR = sumNonLLR + 0.3;

    //         // sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + begin], TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //         // sumNonLLR = sumNonLLR + sort_L_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k];
    //         diff += (k != 0) ? 1 : 0;
    //         if (diff <= Nc)
    //         {
    //             ConstructConf_GPU(TableMultiply_GPU, TableAdd_GPU, Nm, Nc, sumNonele, sumNonLLR, diff, begin + 1, except, end, row, EMS_L_c2v, Variblenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF, sort_Entr_v2c, sort_L_v2c);
    //             // sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + begin], TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //             // sumNonLLR = sumNonLLR - sort_L_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k];

    //             sumNonele = GFAdd_GPU(GFMultiply_GPU(21, 25, TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //             sumNonLLR = sumNonLLR - 0.3;

    //             diff -= (k != 0) ? 1 : 0;
    //         }
    //         else
    //         {
    //             // sumNonele = GFAdd_GPU(GFMultiply_GPU(sort_Entr_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k], Checknode_linkVNs_GF[row * maxdc + begin], TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //             // sumNonLLR = sumNonLLR - sort_L_v2c[Checknode_linkVNs[row * maxdc + begin] * maxdv * GFQ + index * GFQ + k];

    //             sumNonele = GFAdd_GPU(GFMultiply_GPU(34, 42, TableMultiply_GPU), sumNonele, TableAdd_GPU);
    //             sumNonLLR = sumNonLLR - 0.3;

    //             diff -= (k != 0) ? 1 : 0;
    //             break;
    //         }
    //     }
    // }
    // return 0;
}