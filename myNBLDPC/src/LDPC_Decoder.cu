#include <stdlib.h>
#include <stdio.h>
#include <math.h>
//#include <conio.h>
#include <string.h>
#include <memory.h>
#include <time.h>
//#include <direct.h>
#include "define.cuh"
#include "struct.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.cuh"
#include "GF.cuh"

int index_in_VN(CN* Checknode,int CNnum,int index_in_linkVNS,VN* Variablenode)
{
    for(int i=0;i<Variablenode[Checknode[CNnum].linkVNs[index_in_linkVNS]].weight;i++)
    {
        if(Variablenode[Checknode[CNnum].linkVNs[index_in_linkVNS]].linkCNs[i]==CNnum)
        {
            return i;
        }
    }
    printf("index_in_VN error\n");
    exit(0);
} 

int index_in_CN(VN* Variablenode,int VNnum,int index_in_linkCNS,CN* Checknode)
{
    for(int i=0;i<Checknode[Variablenode[VNnum].linkCNs[index_in_linkCNS]].weight;i++)
    {
        if(Checknode[Variablenode[VNnum].linkCNs[index_in_linkCNS]].linkVNs[i]==VNnum)
        {
            return i;
        }
    }
    printf("index_in_CN error\n");
    exit(0);
} 


void Demodulate(LDPCCode* H,AWGNChannel* AWGN,CComplex* CONSTELLATION,VN* Variablenode,CComplex* CComplex_sym_Channelout)
{
    int p_i = 0;
    for(int s = 0; s < H->Variablenode_num; s ++)
    {
            for(int q = 1; q < H->GF; q ++)
            {
                Variablenode[s].LLR[q - 1] = ( (2 * CComplex_sym_Channelout[s - p_i].Real - CONSTELLATION[0].Real - CONSTELLATION[q].Real ) * (CONSTELLATION[q].Real - CONSTELLATION[0].Real) 
                    + (2 * CComplex_sym_Channelout[s - p_i].Image - CONSTELLATION[0].Image - CONSTELLATION[q].Image ) * (CONSTELLATION[q].Image - CONSTELLATION[0].Image) ) / (2 * AWGN->sigma * AWGN->sigma);
            }
    }
}

int ConstructConf(CN *Checknode,VN *Variablenode,int Nm, int Nc, int& sumNonele, double& sumNonLLR, int& diff, int begin, int except, int end, int row)
{
    int index=index_in_VN(Checknode,row,except,Variablenode);
	if (begin > end)
	{
		if (sumNonLLR > Checknode[row].L_c2v[except][sumNonele])
		{
			Checknode[row].L_c2v[except][sumNonele] = sumNonLLR;
		}
	}
	else if (begin == except)
	{
		ConstructConf(Checknode, Variablenode, Nm, Nc, sumNonele, sumNonLLR, diff, begin + 1, except, end, row);
		return 0;
	}
	else
	{
		for (int k = 0; k < Nm; k++)
		{
			sumNonele = GFAdd(GFMultiply(Variablenode[Checknode[row].linkVNs[begin]].sort_Entr_v2c[index][begin], Checknode->linkVNs_GF[begin]), sumNonele);
			sumNonLLR = sumNonLLR + Variablenode[Checknode[row].linkVNs[begin]].sort_Entr_v2c[index][begin];
			diff += (k != 0) ? 1 : 0;
			if (diff <= Nc)
			{
				ConstructConf(Checknode, Variablenode, Nm, Nc, sumNonele, sumNonLLR, diff, begin + 1, except, end, row);
				sumNonele = GFAdd(GFMultiply(Variablenode[Checknode[row].linkVNs[begin]].sort_Entr_v2c[index][begin], Checknode->linkVNs_GF[begin]), sumNonele);
				sumNonLLR = sumNonLLR - Variablenode[Checknode[row].linkVNs[begin]].sort_Entr_v2c[index][begin];
				diff -= (k != 0) ? 1 : 0;
			}
			else
			{
				sumNonele = GFAdd(GFMultiply(Variablenode[Checknode[row].linkVNs[begin]].sort_Entr_v2c[index][begin], Checknode->linkVNs_GF[begin]), sumNonele);
				sumNonLLR = sumNonLLR - Variablenode[Checknode[row].linkVNs[begin]].sort_Entr_v2c[index][begin];
				diff -= (k != 0) ? 1 : 0;
				break;
			}
		}
	}
	return 0;
}
