#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
//#include <conio.h>
#include <string.h>
#include <memory.h>
#include <time.h>
//#include <direct.h>
#include "define.cuh"
#include "struct.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.cuh"
#include "GF.cuh"
#include "float.h"

void BubleSort(float a[], int n, int index[])
{
	int i, j;
	float x;
	for (i = 0; i < n; i++)
	{
		for (j = 1; j < n - i; j++)
		{
			if (a[j - 1] < a[j])
			{
				x = a[j];
				a[j] = a[j - 1];
				a[j - 1] = x;
				x = index[j];
				index[j] = index[j - 1];
				index[j - 1] = x;
			}
		}
	}
}

int SortLLRVector(int GF, float *Entr_v2c, int *index)
{
	BubleSort(Entr_v2c, GF, index);
	return 1;
}

int DecideLLRVector(float *LLR, int GF)
{
	float max = 0;
	int alpha_i;
	for (int q = 0; q < GF - 1; q++)
	{
		if (LLR[q] > max)
		{
			max = LLR[q];
			alpha_i = q + 1;
		}
	}
	if (max <= 0)
	{
		return 0;
	}
	else
	{
		return alpha_i;
	}
}

int index_in_VN(CN *Checknode, int CNnum, int index_in_linkVNS, VN *Variablenode)
{
	for (int i = 0; i < Variablenode[Checknode[CNnum].linkVNs[index_in_linkVNS]].weight; i++)
	{
		if (Variablenode[Checknode[CNnum].linkVNs[index_in_linkVNS]].linkCNs[i] == CNnum)
		{
			return i;
		}
	}
	printf("index_in_VN error\n");
	exit(0);
}

int index_in_CN(VN *Variablenode, int VNnum, int index_in_linkCNS, CN *Checknode)
{
	for (int i = 0; i < Checknode[Variablenode[VNnum].linkCNs[index_in_linkCNS]].weight; i++)
	{
		if (Checknode[Variablenode[VNnum].linkCNs[index_in_linkCNS]].linkVNs[i] == VNnum)
		{
			return i;
		}
	}
	printf("index_in_CN error\n");
	exit(0);
}

void Demodulate(LDPCCode *H, AWGNChannel *AWGN, CComplex *CONSTELLATION, VN *Variablenode, CComplex *CComplex_sym_Channelout)
{
	float *RX_LLR_BIT = (float *)malloc(H->bit_length * sizeof(float));
	if (n_QAM == 2)
	{
		// RX_MOD_SYM --> RX_LLR_BIT --> RX_LLR_SYM
		// only support bpsk now
		int p_i = 0;
		for (int b = 0; b < H->bit_length; b++)
		{

			RX_LLR_BIT[b] = -2 * CComplex_sym_Channelout[b - p_i].Real / (AWGN->sigma * AWGN->sigma);
		}
		// RX_LLE_BIT --> RX_LLR_SYM
		for (int s = 0; s < H->Variablenode_num; s++)
		{
			for (int q = 1; q < H->GF; q++)
			{
				Variablenode[s].L_ch[q - 1] = 0;
				for (int b_p_s = 0; b_p_s < H->q_bit; b_p_s++)
				{
					if ((q & (1 << b_p_s)) != 0)
					{
						Variablenode[s].L_ch[q - 1] += RX_LLR_BIT[s * H->q_bit + b_p_s];
					}
				}
			}
		}
	}
	else
	{
		for (int s = 0; s < H->Variablenode_num; s++)
		{
			for (int q = 1; q < H->GF; q++)
			{
				Variablenode[s].L_ch[q - 1] = ((2 * CComplex_sym_Channelout[s].Real - CONSTELLATION[0].Real - CONSTELLATION[q].Real) * (CONSTELLATION[q].Real - CONSTELLATION[0].Real) + (2 * CComplex_sym_Channelout[s].Image - CONSTELLATION[0].Image - CONSTELLATION[q].Image) * (CONSTELLATION[q].Image - CONSTELLATION[0].Image)) / (2 * AWGN->sigma * AWGN->sigma);
			}
		}
	}
}
int Decoding_EMS(LDPCCode *H, VN *Variablenode, CN *Checknode, int EMS_Nm, int EMS_Nc, int *DecodeOutput)
{
	for (int col = 0; col < H->Variablenode_num; col++)
	{
		for (int d = 0; d < Variablenode[col].weight; d++)
		{
			for (int q = 0; q < H->GF; q++)
			{
				Variablenode[col].Entr_v2c[d][q] = Variablenode[col].L_ch[q];
			}
		}
	}
	for (int row = 0; row < H->Checknode_num; row++)
	{
		for (int d = 0; d < Checknode[row].weight; d++)
		{
			for (int q = 0; q < H->GF - 1; q++)
			{
				Checknode[row].L_c2v[d][q] = 0;
			}
		}
	}

	int iter_number = 0;
	bool decode_correct = true;
	while (iter_number++ < maxIT)
	{
		// printf("it_time: %d\n",iter_number);
		for (int col = 0; col < H->Variablenode_num; col++)
		{
			for (int d = 0; d < Variablenode[col].weight; d++)
			{
				for (int q = 0; q < H->GF - 1; q++)
				{
					Variablenode[col].LLR[q] = Variablenode[col].L_ch[q];
				}
			}
		}
		for (int col = 0; col < H->Variablenode_num; col++)
		{
			for (int d = 0; d < Variablenode[col].weight; d++)
			{
				for (int q = 0; q < H->GF - 1; q++)
				{
					Variablenode[col].LLR[q] += Checknode[Variablenode[col].linkCNs[d]].L_c2v[index_in_CN(Variablenode, col, d, Checknode)][q];
				}
			}
			DecodeOutput[col] = DecideLLRVector(Variablenode[col].LLR, H->GF);
			// printf("%d ",DecodeOutput[col]);
		}
		// printf("\n");
		// exit(0);

		decode_correct = true;
		int sum_temp = 0;
		for (int row = 0; row < H->Checknode_num; row++)
		{
			for (int i = 0; i < Checknode[row].weight; i++)
			{
				sum_temp = GFAdd(sum_temp, GFMultiply(DecodeOutput[Checknode[row].linkVNs[i]], Checknode[row].linkVNs_GF[i]));
			}
			if (sum_temp)
			{
				decode_correct = false;
				break;
			}
		}
		if (decode_correct)
		{
			return 1;
		}

		// message from var to check
		for (int col = 0; col < H->Variablenode_num; col++)
		{
			for (int dv = 0; dv < Variablenode[col].weight; dv++)
			{
				for (int q = 0; q < H->GF - 1; q++)
				{
					Variablenode[col].Entr_v2c[dv][q] = Variablenode[col].LLR[q] - Checknode[Variablenode[col].linkCNs[dv]].L_c2v[index_in_CN(Variablenode, col, dv, Checknode)][q];
				}
			}
		}

		int *index = (int *)malloc((H->GF) * sizeof(int));
		for (int col = 0; col < H->Variablenode_num; col++)
		{
			memcpy(Variablenode[col].sort_L_v2c[0], Variablenode[col].Entr_v2c[0], Variablenode[col].weight * H->GF * sizeof(float));

			for (int dv = 0; dv < Variablenode[col].weight; dv++)
			{
				for (int i = 0; i < H->GF - 1; i++)
				{
					index[i] = i + 1;
				}
				index[H->GF - 1] = 0;
				SortLLRVector(H->GF, Variablenode[col].sort_L_v2c[dv], index);
				for (int i = 0; i < H->GF; i++)
				{
					Variablenode[col].sort_Entr_v2c[dv][i] = index[i];
				}
			}
		}

		float *EMS_L_c2v = (float *)malloc(H->GF * sizeof(float));

		// message from check to var
		for (int row = 0; row < H->Checknode_num; row++)
		{

			for (int dc = 0; dc < Checknode[row].weight; dc++)
			{
				// reset the sum store vector to the munimum
				for (int q = 0; q < H->GF; q++)
				{
					EMS_L_c2v[q] = -DBL_MAX;
				}

				// recursly exhaustly
				int sumNonele, diff;
				float sumNonLLR;
				// conf(q, 1)
				sumNonele = 0;
				sumNonLLR = 0;
				diff = 0;
				ConstructConf(Checknode, Variablenode, H->GF, 1, sumNonele, sumNonLLR, diff, 0, dc, Checknode[row].weight - 1, row, EMS_L_c2v);

				// conf(nm, nc)
				sumNonele = 0;
				sumNonLLR = 0;
				diff = 0;
				ConstructConf(Checknode, Variablenode, EMS_Nm, EMS_Nc, sumNonele, sumNonLLR, diff, 0, dc, Checknode[row].weight - 1, row, EMS_L_c2v);

				// calculate each c2v LLR
				int v = 0;
				for (int k = 1; k < H->GF; k++)
				{
					v = GFMultiply(k, Checknode[row].linkVNs_GF[dc]);
					Checknode[row].L_c2v[dc][k - 1] = (EMS_L_c2v[v] - EMS_L_c2v[0]) / 1.2;
				}
			}
		}
		free(EMS_L_c2v);
	}
	return 0;
}

int ConstructConf(CN *Checknode, VN *Variablenode, int Nm, int Nc, int &sumNonele, float &sumNonLLR, int &diff, int begin, int except, int end, int row, float *EMS_L_c2v)
{
	int index;
	if (begin > end)
	{
		if (sumNonLLR > EMS_L_c2v[sumNonele])
		{
			EMS_L_c2v[sumNonele] = sumNonLLR;
		}
	}
	else if (begin == except)
	{
		ConstructConf(Checknode, Variablenode, Nm, Nc, sumNonele, sumNonLLR, diff, begin + 1, except, end, row, EMS_L_c2v);
		return 0;
	}
	else
	{
		index = index_in_VN(Checknode, row, begin, Variablenode);
		for (int k = 0; k < Nm; k++)
		{
			sumNonele = GFAdd(GFMultiply(Variablenode[Checknode[row].linkVNs[begin]].sort_Entr_v2c[index][k], Checknode[row].linkVNs_GF[begin]), sumNonele);
			sumNonLLR = sumNonLLR + Variablenode[Checknode[row].linkVNs[begin]].sort_L_v2c[index][k];
			diff += (k != 0) ? 1 : 0;
			if (diff <= Nc)
			{
				ConstructConf(Checknode, Variablenode, Nm, Nc, sumNonele, sumNonLLR, diff, begin + 1, except, end, row, EMS_L_c2v);
				sumNonele = GFAdd(GFMultiply(Variablenode[Checknode[row].linkVNs[begin]].sort_Entr_v2c[index][k], Checknode[row].linkVNs_GF[begin]), sumNonele);
				sumNonLLR = sumNonLLR - Variablenode[Checknode[row].linkVNs[begin]].sort_L_v2c[index][k];
				diff -= (k != 0) ? 1 : 0;
			}
			else
			{
				sumNonele = GFAdd(GFMultiply(Variablenode[Checknode[row].linkVNs[begin]].sort_Entr_v2c[index][k], Checknode[row].linkVNs_GF[begin]), sumNonele);
				sumNonLLR = sumNonLLR - Variablenode[Checknode[row].linkVNs[begin]].sort_L_v2c[index][k];
				diff -= (k != 0) ? 1 : 0;
				break;
			}
		}
	}
	return 0;
}
