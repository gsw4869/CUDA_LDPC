#include <stdlib.h>
#include <stdio.h>
#include <math.h>
//#include <conio.h>
#include <string.h>
#include <memory.h>
#include <time.h>
//#include <direct.h>
#include "define.cuh"
#include "struct.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.cuh"

void Demodulate(LDPCCode* H,AWGNChannel* AWGN,CComplex* CONSTELLATION,VN* Variablenode,CComplex* CComplex_sym_Channelout)
{
    int p_i = 0;
    for(int s = 0; s < H->Variablenode_num; s ++)
    {
            for(int q = 1; q < H->GF; q ++)
            {
                Variablenode[s].LLR[q - 1] = ( (2 * CComplex_sym_Channelout[s - p_i].Real - CONSTELLATION[0].Real - CONSTELLATION[q].Real ) * (CONSTELLATION[q].Real - CONSTELLATION[0].Real) 
                    + (2 * CComplex_sym_Channelout[s - p_i].Image - CONSTELLATION[0].Image - CONSTELLATION[q].Image ) * (CONSTELLATION[q].Image - CONSTELLATION[0].Image) ) / (2 * AWGN->sigma * AWGN->sigma);
            }
    }
}