#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
//#include <conio.h>
#include <string.h>
#include <memory.h>
#include <time.h>
//#include <direct.h>
#include "define.cuh"
#include "struct.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Decoder.cuh"

/*
* D:一码输出
* Channel_Out:经过AWGN信道的信号
* Weight_Checknode:校验节点重量
* Weight_Variablenode:变量节点重量
* Address_Variablenode:每个变量节点所对应校验节点的memory_rq的地址
* LDPC:迭代次数
*/
void LDPC_Decoder_GPU(int* D, float* Channel_Out, hipDeviceProp_t prop, int* Address_Variablenode, int* Weight_Checknode, int* Weight_Variablenode, LDPCCode *LDPC)
{
	hipError_t cudaStatus;
	int index0, index1, Length;
	int ThreadPerBlock, Num_Block;
	float* Memory_RQ;
	int* Weight_Checknode_GPU, *Weight_Variablenode_GPU;
	int* D_GPU;
	hipEvent_t GPU_start;			// GPU速率统计参数
	hipEvent_t GPU_stop;
	hipEventCreate(&GPU_start);
	hipEventCreate(&GPU_stop);

	Length = (Message_CW == 0) ? msgLen : CW_Len;

	cudaStatus = hipMalloc((void**)&Memory_RQ, parLen * Weight_Checknode[J] * Num_Frames_OneTime * sizeof(float));	// 锟斤拷锟斤拷锟斤拷GPU锟斤拷global memory锟斤拷
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc Memory_RQ in LDPC_Decoder_GPU on device, exit!\n");
		//getch();
		exit(0);
	}
	cudaStatus = hipMalloc((void**)&D_GPU, (CW_Len + 1) * Num_Frames_OneTime * sizeof(int));		// 锟斤拷锟斤拷锟斤拷GPU锟斤拷global memory锟斤拷
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc D_GPU in LDPC_Decoder_GPU on device, exit!\n");
		//getch();
		exit(0);
	}
	cudaStatus = hipMalloc((void**)&Weight_Checknode_GPU, (J + 1) * sizeof(int));		// 锟斤拷锟斤拷锟斤拷GPU锟斤拷global memory锟斤拷
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc Weight_Checknode_GPU in LDPC_Decoder_GPU on device, exit!\n");
		//getch();
		exit(0);
	}
	cudaStatus = hipMalloc((void**)&Weight_Variablenode_GPU, (L + 1) * sizeof(int));		// 锟斤拷锟斤拷锟斤拷GPU锟斤拷global memory锟斤拷
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot malloc Weight_Checknode_GPU in LDPC_Decoder_GPU on device, exit!\n");
		//getch();
		exit(0);
	}
	cudaStatus = hipMemcpy(Weight_Checknode_GPU, Weight_Checknode, (J + 1) * sizeof(int), hipMemcpyHostToDevice);//J锟斤拷 L锟叫ｏ拷锟斤拷锟揭伙拷锟轿?锟斤拷锟斤拷锟斤拷锟?
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Weight_Checknode to Weight_Checknode_GPU in LDPC_Decoder_GPU, exit!\n");
		//getch();
		exit(0);
	}
	cudaStatus = hipMemcpy(Weight_Variablenode_GPU, Weight_Variablenode, (L + 1) * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot copy Weight_Variablenode to Weight_Variablenode_GPU in LDPC_Decoder_GPU, exit!\n");
		//getch();
		exit(0);
	}

	// 初始化
	cudaStatus = hipMemset(Memory_RQ, 0, parLen * Weight_Checknode[J] * Num_Frames_OneTime * sizeof(float));	// 锟芥储锟斤拷锟斤拷锟斤拷,为锟斤拷一锟轿碉拷锟斤拷锟斤拷准锟斤拷,parlen校锟斤拷位锟斤拷锟饺ｏ拷J*Z锟斤拷
	if (cudaStatus != hipSuccess)
	{
		printf("Cannot memset Memory_RQ in LDPC_Decoder_GPU on device, exit!\n");
		//getch();
		exit(0);
	}

	LDPC->iteraTime = 0;

	hipDeviceSynchronize();

	while (LDPC->iteraTime < maxIT)
	{
		LDPC->iteraTime = LDPC->iteraTime + 1;		
		if ((Z * Num_Frames_OneTime) % prop.maxThreadsPerBlock == 0)	// 此时根据prop.maxThreadsPerBlock分成的每个线程块均属于一个列块或者行块的节点,一个块之间可以共享重量
		{
			hipDeviceSynchronize();
			ThreadPerBlock = prop.maxThreadsPerBlock;
			Num_Block = ((Num_Frames_OneTime * CW_Len) % ThreadPerBlock == 0) ? (Num_Frames_OneTime * CW_Len) / ThreadPerBlock : ((Num_Frames_OneTime * CW_Len) / ThreadPerBlock) + 1;
			Variablenode_Shared_Kernel<<<Num_Block, ThreadPerBlock>>>(Memory_RQ, D_GPU, Channel_Out, Address_Variablenode, Weight_Variablenode_GPU);//变量节点计算，得到L

			ThreadPerBlock = prop.maxThreadsPerBlock;
			Num_Block = ((Num_Frames_OneTime * parLen) % ThreadPerBlock == 0) ? (Num_Frames_OneTime * parLen) / ThreadPerBlock : ((Num_Frames_OneTime * parLen) / ThreadPerBlock) + 1;

			if (decoder_method == 0)
			{
				Checknode_Shared_Kernel<<<Num_Block, ThreadPerBlock>>>(Memory_RQ, Weight_Checknode_GPU);
			}

			hipDeviceSynchronize();
		}
		else
		{
			hipDeviceSynchronize();
			ThreadPerBlock = prop.maxThreadsPerBlock;
			Num_Block = ((Num_Frames_OneTime * CW_Len) % ThreadPerBlock == 0) ? (Num_Frames_OneTime * CW_Len) / ThreadPerBlock : ((Num_Frames_OneTime * CW_Len) / ThreadPerBlock) + 1;
			Variablenode_Kernel<<<Num_Block, ThreadPerBlock>>>(Memory_RQ, D_GPU, Channel_Out, Address_Variablenode, Weight_Variablenode_GPU);//变量节点计算，得到L

			hipDeviceSynchronize();

			ThreadPerBlock = prop.maxThreadsPerBlock;
			Num_Block = ((Num_Frames_OneTime * parLen) % ThreadPerBlock == 0) ? (Num_Frames_OneTime * parLen) / ThreadPerBlock : ((Num_Frames_OneTime * parLen) / ThreadPerBlock) + 1;
			if (decoder_method == 0)
			{
				Checknode_Kernel<<<Num_Block, ThreadPerBlock>>>(Memory_RQ, Weight_Checknode_GPU);
			}
			
			hipDeviceSynchronize();
		}
		hipDeviceSynchronize();

		memset(D + CW_Len * Num_Frames_OneTime, 0, Num_Frames_OneTime * sizeof(int));
		hipMemcpy(D, D_GPU, CW_Len * Num_Frames_OneTime * sizeof(int), hipMemcpyDeviceToHost);

		for (index0 = 0; index0 < Length; index0++)
		{
			for (index1 = 0; index1 < Num_Frames_OneTime; index1++)
			{
				D[index1 + CW_Len * Num_Frames_OneTime] += D[index0 * Num_Frames_OneTime + index1];//把每一帧的所有字节加起来求和
			}
		}
		index0 = 0;
		for (index1 = 0; index1 < Num_Frames_OneTime; index1++)
		{
			D[index1 + CW_Len * Num_Frames_OneTime] = (D[index1 + CW_Len * Num_Frames_OneTime] == 0) ? 1 : 0;//全零序列加起来为全0，正确结果为1
			index0 += D[index1 + CW_Len * Num_Frames_OneTime];//统计对的帧数
		}
		if (index0 == Num_Frames_OneTime)
		{
			break;//所有帧都解码完
		}
		hipDeviceSynchronize();

	}
	hipEventDestroy(GPU_start);
	hipEventDestroy(GPU_stop);

	hipFree(Memory_RQ);
	hipFree(Weight_Checknode_GPU);
	hipFree(Weight_Variablenode_GPU);
	hipFree(D_GPU);
}

/*
* Memory_RQ:用于存储变量节点和校验节点计算时得到的R与Q值
* D:译码输出
* Weight_Variablenode:变量节点重量
* Address_Variablenode:每个变量节点所对应校验节点的memory_rq的地址
*/
__global__ void Variablenode_Kernel(float* Memory_RQ, int* D, float* Channel_Out, int* Address_Variablenode, int* Weight_Variablenode)
{
	int offset, num_Variablenode, num_Frames, num_VariablenodeZ;
	float R[15];
	float Add_result;
	int Ad[15];
	int Weight;

	offset = threadIdx.x + blockIdx.x * blockDim.x;			// 线程号
	num_Variablenode = offset / Num_Frames_OneTime;		// 变量节点序号（16个帧的第一个变量节点-16个帧的第二个节点-————16个帧的最后一个节点）
	num_Frames = offset % Num_Frames_OneTime;		// 帧号
	num_VariablenodeZ = num_Variablenode / Z;					// 分块式校验矩阵中对应的列块号,等于offset / (Z*Num_Frames_OneTime_define)（1个z维矩阵块z个节点）
	num_Variablenode = num_Variablenode * Weight_Variablenode[L];//转换到在Address_Variablenode_GPU里的位置（每个变量节点对应的连接关系，Address_Variablenode_GPU每一块是一个变量节点和所有的连接）

	

	if (offset < CW_Len * Num_Frames_OneTime)//memory数组里是（帧1变量节点1连接的节点——帧2节点的1连接的节点——帧3——————————）
	{
		Weight = Weight_Variablenode[num_VariablenodeZ];
		for (int i = 0; i < Weight; i++)
		{
			Ad[i] = Address_Variablenode[num_Variablenode + i] * Num_Frames_OneTime + num_Frames;
		}
		for (int i = 0; i < Weight; i++)
		{
			R[i] = Memory_RQ[(Ad[i])];
		}
		for (int i = 0; i < Weight; i++)
		{
			Add_result += R[i];
		}
		Add_result += Channel_Out[offset];
		D[offset] = (Add_result < 0) ? 1 : 0;//根据R计算Q，变量节点;
		for (int i = 0; i < Weight;i++)
		{
			Memory_RQ[Ad[i]] = Add_result - R[i];

		}
	}
}
/*
* Memory_RQ:用于存储变量节点和校验节点计算时得到的R与Q值
* D:译码输出
* Weight_Variablenode:变量节点重量
* Address_Variablenode:每个变量节点所对应校验节点的memory_rq的地址
*/
__global__ void Variablenode_Shared_Kernel(float* Memory_RQ, int* D, float* Channel_Out, int* Address_Variablenode, int* Weight_Variablenode)
{
	int offset, num_Variablenode, num_Frames, num_VariablenodeZ;
	float R[15];
	float Add_result;
	int Ad[15];
	__shared__ int Weight;

	offset = threadIdx.x + blockIdx.x * blockDim.x;			// 线程号
	num_Variablenode = offset / Num_Frames_OneTime;		// 变量节点序号（16个帧的第一个变量节点-16个帧的第二个节点-————16个帧的最后一个节点）
	num_Frames = offset % Num_Frames_OneTime;		// 帧号
	num_VariablenodeZ = num_Variablenode / Z;					// 分块式校验矩阵中对应的列块号,等于offset / (Z*Num_Frames_OneTime_define)（1个z维矩阵块z个节点）
	num_Variablenode = num_Variablenode * Weight_Variablenode[L];//转换到在Address_Variablenode_GPU里的位置（每个变量节点对应的连接关系，Address_Variablenode_GPU每一块是一个变量节点和所有的连接）

	if (threadIdx.x == 0 && num_VariablenodeZ < L)
	{
		Weight = Weight_Variablenode[num_VariablenodeZ];//只需要赋一次值
	}
	__syncthreads();

	if (offset < CW_Len * Num_Frames_OneTime)//memory数组里是（帧1变量节点1连接的节点——帧2节点的1连接的节点——帧3——————————）
	{
		// 计算地址的时候不需要根据重量
		for (int i = 0; i < Weight; i++)
		{
			Ad[i] = Address_Variablenode[num_Variablenode + i] * Num_Frames_OneTime + num_Frames;
		}
		for (int i = 0; i < Weight; i++)
		{
			R[i] = Memory_RQ[(Ad[i])];
		}
		for (int i = 0; i < Weight; i++)
		{
			Add_result += R[i];
		}
		Add_result += Channel_Out[offset];
		D[offset] = (Add_result < 0) ? 1 : 0;//根据R计算Q，变量节点;
		for (int i = 0; i < Weight; i++)
		{
			Memory_RQ[Ad[i]] = Add_result - R[i];

		}
	}
}
__global__ void Checknode_Kernel(float* Memory_RQ, int* Weight_Checknode)
{
	int offset, num_Checknode, num_Frames, num_ChecknodeZ;
	__shared__ int Weight;
	float Q[25], Q0[25];
	int Sign[26];
	float MinQ, SubMinQ;
	int Index_minQ;

	offset = threadIdx.x + blockIdx.x * blockDim.x;
	num_Checknode = offset / Num_Frames_OneTime;													// 校验节点序号
	num_Frames = offset % Num_Frames_OneTime;													// 帧号
	num_Frames = num_Frames + num_Checknode * Num_Frames_OneTime * Weight_Checknode[J];	// 当前帧的该校验节点第0个Q值的存放地址
	num_ChecknodeZ = num_Checknode / Z;																	// 当前校验节点所在的列块号

	

	if (offset < Num_Frames_OneTime * parLen)//q就是memory_rq的一行
	{
		Weight = Weight_Checknode[num_ChecknodeZ];
		for (int i = 0; i < Weight; i++)
		{
			Q[i] = Memory_RQ[num_Frames + i * Num_Frames_OneTime];
		}
		for (int i = 0; i < Weight; i++)
		{
			Sign[i] = (Q[i] < 0) ? -1 : 1;
			Q[i] = (Q[i] < 0) ? -Q[i] : Q[i];
			Q0[i] = Q[i];
		}
		Sign[25] = 1;
		for (int i = 0; i < Weight; i++)
		{
			Sign[25] *= Sign[i];
		}
		sortQ(&MinQ, &SubMinQ, Q, Weight);
		for (int i = 0; i < Weight; i++)
		{
			if (Q0[i] == MinQ)
			{
				Index_minQ = i;
				break;
			}
		}
		for (int i = 0; i < Weight; i++)
		{
			if (i != Index_minQ)
			{
				Memory_RQ[num_Frames + i * Num_Frames_OneTime] = Sign[25] * Sign[i] * MinQ;
			}
			else Memory_RQ[num_Frames + i * Num_Frames_OneTime] = Sign[25] * Sign[i] * SubMinQ;
		}
	}
}
__global__ void Checknode_Shared_Kernel(float* Memory_RQ, int* Weight_Checknode)
{
	int offset, num_Checknode, num_Frames, num_ChecknodeZ;
	__shared__ int Weight;
	float Q[25],Q0[25];
	int Sign[26];
	float MinQ, SubMinQ;
	int Index_minQ;

	offset = threadIdx.x + blockIdx.x * blockDim.x;
	num_Checknode = offset / Num_Frames_OneTime;													// 校验节点序号
	num_Frames = offset % Num_Frames_OneTime;													// 帧号
	num_Frames = num_Frames + num_Checknode * Num_Frames_OneTime * Weight_Checknode[J];	// 当前帧的该校验节点第0个Q值的存放地址
	num_ChecknodeZ = num_Checknode / Z;																	// 当前校验节点所在的列块号

	if (threadIdx.x == 0 && num_ChecknodeZ < J)		// 用第0个线程找到该线程块中所有线程对应的重量
	{
		Weight = Weight_Checknode[num_ChecknodeZ];
	}
	__syncthreads();

	if (offset < Num_Frames_OneTime * parLen)//q就是memory_rq的一行
	{
		for (int i = 0; i < Weight; i++)
		{
			Q[i] = Memory_RQ[num_Frames + i * Num_Frames_OneTime];
		}
		for (int i = 0; i < Weight; i++)
		{
			Sign[i]= (Q[i] < 0) ? -1 : 1;
			Q[i] = (Q[i] < 0) ? -Q[i] : Q[i];
			Q0[i] = Q[i];
		}
		Sign[25] = 1;
		for (int i = 0; i < Weight; i++)
		{
			Sign[25] *= Sign[i];
		}
		sortQ(&MinQ, &SubMinQ, Q, Weight);
		for (int i = 0; i < Weight;i++)
		{
			if (Q0[i] == MinQ)
			{
				Index_minQ = i;
				break;
			}
		}
		for (int i = 0; i < Weight; i++)
		{
			if (i != Index_minQ)
			{
				Memory_RQ[num_Frames + i * Num_Frames_OneTime] = Sign[25] * Sign[i] * MinQ;
			}
			else Memory_RQ[num_Frames + i * Num_Frames_OneTime] = Sign[25] * Sign[i] * SubMinQ;
		}
	}
}

__device__ void sortQ(float* MinQ, float* SubMinQ, float* Q,int Weight)
{
	float tmp;
	for (int i = 0; i < 2; i++) {

		for (int j = 0; j < Weight-1; j++) 
		{

			if (Q[j] < Q[j + 1]) 
			{

				tmp = Q[j];

				Q[j] = Q[j + 1];

				Q[j + 1] = tmp;

			}

		}

	}
	*MinQ = Q[Weight - 1];
	*SubMinQ = Q[Weight - 2];
}