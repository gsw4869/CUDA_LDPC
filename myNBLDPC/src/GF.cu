/*
 * @Author: your name
 * @Date: 2021-04-09 16:52:00
 * @LastEditTime: 2021-04-09 17:18:09
 * @LastEditors: Please set LastEditors
 * @Description: In User Settings Edit
 * @FilePath: /myNBLDPC/src/GF.cpp
 */
#include "GF.cuh"
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <assert.h>
using namespace std;

unsigned **TableAdd;
unsigned **TableMultiply;
unsigned *TableInverse;

unsigned **malloc_2(int xDim, int yDim)
{
	unsigned **a = (unsigned **)malloc(xDim * sizeof(unsigned *));
	a[0] = (unsigned *)malloc(xDim * yDim * sizeof(unsigned));
	memset(a[0], 0, xDim * yDim * sizeof(unsigned));
	for (int i = 1; i < xDim; i++)
	{
		a[i] = a[i - 1] + yDim;
	}
	assert(a != NULL);
	return a;
}

float **malloc_2_float(int xDim, int yDim)
{
	float **a = (float **)malloc(xDim * sizeof(float *));
	a[0] = (float *)malloc(xDim * yDim * sizeof(float));
	memset(a[0], 0, xDim * yDim * sizeof(float));
	for (int i = 1; i < xDim; i++)
	{
		a[i] = a[i - 1] + yDim;
	}
	assert(a != NULL);
	return a;
}

int GFAdd(int ele1, int ele2)
{
	return ele1 ^ ele2;
}

int GFMultiply(int ele1, int ele2)
{
	return TableMultiply[ele1][ele2];
}

int GFInverse(int ele)
{
	if (ele == 0)
	{
		printf("Div 0 Error!\n");
		exit(-1);
	}
	return TableInverse[ele];
}

bool GFInitial(int GFq)
{
	// calculate order
	int q = GFq;
	// allocate memory space
	TableAdd = malloc_2(q, q);
	TableMultiply = malloc_2(q, q);
	TableInverse = new unsigned[q];

	// read profile
	stringstream ss;
	ss << q << ".txt";
	//Arithmetic Table
	string ArithTableFileName = "./GF/Arith.Table.GF.";
	ArithTableFileName += ss.str();
	ifstream ArithFin(ArithTableFileName);
	if (!ArithFin.is_open())
	{
		cerr << "Cannot open " << ArithTableFileName << endl;
		exit(-1);
	}
	string rub;
	getline(ArithFin, rub);
	//	cout << "Read Arithmetic Table File: " << rub << "..." << endl;
	ArithFin >> rub >> rub;
	for (int i = 0; i < q; i++)
	{
		for (int j = 0; j < q; j++)
		{
			ArithFin >> TableMultiply[i][j];
		}
	}
	ArithFin >> rub >> rub;
	for (int i = 0; i < q; i++)
	{
		for (int j = 0; j < q; j++)
		{
			ArithFin >> TableAdd[i][j];
		}
	}
	ArithFin >> rub >> rub;
	for (int i = 0; i < q; i++)
	{
		ArithFin >> TableInverse[i];
	}
	ArithFin.close();
	//	cout << "done." << endl;

	return true;
}
