#include "Simulation.cuh"
#include "LDPC_Encoder.cuh"
#include "LDPC_Decoder.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <assert.h>
#include <fstream>
#include <string>
#include "Decode_GPU.cuh"
#include <ctime>
#include <thread>

/*
* 仿真函数
* AWGN:AWGNChannel类变量，包含噪声种子等
* 
*/
void Simulation_CPU(LDPCCode *H, AWGNChannel *AWGN, Simulation *SIM, CComplex *CONSTELLATION, VN *Variablenode, CN *Checknode, CComplex *CComplex_sym, int *CodeWord_sym, int *DecodeOutput)
{
	int iter_number = 0;

	CComplex *CComplex_sym_Channelout;
	if (n_QAM != 2)
	{
		CComplex_sym_Channelout = (CComplex *)malloc(H->Variablenode_num * sizeof(CComplex));
	}
	else
	{
		CComplex_sym_Channelout = (CComplex *)malloc(H->bit_length * sizeof(CComplex));
	}
	std::chrono::_V2::steady_clock::time_point start = std::chrono::steady_clock::now();
	std::chrono::_V2::steady_clock::time_point end;
	while (SIM->num_Error_Frames < leastErrorFrames)
	{
		// printf("%d\n",SIM->num_Frames);
		SIM->num_Frames += 1;

		AWGNChannel_CPU(H, AWGN, CComplex_sym_Channelout, CComplex_sym);

		Demodulate(H, AWGN, CONSTELLATION, Variablenode, CComplex_sym_Channelout);

		Decoding_EMS(H, Variablenode, Checknode, H->GF, 1, DecodeOutput, iter_number);

		end = std::chrono::steady_clock::now();

		SIM->sumTime = (end - start).count() / 1000000000.0;

		SIM->Total_Iteration += iter_number;

		Statistic(SIM, CodeWord_sym, DecodeOutput, H);
	}
	free(CComplex_sym_Channelout);
}

/*
* 仿真函数
* AWGN:AWGNChannel类变量，包含噪声种子等
* 
*/
void Simulation_GPU(LDPCCode *H, AWGNChannel *AWGN, Simulation *SIM, CComplex *CONSTELLATION, VN *Variablenode, CN *Checknode, CComplex *CComplex_sym, int *CodeWord_sym, int *DecodeOutput, unsigned *TableMultiply_GPU, unsigned *TableAdd_GPU, int *Checknode_weight, int *Variablenode_linkCNs, int *Checknode_linkVNs, int *Checknode_linkVNs_GF)
{
	int iter_number = 0;
	CComplex *CComplex_sym_Channelout;
	if (n_QAM != 2)
	{
		CComplex_sym_Channelout = (CComplex *)malloc(H->Variablenode_num * sizeof(CComplex));
	}
	else
	{
		CComplex_sym_Channelout = (CComplex *)malloc(H->bit_length * sizeof(CComplex));
	}
	std::chrono::_V2::steady_clock::time_point start = std::chrono::steady_clock::now();
	std::chrono::_V2::steady_clock::time_point end;
	while (SIM->num_Error_Frames < leastErrorFrames)
	{
		// printf("%d\n",SIM->num_Frames);
		SIM->num_Frames += 1;

		AWGNChannel_CPU(H, AWGN, CComplex_sym_Channelout, CComplex_sym);

		Demodulate(H, AWGN, CONSTELLATION, Variablenode, CComplex_sym_Channelout);

		Decoding_EMS_GPU(H, Variablenode, Checknode, H->GF / 2, 2, DecodeOutput, TableMultiply_GPU, TableAdd_GPU, Checknode_weight, Variablenode_linkCNs, Checknode_linkVNs, Checknode_linkVNs_GF, iter_number);

		end = std::chrono::steady_clock::now();

		SIM->sumTime = (end - start).count() / 1000000000.0;

		SIM->Total_Iteration += iter_number;

		Statistic(SIM, CodeWord_sym, DecodeOutput, H);
	}
	free(CComplex_sym_Channelout);
}

/*
* 统计函数，统计仿真结果
*/
int Statistic(Simulation *SIM, int *CodeWord_Frames, int *D, LDPCCode *H)
{
	int index1;
	int Error_msgBit = 0;

	for (index1 = 0; index1 < H->Variablenode_num; index1++)
	{
		Error_msgBit = (D[index1] != CodeWord_Frames[index1]) ? Error_msgBit + 1 : Error_msgBit;
	}
	SIM->num_Error_Bits += Error_msgBit;
	SIM->num_Error_Frames = (Error_msgBit != 0) ? SIM->num_Error_Frames + 1 : SIM->num_Error_Frames;
	// SIM->num_Error_Frames = (Error_msgBit!= 0 || D[index0 + CW_Len * Num_Frames_OneTime] == 0) ? SIM->num_Error_Frames + 1 : SIM->num_Error_Frames;
	// SIM->num_Alarm_Frames = (Error_msgBit[index0] == 0 && D[index0 + CW_Len * Num_Frames_OneTime] == 0) ? SIM->num_Alarm_Frames + 1 : SIM->num_Alarm_Frames;
	// SIM->num_False_Frames = (Error_msgBit[index0] != 0 && D[index0 + CW_Len * Num_Frames_OneTime] == 1) ? SIM->num_False_Frames + 1 : SIM->num_False_Frames;

	if (SIM->num_Frames % displayStep == 0)
	{
		SIM->BER = ((double)SIM->num_Error_Bits / (double)(SIM->num_Frames)) / (double)(H->Variablenode_num);
		SIM->FER = (double)SIM->num_Error_Frames / (double)SIM->num_Frames;
		SIM->AverageIT = (double)SIM->Total_Iteration / (double)SIM->num_Frames;
		// SIM->FER_Alarm = (double)SIM->num_Alarm_Frames / (double)SIM->num_Frames;
		// SIM->FER_False = (double)SIM->num_False_Frames / (double)SIM->num_Frames;
		printf(" %.1f %8d  %4d  %6.4e  %6.4e  %.2f  %6.4esec\n", SIM->SNR, SIM->num_Frames, SIM->num_Error_Frames, SIM->FER, SIM->BER, SIM->AverageIT, SIM->sumTime / SIM->num_Frames);
		FILE *fp_H;
		if (NULL == (fp_H = fopen("results.txt", "a")))
		{
			printf("can not open file: results.txt\n");
			exit(0);
		}
		fprintf(fp_H, " %.1f %8d  %4d  %6.4e  %6.4e  %.2f  %6.4esec\n", SIM->SNR, SIM->num_Frames, SIM->num_Error_Frames, SIM->FER, SIM->BER, SIM->AverageIT, SIM->sumTime / SIM->num_Frames);
		fclose(fp_H);
	}

	if (SIM->num_Error_Frames >= leastErrorFrames && SIM->num_Frames >= leastTestFrames)
	{
		SIM->BER = ((double)SIM->num_Error_Bits / (double)(SIM->num_Frames)) / (double)(H->Variablenode_num);
		SIM->FER = (double)SIM->num_Error_Frames / (double)SIM->num_Frames;
		SIM->AverageIT = (double)SIM->Total_Iteration / (double)SIM->num_Frames;
		// SIM->FER_Alarm = (double)SIM->num_Alarm_Frames / (double)SIM->num_Frames;
		// SIM->FER_False = (double)SIM->num_False_Frames / (double)SIM->num_Frames;
		printf(" %.1f %8d  %4d  %6.4e  %6.4e  %.2f  %6.4esec\n", SIM->SNR, SIM->num_Frames, SIM->num_Error_Frames, SIM->FER, SIM->BER, SIM->AverageIT, SIM->sumTime / SIM->num_Frames);
		FILE *fp_H;
		if (NULL == (fp_H = fopen("results.txt", "a")))
		{
			printf("can not open file: results.txt\n");
			exit(0);
		}
		fprintf(fp_H, " %.1f %8d  %4d  %6.4e  %6.4e  %.2f  %6.4esec\n", SIM->SNR, SIM->num_Frames, SIM->num_Error_Frames, SIM->FER, SIM->BER, SIM->AverageIT, SIM->sumTime / SIM->num_Frames);
		fclose(fp_H);
		return 1;
	}
	return 0;
}

CComplex *Get_CONSTELLATION(LDPCCode *H)
{
	CComplex *CONSTELLATION = new CComplex[H->GF];

	char temp[100];
	FILE *fp_H;
	if (NULL == (fp_H = fopen(Constellationfile, "r")))
	{
		printf("can not open file: %s\n", Constellationfile);
		exit(0);
	}

	int index;
	for (int k = 0; k < n_QAM; k++)
	{
		fscanf(fp_H, "%s", temp);
		fscanf(fp_H, "%d", &index);
		fscanf(fp_H, "%s", temp);
		fscanf(fp_H, "%f", &CONSTELLATION[index].Real); // GF域
		fscanf(fp_H, "%s", temp);
		fscanf(fp_H, "%f", &CONSTELLATION[index].Image); // GF域
	}
	fclose(fp_H);

	return CONSTELLATION;
}

/*
H:校验矩阵
Weight_Checknode:按顺序记录每个校验节点的重量
Weight_Variablenode:按顺序记录每个变量节点的重量
Address_Variablenode:变量节点相连的校验节点的序号
Address_Checknode:校验节点相连的变量节点的序号
*/
void Get_H(LDPCCode *H, VN *Variablenode, CN *Checknode)
{
	int index1;

	FILE *fp_H;

	if (NULL == (fp_H = fopen(Matrixfile, "r")))
	{
		printf("can not open file: %s\n", Matrixfile);
		exit(0);
	}

	fscanf(fp_H, "%d", &H->Variablenode_num); // 变量节点个数（行数）
	// Variablenode=(VN *)malloc(H->Variablenode_num*sizeof(VN));

	fscanf(fp_H, "%d", &H->Checknode_num); // 校验节点个数（列数）
	// Checknode=(CN *)malloc(H->Checknode_num*sizeof(CN));

	H->rate = (float)(H->Variablenode_num - H->Checknode_num) / H->Variablenode_num;

	fscanf(fp_H, "%d", &H->GF); // GF域

	switch (H->GF)
	{
	case 4:
		H->q_bit = 2;
		break;
	case 8:
		H->q_bit = 3;
		break;
	case 16:
		H->q_bit = 4;
		break;
	case 32:
		H->q_bit = 5;
		break;
	case 64:
		H->q_bit = 6;
		break;
	case 128:
		H->q_bit = 7;
		break;
	case 256:
		H->q_bit = 8;
		break;
	default:
		printf("error");
		exit(0);
	}

	H->bit_length = H->Variablenode_num * H->q_bit;

	fscanf(fp_H, "%d", &H->maxWeight_variablenode); //变量节点相连的校验节点的个数

	fscanf(fp_H, "%d", &H->maxWeight_checknode); //校验节点相连的变量节点的个数

	for (int i = 0; i < H->Variablenode_num; i++)
	{
		fscanf(fp_H, "%d", &index1);
		Variablenode[i].weight = index1;
		Variablenode[i].linkCNs = (int *)malloc(Variablenode[i].weight * sizeof(int));
		Variablenode[i].linkCNs_GF = (int *)malloc(Variablenode[i].weight * sizeof(int));
		Variablenode[i].L_ch = (float *)malloc((H->GF - 1) * sizeof(float));
		Variablenode[i].LLR = (float *)malloc((H->GF - 1) * sizeof(float));
		Variablenode[i].Entr_v2c = malloc_2_float(Variablenode[i].weight, H->GF);
		Variablenode[i].sort_L_v2c = malloc_2_float(Variablenode[i].weight, H->GF);
		Variablenode[i].sort_Entr_v2c = malloc_2(Variablenode[i].weight, H->GF);
	}

	for (int i = 0; i < H->Checknode_num; i++)
	{
		fscanf(fp_H, "%d", &index1);
		Checknode[i].weight = index1;
		Checknode[i].linkVNs = (int *)malloc(Checknode[i].weight * sizeof(int));
		Checknode[i].linkVNs_GF = (int *)malloc(Checknode[i].weight * sizeof(int));
		Checknode[i].L_c2v = malloc_2_float(Checknode[i].weight, H->GF);
	}

	for (int i = 0; i < H->Variablenode_num; i++)
	{
		for (int j = 0; j < Variablenode[i].weight; j++)
		{
			fscanf(fp_H, "%d", &index1);
			Variablenode[i].linkCNs[j] = index1 - 1;
			fscanf(fp_H, "%d", &index1);
			Variablenode[i].linkCNs_GF[j] = index1;
		}
	}

	for (int i = 0; i < H->Checknode_num; i++)
	{
		for (int j = 0; j < Checknode[i].weight; j++)
		{
			fscanf(fp_H, "%d", &index1);
			Checknode[i].linkVNs[j] = index1 - 1;
			fscanf(fp_H, "%d", &index1);
			Checknode[i].linkVNs_GF[j] = index1;
		}
	}

	fclose(fp_H);
}