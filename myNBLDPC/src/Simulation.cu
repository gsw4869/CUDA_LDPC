#include "Simulation.cuh"
#include "LDPC_Encoder.cuh"
#include "LDPC_Decoder.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <assert.h>

/*
* 仿真函数
* AWGN:AWGNChannel类变量，包含噪声种子等
* 
*/
void Simulation_GPU(Simulation* SIM, VN* Variablenode, CN* Checknode, float* Channel_Out)
{	
	// while (SIM->num_Frames<50)
	// {
	// 	SIM->num_Frames += 1;
	// }
	SIM->num_Frames = 40960;
}

/*
* 统计函数，统计仿真结果
*/
int Statistic(Simulation* SIM, int* CodeWord_Frames, int* D,LDPCCode *H)
{
	int index1;
	int Error_msgBit=0;	

	
	for (index1 = 0; index1 < H->length; index1++)
	{
		Error_msgBit = (D[index1] != CodeWord_Frames[index1]) ? Error_msgBit + 1 : Error_msgBit;
	}
	SIM->num_Error_Bits += Error_msgBit;
	SIM->num_Error_Frames = (Error_msgBit!= 0) ? SIM->num_Error_Frames + 1 : SIM->num_Error_Frames;
	// SIM->num_Error_Frames = (Error_msgBit!= 0 || D[index0 + CW_Len * Num_Frames_OneTime] == 0) ? SIM->num_Error_Frames + 1 : SIM->num_Error_Frames;
	// SIM->num_Alarm_Frames = (Error_msgBit[index0] == 0 && D[index0 + CW_Len * Num_Frames_OneTime] == 0) ? SIM->num_Alarm_Frames + 1 : SIM->num_Alarm_Frames;
	// SIM->num_False_Frames = (Error_msgBit[index0] != 0 && D[index0 + CW_Len * Num_Frames_OneTime] == 1) ? SIM->num_False_Frames + 1 : SIM->num_False_Frames;
	SIM->Total_Iteration += H->iteraTime;
	
	if (SIM->num_Frames % displayStep == 0)
	{
		SIM->BER = ((double)SIM->num_Error_Bits / (double)(SIM->num_Frames)) / (double)(H->length);
		SIM->FER = (double)SIM->num_Error_Frames / (double)SIM->num_Frames;
		SIM->AverageIT = (double)SIM->Total_Iteration / (double)SIM->num_Frames;
		// SIM->FER_Alarm = (double)SIM->num_Alarm_Frames / (double)SIM->num_Frames;
		// SIM->FER_False = (double)SIM->num_False_Frames / (double)SIM->num_Frames;
		printf(" %.1f %8d  %4d  %6.4e  %6.4e  %.2f  %6.4e %6.4e\n", SIM->SNR, SIM->num_Frames, SIM->num_Error_Frames, SIM->FER, SIM->BER, SIM->AverageIT, SIM->FER_False, SIM->FER_Alarm);
	}

	if (SIM->num_Error_Frames >= leastErrorFrames && SIM->num_Frames >= leastTestFrames)
	{
		SIM->BER = ((double)SIM->num_Error_Bits / (double)(SIM->num_Frames)) / (double)(H->length);
		SIM->FER = (double)SIM->num_Error_Frames / (double)SIM->num_Frames;
		SIM->AverageIT = (double)SIM->Total_Iteration / (double)SIM->num_Frames;
		// SIM->FER_Alarm = (double)SIM->num_Alarm_Frames / (double)SIM->num_Frames;
		// SIM->FER_False = (double)SIM->num_False_Frames / (double)SIM->num_Frames;
		printf(" %.1f %8d  %4d  %6.4e  %6.4e  %.2f  %6.4e %6.4e\n", SIM->SNR, SIM->num_Frames, SIM->num_Error_Frames, SIM->FER, SIM->BER, SIM->AverageIT, SIM->FER_False, SIM->FER_Alarm);
		return 1;
	}
	return 0;
}

/*
H:校验矩阵
Weight_Checknode:按顺序记录每个校验节点的重量
Weight_Variablenode:按顺序记录每个变量节点的重量
Address_Variablenode:变量节点相连的校验节点的序号
Address_Checknode:校验节点相连的变量节点的序号
*/
void Get_H(LDPCCode* H,VN* Variablenode,CN* Checknode)
{
	int index1;
	char file[100]="Tanner_74_9_Z128_GF16.txt";
	FILE* fp_H;
	
	if (NULL == (fp_H = fopen(file, "r")))
	{
		printf("can not open file: %s\n", file);
		exit(0);
	}

	fscanf(fp_H, "%d", &H->Variablenode_num);// 变量节点个数（行数）
	Variablenode=(VN *)malloc(H->Variablenode_num*sizeof(VN));

	fscanf(fp_H, "%d", &H->Checknode_num);// 校验节点个数（列数）
	Checknode=(CN *)malloc(H->Checknode_num*sizeof(CN));

	H->rate=(float)(H->Variablenode_num-H->Checknode_num)/H->Variablenode_num;
    H->length=H->Variablenode_num;
	fscanf(fp_H, "%d", &index1);// GF域

	fscanf(fp_H, "%d", &H->maxWeight_variablenode);//最大行重

	fscanf(fp_H, "%d", &H->maxWeight_checknode);//最大列重


	for(int i=0;i<H->Variablenode_num;i++)
	{
		fscanf(fp_H, "%d", &index1);
		Variablenode[i].weight=index1;
		Variablenode[i].linkCNs=(int *)malloc(Variablenode[i].weight*sizeof(int));
		Variablenode[i].linkCNs_GF=(int *)malloc(Variablenode[i].weight*sizeof(int));
	}

	
	for(int i=0;i<H->Checknode_num;i++)
	{
		fscanf(fp_H, "%d", &index1);
		Checknode[i].weight=index1;
		Checknode[i].linkVNs=(int *)malloc(Checknode[i].weight*sizeof(int));
		Checknode[i].linkVNs_GF=(int *)malloc(Checknode[i].weight*sizeof(int));
	}
	
	for(int i=0;i<H->Variablenode_num;i++)
	{
		for(int j=0;j<Variablenode[i].weight;j++)
		{
			fscanf(fp_H, "%d", &index1);
			Variablenode[i].linkCNs[j]=index1;
			fscanf(fp_H, "%d", &index1);
			Variablenode[i].linkCNs_GF[j]=index1;

		}
	}

	for(int i=0;i<H->Checknode_num;i++)
	{
		for(int j=0;j<Checknode[i].weight;j++)
		{
			fscanf(fp_H, "%d", &index1);
			Checknode[i].linkVNs[j]=index1;
			fscanf(fp_H, "%d", &index1);
			Checknode[i].linkVNs_GF[j]=index1;

		}
	}

	fclose(fp_H);

	// for(int i=0;i<H->Checknode_num;i++)
	// {
	// 	for(int j=0;j<Checknode[i].weight;j++)
	// 	{
	// 		printf("%d ",Checknode[i].linkVNs_GF[j]);

	// 	}
	// 	printf("\n");
	// }

}