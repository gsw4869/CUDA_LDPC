#include "Simulation.cuh"
#include "LDPC_Encoder.cuh"
#include "LDPC_Decoder.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <assert.h>


int **malloc_2(int xDim, int yDim) {
    int **a = (int **)malloc(xDim * sizeof(int *));
    a[0] = (int *)malloc(xDim * yDim * sizeof(int));
    memset(a[0], 0, xDim * yDim * sizeof(int));
    for (int i = 1; i < xDim; i++) {
        a[i] = a[i - 1] + yDim;
    }
    assert(a != NULL);
    return a;
}

/*
H:校验矩阵
Weight_Checknode:按顺序记录每个校验节点的重量
Weight_Variablenode:按顺序记录每个变量节点的重量
Address_Variablenode:变量节点相连的校验节点的序号
Address_Checknode:校验节点相连的变量节点的序号
*/
void Get_H(VN* Variablenode,CN* Checknode)
{
	int Weight_Variablenode_num;
	int Weight_Checknode_num;
	int max_Weight_checknode;
	int max_Weight_variablenode;
	int index1;
	char file[100]="Tanner_74_9_Z128_GF16.txt";
	FILE* fp_H;
	
	if (NULL == (fp_H = fopen(file, "r")))
	{
		printf("can not open file: %s\n", file);
		exit(0);
	}

	fscanf(fp_H, "%d", &Weight_Variablenode_num);// 变量节点个数（行数）
	Variablenode=(VN *)malloc(Weight_Variablenode_num*sizeof(VN));

	fscanf(fp_H, "%d", &Weight_Checknode_num);// 校验节点个数（列数）
	Checknode=(CN *)malloc(Weight_Checknode_num*sizeof(CN));

	fscanf(fp_H, "%d", &index1);// GF域

	fscanf(fp_H, "%d", &max_Weight_variablenode);//最大行重
	// Address_Variablenode=malloc_2(Weight_Variablenode_num,max_Weight_variablenode);

	fscanf(fp_H, "%d", &max_Weight_checknode);//最大列重
	// Address_Checknode=malloc_2(Weight_Checknode_num,max_Weight_checknode);

	for(int i=0;i<Weight_Variablenode_num;i++)
	{
		fscanf(fp_H, "%d", &index1);
		Variablenode[i].weight=index1;
		Variablenode[i].linkCNs=(int *)malloc(Variablenode[i].weight*sizeof(int));
		Variablenode[i].linkCNs_GF=(int *)malloc(Variablenode[i].weight*sizeof(int));
	}

	
	for(int i=0;i<Weight_Checknode_num;i++)
	{
		fscanf(fp_H, "%d", &index1);
		Checknode[i].weight=index1;
		Checknode[i].linkVNs=(int *)malloc(Checknode[i].weight*sizeof(int));
		Checknode[i].linkVNs_GF=(int *)malloc(Checknode[i].weight*sizeof(int));
	}
	
	for(int i=0;i<Weight_Variablenode_num;i++)
	{
		for(int j=0;j<Variablenode[i].weight;j++)
		{
			fscanf(fp_H, "%d", &index1);
			Variablenode[i].linkCNs[j]=index1;
			fscanf(fp_H, "%d", &index1);
			Variablenode[i].linkCNs_GF[j]=index1;

		}
	}

	for(int i=0;i<Weight_Checknode_num;i++)
	{
		for(int j=0;j<Checknode[i].weight;j++)
		{
			fscanf(fp_H, "%d", &index1);
			Checknode[i].linkVNs[j]=index1;
			fscanf(fp_H, "%d", &index1);
			Checknode[i].linkVNs_GF[j]=index1;

		}
	}

	fclose(fp_H);

	for(int i=0;i<Weight_Checknode_num;i++)
	{
		for(int j=0;j<Checknode[i].weight;j++)
		{
			printf("%d ",Checknode[i].linkVNs_GF[j]);
		}
		printf("\n");
	}
}