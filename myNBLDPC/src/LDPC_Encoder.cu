#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Encoder.cuh"
#include "struct.cuh"


/*
* CodeWord：原始码组
* Channel_Out：经过BPSK调制的输出信号
*/
void AWGNChannel_CPU(LDPCCode* H, AWGNChannel* AWGN, float* Channel_Out,int* CodeWord)
{
	int index0;
	float u1, u2, temp;
	for (index0 = 0; index0 < H->length; index0++)
	{
		u1 = RandomModule(AWGN->seed);
		u2 = RandomModule(AWGN->seed);

		temp = (float)sqrt((float)(-2) * log((float)1 - u1));
		*(Channel_Out + index0) = (AWGN->sigma) * sin(2 * PI * u2) * temp + 1.0 - 2 * (*(CodeWord + index0));//产生高斯白噪声信号(https://www.cnblogs.com/tsingke/p/6194737.html)
	}
}



float RandomModule(int* seed)
{
	float temp = 0.0;
	seed[0] = (seed[0] * 249) % 61967;
	seed[1] = (seed[1] * 251) % 63443;
	seed[2] = (seed[2] * 252) % 63599;
	temp = (((float)seed[0]) / ((float)61967)) + (((float)seed[1]) / ((float)63443))
		+ (((float)seed[2]) / ((float)63599));
	temp -= (int)temp;
	return (temp);
}
