#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Encoder.cuh"
#include "struct.cuh"


void BitToSym(LDPCCode* H,int* CodeWord_sym,int* CodeWord_bit)
{
	for(int s = 0; s < H->Variablenode_num; s ++)
	{
		CodeWord_sym[s] = 0;
		for (int i = 0; i < H->q_bit; i++) {
			CodeWord_sym[s] = 2 * CodeWord_sym[s] + CodeWord_bit[H->q_bit*s+ H->q_bit -1-i];
 		}
			
	}
}

void Modulate(LDPCCode* H,CComplex* CONSTELLATION,CComplex* CComplex_sym,int* CodeWord_sym)
{
	for(int s = 0; s < H->Variablenode_num; s ++)
	{
		CComplex_sym[s].Real = CONSTELLATION[CodeWord_sym[s]].Real;
		CComplex_sym[s].Image = CONSTELLATION[CodeWord_sym[s]].Image;
	}	
}
/*
* CodeWord：原始码组
* Channel_Out：经过BPSK调制的输出信号
*/
void AWGNChannel_CPU(LDPCCode* H, AWGNChannel* AWGN, CComplex* CComplex_sym_Channelout,CComplex* CComplex_sym)
{
	int index0;
	float u1, u2, temp;
	for (index0 = 0; index0 < H->Variablenode_num; index0++)
	{
		
		u1 = RandomModule(AWGN->seed);
		u2 = RandomModule(AWGN->seed);

		temp = (float)sqrt((float)(-2) * log((float)1 - u1));
		CComplex_sym_Channelout[index0].Real = (AWGN->sigma) * sin(2 * PI * u2) * temp + CComplex_sym[index0].Real;//产生高斯白噪声信号(https://www.cnblogs.com/tsingke/p/6194737.html)

		u1 = RandomModule(AWGN->seed);
		u2 = RandomModule(AWGN->seed);

		temp = (float)sqrt((float)(-2) * log((float)1 - u1));
		CComplex_sym_Channelout[index0].Image = (AWGN->sigma) * sin(2 * PI * u2) * temp + CComplex_sym[index0].Image;
		
	}
}



float RandomModule(int* seed)
{
	float temp = 0.0;
	seed[0] = (seed[0] * 249) % 61967;
	seed[1] = (seed[1] * 251) % 63443;
	seed[2] = (seed[2] * 252) % 63599;
	temp = (((float)seed[0]) / ((float)61967)) + (((float)seed[1]) / ((float)63443))
		+ (((float)seed[2]) / ((float)63599));
	temp -= (int)temp;
	return (temp);
}
