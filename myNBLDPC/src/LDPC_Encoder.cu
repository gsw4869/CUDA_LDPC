#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "LDPC_Encoder.cuh"
#include "struct.cuh"

/*
* CodeWord：原始码组
* BPSK_Out：经过BPSK调制的输出信号
*/
__global__ void BPSK(float* BPSK_Out, int* CodeWord)
{
	int offset;
	offset = threadIdx.x + blockDim.x * blockIdx.x;

	if (offset < Num_Frames_OneTime * CW_Len)
	{
		*(BPSK_Out + offset) = 1.0 - 2*(*(CodeWord + offset));
	}
}

/*
* CodeWord：原始码组
* Channel_Out：经过BPSK调制的输出信号
*/
void AWGNChannel_CPU(AWGNChannel* AWGN, float* Channel_Out,int* CodeWord)
{
	int index0, index1;
	float u1, u2, temp;

	for (index0 = 0; index0 < Num_Frames_OneTime; index0++)
	{
		for (index1 = 0; index1 < CW_Len; index1++)
		{
			u1 = RandomModule(AWGN->seed);
			u2 = RandomModule(AWGN->seed);

			temp = (float)sqrt((float)(-2) * log((float)1 - u1));
			*(Channel_Out + index1 * Num_Frames_OneTime + index0) = (AWGN->sigma) * sin(2 * PI * u2) * temp + 1.0 - 2 * (*(CodeWord + index1 * Num_Frames_OneTime + index0));//产生高斯白噪声信号(https://www.cnblogs.com/tsingke/p/6194737.html)
		}
	}
}

/*

*/
float RandomModule(int* seed)
{
	float temp = 0.0;
	seed[0] = (seed[0] * 249) % 61967;
	seed[1] = (seed[1] * 251) % 63443;
	seed[2] = (seed[2] * 252) % 63599;
	temp = (((float)seed[0]) / ((float)61967)) + (((float)seed[1]) / ((float)63443))
		+ (((float)seed[2]) / ((float)63599));
	temp -= (int)temp;
	return (temp);
}
